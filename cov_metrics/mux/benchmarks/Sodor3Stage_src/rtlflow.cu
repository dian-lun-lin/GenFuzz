#include "hip/hip_runtime.h"

#include <taskflow.hpp>

#include <cuda/algorithm/reduce.hpp>

#include "rtlflow.h"


#include "Top.h"


#include <hiprand.h>

#include <assert.h>

// begin of namespace RF =====================================
namespace RF {
    inline
    hipError_t checkCuda(hipError_t result) {
        if (result != hipSuccess) {
            using namespace std::literals::string_literals;
            throw std::runtime_error("CUDA Runtime Error: "s + hipGetErrorString(result));
        }
        return result;
    }
    
    __global__ void _eval_settle(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals);
    
    RTLflow::RTLflow(Top* dut): dut{dut} {
        checkCuda(hipMallocManaged(&_csignals, BATCH_SIZE * cuda_cmem_size * sizeof(CData)));
        checkCuda(hipMallocManaged(&_ssignals, BATCH_SIZE * cuda_smem_size * sizeof(SData)));
        checkCuda(hipMallocManaged(&_qsignals, BATCH_SIZE * cuda_qmem_size * sizeof(QData)));
        checkCuda(hipMallocManaged(&_isignals, BATCH_SIZE * cuda_imem_size * sizeof(IData)));
        checkCuda(hipMallocManaged(&change, BATCH_SIZE * sizeof(IData)));
        checkCuda(hipMallocManaged(&done, BATCH_SIZE * sizeof(bool)));
        checkCuda(hipMemset(change, 1, BATCH_SIZE * sizeof(IData)));
        checkCuda(hipMemset(done, 0, BATCH_SIZE * sizeof(bool)));
    }
    RTLflow::RTLflow(const RTLflow& rtlflow) {
        dut = rtlflow.dut;
        checkCuda(hipMallocManaged(&_csignals, BATCH_SIZE * cuda_cmem_size * sizeof(CData)));
        checkCuda(hipMallocManaged(&_ssignals, BATCH_SIZE * cuda_smem_size * sizeof(SData)));
        checkCuda(hipMallocManaged(&_qsignals, BATCH_SIZE * cuda_qmem_size * sizeof(QData)));
        checkCuda(hipMallocManaged(&_isignals, BATCH_SIZE * cuda_imem_size * sizeof(IData)));
        checkCuda(hipMallocManaged(&change, BATCH_SIZE * sizeof(IData)));
        checkCuda(hipMallocManaged(&done, BATCH_SIZE * sizeof(bool)));
        checkCuda(hipMemset(change, 1, BATCH_SIZE * sizeof(IData)));
        checkCuda(hipMemset(done, 0, BATCH_SIZE * sizeof(bool)));
    }
    RTLflow::~RTLflow() {
        checkCuda(hipFree(_csignals));
        checkCuda(hipFree(_ssignals));
        checkCuda(hipFree(_qsignals));
        checkCuda(hipFree(_isignals));
        checkCuda(hipFree(change));
        checkCuda(hipFree(done));
    }
    
    // idx: index of testbenches
    CData* RTLflow::get(CDataLoc cdl, size_t idx) {
        return _csignals + idx * cdl.size + cdl.memloc;
    }
    SData* RTLflow::get(SDataLoc sdl, size_t idx) {
        return _ssignals + idx * sdl.size + sdl.memloc;
    }
    QData* RTLflow::get(QDataLoc qdl, size_t idx) {
        return _qsignals + idx * qdl.size + qdl.memloc;
    }
    IData* RTLflow::get(IDataLoc idl, size_t idx) {
        return _isignals + idx * idl.size + idl.memloc;
    }
    
    void RTLflow::_randomize() {
        hiprandGenerator_t generator;
        hiprandCreateGenerator(&generator,HIPRAND_RNG_PSEUDO_XORWOW);
        hiprandSetPseudoRandomGeneratorSeed(generator,(int)time(NULL));
        hiprandGenerate(generator, (unsigned int*)_csignals, BATCH_SIZE * cuda_cmem_size / 4);
        hiprandGenerate(generator, (unsigned int*)_ssignals, BATCH_SIZE * cuda_smem_size / 2);
        hiprandGenerate(generator, (unsigned int*)_isignals, BATCH_SIZE * cuda_imem_size);
        hiprandGenerate(generator, (unsigned int*)_qsignals, BATCH_SIZE * cuda_qmem_size * 2);
    }
    
    void RTLflow::run() { _executor.run(_taskflow).wait(); }
    
    tf::Taskflow& RTLflow::taskflow() { return _taskflow; }
    
    void RTLflow::_ctor_var_reset(Top__Syms* VlSymsp) {
        size_t offsetc{0};
        size_t offsets{0};
        size_t offseti{0};
        size_t offsetq{0};
        offsetc = 0;
        offsets = 0;
        offseti = 0;
        offsetq = 0;
        VlSymsp->TOPp->_ctor_var_reset(_csignals + offsetc, _ssignals + offsets, _isignals + offseti, _qsignals + offsetq);
    }
    
    void RTLflow::initialize() {
        Top__Syms* VlSymsp = dut-> __VlSymsp;
         _ctor_var_reset(VlSymsp);
        const size_t num_threads = (BATCH_SIZE < 128) ? BATCH_SIZE : 128;
        const size_t num_blocks = (num_threads < 128) ? 1 : BATCH_SIZE / num_threads;
        auto reset_cut = _cudaflow.memset(change, 1, sizeof(IData) * BATCH_SIZE);
        auto change_cut = _cudaflow.kernel(num_blocks, num_threads, 0, _change_request, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change);
        auto last_assign_cut = _cudaflow.kernel(num_blocks, num_threads, 0, _last_assign, VlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        auto reduce_cut = _cudaflow.reduce(change, change + BATCH_SIZE, change, [] __device__ (IData a, IData b){ return a | b; });
        last_assign_cut.precede(change_cut);
        
        change_cut.precede(reduce_cut);
        
        auto id_4_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__4, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_4");
        auto id_6_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__6, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_6");
        auto id_7_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__7, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_7");
        auto id_8_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__8, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_8");
        auto id_9_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__9, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_9");
        auto id_10_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__10, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_10");
        auto id_16_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__16, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_16");
        auto id_17_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__17, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_17");
        auto id_18_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__18, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_18");
        auto id_19_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__19, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_19");
        auto id_20_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__20, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_20");
        auto id_21_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__21, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_21");
        auto id_22_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__22, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_22");
        auto id_23_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__23, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_23");
        auto id_25_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__25, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_25");
        auto id_26_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__26, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_26");
        auto id_27_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__27, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_27");
        auto id_29_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__29, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_29");
        auto id_30_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__30, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_30");
        auto id_31_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__31, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_31");
        auto id_32_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__32, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_32");
        auto id_1_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__1, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_1");
        auto id_2_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__2, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_2");
        auto id_24_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__24, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_24");
        auto id_28_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__28, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_28");
        auto id_36_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__36, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_36");
        auto id_37_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__37, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_37");
        auto id_38_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__38, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_38");
        auto id_40_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__40, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_40");
        auto id_41_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__41, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_41");
        auto id_42_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__42, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_42");
        auto id_43_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__43, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_43");
        auto id_44_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__44, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_44");
        auto id_45_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__45, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_45");
        auto id_46_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__46, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_46");
        auto id_47_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__47, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_47");
        auto id_48_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__48, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_48");
        auto id_49_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__49, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_49");
        auto id_50_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__50, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_50");
        auto id_34_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__34, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_34");
        auto id_35_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__35, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_35");
        auto id_39_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__39, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_39");
        auto id_52_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__52, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_52");
        auto id_5_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__5, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_5");
        auto id_11_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__11, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_11");
        auto id_12_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__12, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_12");
        auto id_13_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__13, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_13");
        auto id_14_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__14, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_14");
        auto id_15_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__15, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_15");
        auto id_51_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__51, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_51");
        auto id_53_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__53, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_53");
        auto id_54_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__54, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_54");
        auto id_55_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__55, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_55");
        auto id_56_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__56, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_56");
        auto id_57_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__57, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_57");
        auto id_58_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__58, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_58");
        auto id_59_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__59, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_59");
        auto id_60_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__60, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_60");
        auto id_61_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__61, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_61");
        auto id_62_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__62, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_62");
        auto id_63_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__63, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_63");
        auto id_64_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__64, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_64");
        auto id_65_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__65, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_65");
        auto id_66_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__66, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_66");
        auto id_67_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__67, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_67");
        auto id_68_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__68, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_68");
        auto id_69_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__69, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_69");
        auto id_70_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__70, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_70");
        auto id_71_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__71, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_71");
        auto id_72_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__72, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_72");
        id_4_cut.precede(id_16_cut);
        id_4_cut.precede(id_25_cut);
        id_4_cut.precede(id_26_cut);
        id_4_cut.precede(id_27_cut);
        id_4_cut.precede(id_29_cut);
        id_4_cut.precede(id_30_cut);
        id_4_cut.precede(id_31_cut);
        id_4_cut.precede(id_32_cut);
        reset_cut.precede(id_4_cut);
        id_6_cut.precede(id_53_cut);
        id_6_cut.precede(id_54_cut);
        reset_cut.precede(id_6_cut);
        id_7_cut.precede(id_11_cut);
        id_7_cut.precede(id_54_cut);
        id_7_cut.precede(id_55_cut);
        id_7_cut.precede(id_56_cut);
        id_7_cut.precede(id_57_cut);
        id_7_cut.precede(id_58_cut);
        id_7_cut.precede(id_59_cut);
        reset_cut.precede(id_7_cut);
        id_8_cut.precede(id_60_cut);
        reset_cut.precede(id_8_cut);
        id_9_cut.precede(id_61_cut);
        reset_cut.precede(id_9_cut);
        id_10_cut.precede(id_62_cut);
        reset_cut.precede(id_10_cut);
        id_16_cut.precede(id_37_cut);
        id_16_cut.precede(id_42_cut);
        id_17_cut.precede(id_2_cut);
        id_17_cut.precede(id_34_cut);
        reset_cut.precede(id_17_cut);
        id_18_cut.precede(id_2_cut);
        id_18_cut.precede(id_34_cut);
        reset_cut.precede(id_18_cut);
        id_19_cut.precede(id_2_cut);
        id_19_cut.precede(id_34_cut);
        reset_cut.precede(id_19_cut);
        id_20_cut.precede(id_2_cut);
        id_20_cut.precede(id_34_cut);
        reset_cut.precede(id_20_cut);
        id_21_cut.precede(id_2_cut);
        id_21_cut.precede(id_34_cut);
        reset_cut.precede(id_21_cut);
        id_22_cut.precede(id_2_cut);
        id_22_cut.precede(id_34_cut);
        reset_cut.precede(id_22_cut);
        id_23_cut.precede(id_2_cut);
        id_23_cut.precede(id_34_cut);
        reset_cut.precede(id_23_cut);
        id_25_cut.precede(id_47_cut);
        id_26_cut.precede(id_28_cut);
        id_27_cut.precede(id_28_cut);
        id_29_cut.precede(id_24_cut);
        id_29_cut.precede(id_28_cut);
        id_30_cut.precede(id_24_cut);
        id_30_cut.precede(id_28_cut);
        id_31_cut.precede(id_24_cut);
        id_31_cut.precede(id_28_cut);
        id_32_cut.precede(id_24_cut);
        id_32_cut.precede(id_28_cut);
        id_1_cut.precede(id_2_cut);
        id_1_cut.precede(id_34_cut);
        reset_cut.precede(id_1_cut);
        id_2_cut.precede(id_35_cut);
        id_24_cut.precede(id_47_cut);
        id_28_cut.precede(id_36_cut);
        id_28_cut.precede(id_38_cut);
        id_28_cut.precede(id_40_cut);
        id_28_cut.precede(id_41_cut);
        id_28_cut.precede(id_42_cut);
        id_28_cut.precede(id_43_cut);
        id_28_cut.precede(id_44_cut);
        id_28_cut.precede(id_45_cut);
        id_28_cut.precede(id_47_cut);
        id_36_cut.precede(id_37_cut);
        id_36_cut.precede(id_46_cut);
        id_36_cut.precede(id_34_cut);
        id_36_cut.precede(id_39_cut);
        id_37_cut.precede(id_48_cut);
        id_37_cut.precede(id_52_cut);
        id_38_cut.precede(id_46_cut);
        id_40_cut.precede(id_52_cut);
        id_41_cut.precede(id_52_cut);
        id_42_cut.precede(id_48_cut);
        id_43_cut.precede(id_52_cut);
        id_44_cut.precede(id_52_cut);
        id_45_cut.precede(id_34_cut);
        id_46_cut.precede(id_52_cut);
        id_47_cut.precede(id_48_cut);
        id_47_cut.precede(id_34_cut);
        id_47_cut.precede(id_39_cut);
        id_48_cut.precede(id_49_cut);
        id_48_cut.precede(id_50_cut);
        id_49_cut.precede(last_assign_cut);
        id_50_cut.precede(last_assign_cut);
        id_34_cut.precede(id_35_cut);
        id_34_cut.precede(id_52_cut);
        id_35_cut.precede(id_5_cut);
        id_35_cut.precede(id_51_cut);
        id_39_cut.precede(last_assign_cut);
        id_52_cut.precede(id_5_cut);
        id_52_cut.precede(id_51_cut);
        id_5_cut.precede(id_11_cut);
        id_5_cut.precede(id_12_cut);
        id_5_cut.precede(id_13_cut);
        id_5_cut.precede(id_14_cut);
        id_5_cut.precede(id_15_cut);
        id_5_cut.precede(id_53_cut);
        id_5_cut.precede(id_54_cut);
        id_5_cut.precede(id_55_cut);
        id_5_cut.precede(id_56_cut);
        id_5_cut.precede(id_57_cut);
        id_5_cut.precede(id_58_cut);
        id_5_cut.precede(id_59_cut);
        id_5_cut.precede(id_60_cut);
        id_5_cut.precede(id_61_cut);
        id_5_cut.precede(id_62_cut);
        id_5_cut.precede(id_63_cut);
        id_5_cut.precede(id_64_cut);
        id_5_cut.precede(id_65_cut);
        id_5_cut.precede(id_66_cut);
        id_5_cut.precede(id_67_cut);
        id_5_cut.precede(id_68_cut);
        id_5_cut.precede(id_69_cut);
        id_5_cut.precede(id_70_cut);
        id_5_cut.precede(id_71_cut);
        id_5_cut.precede(id_72_cut);
        id_11_cut.precede(last_assign_cut);
        id_12_cut.precede(last_assign_cut);
        id_13_cut.precede(last_assign_cut);
        id_14_cut.precede(last_assign_cut);
        id_15_cut.precede(last_assign_cut);
        id_51_cut.precede(last_assign_cut);
        id_53_cut.precede(last_assign_cut);
        id_54_cut.precede(last_assign_cut);
        id_55_cut.precede(last_assign_cut);
        id_56_cut.precede(last_assign_cut);
        id_57_cut.precede(last_assign_cut);
        id_58_cut.precede(last_assign_cut);
        id_59_cut.precede(last_assign_cut);
        id_60_cut.precede(last_assign_cut);
        id_61_cut.precede(last_assign_cut);
        id_62_cut.precede(last_assign_cut);
        id_63_cut.precede(last_assign_cut);
        id_64_cut.precede(last_assign_cut);
        id_65_cut.precede(last_assign_cut);
        id_66_cut.precede(last_assign_cut);
        id_67_cut.precede(last_assign_cut);
        id_68_cut.precede(last_assign_cut);
        id_69_cut.precede(last_assign_cut);
        id_70_cut.precede(last_assign_cut);
        id_71_cut.precede(last_assign_cut);
        id_72_cut.precede(last_assign_cut);
        auto start_t = _taskflow.emplace([=](){
                if(VL_UNLIKELY(!init)) {
                    Top::_eval_initial(VlSymsp, _csignals, _ssignals, _isignals, _qsignals);
                    int device;
                    checkCuda(hipGetDevice(&device));
                    checkCuda(hipMemPrefetchAsync(_csignals, BATCH_SIZE * cuda_cmem_size * sizeof(CData), device));
                    checkCuda(hipMemPrefetchAsync(_ssignals, BATCH_SIZE * cuda_smem_size * sizeof(SData), device));
                    checkCuda(hipMemPrefetchAsync(_isignals, BATCH_SIZE * cuda_imem_size * sizeof(IData), device));
                    checkCuda(hipMemPrefetchAsync(_qsignals, BATCH_SIZE * cuda_qmem_size * sizeof(QData), device));
                    checkCuda(hipMemPrefetchAsync(change, BATCH_SIZE * sizeof(IData), device));
                    checkCuda(hipMemPrefetchAsync(done, BATCH_SIZE * sizeof(bool), device));
                    init = true;
                    return 0;
                }
                else {
                    return 1;
                }
        });
        
        auto init_detect_t = _taskflow.emplace([=](){
                if(++loop > 100) {
                    _change_request<<<num_blocks, num_threads, 0>>>(VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change);
                    checkCuda(hipDeviceSynchronize());
                    VL_FATAL_MT("add.v", 2, "",
                        "Verilated model didn't converge"
                        "- See https://verilator.org/warn/DIDNOTCONVERGE");
                }
                return (bool)change[0];
        });
        auto init_sim_t = _taskflow.emplace([=](){
                _eval_settle<<<num_blocks, num_threads, 0>>>(VlSymsp, _csignals, _ssignals, _isignals, _qsignals);
                checkCuda(hipDeviceSynchronize());
                _cudaflow.offload();
        });
        auto sim_t = _taskflow.emplace([=](){
                _cudaflow.offload();
        });
        auto end_t = _taskflow.emplace([=](){
                loop = 0;
        });
        
        auto detect_t = _taskflow.emplace([=](){
                if(++loop > 100) {
                    _change_request<<<num_blocks, num_threads, 0>>>(VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change);
                    checkCuda(hipDeviceSynchronize());
                    VL_FATAL_MT("add.v", 2, "",
                        "Verilated model didn't converge"
                        "- See https://verilator.org/warn/DIDNOTCONVERGE");
                }
                return (bool)change[0];
        });
        start_t.precede(init_sim_t, sim_t);
        init_sim_t.precede(init_detect_t);
        init_detect_t.precede(end_t, init_sim_t);
        
        sim_t.precede(detect_t);
        detect_t.precede(end_t, sim_t);
        std::ofstream ofs("./cudaflow.out");
        _cudaflow.dump(ofs);
    }
} // end of namespace RF ==================================== 
