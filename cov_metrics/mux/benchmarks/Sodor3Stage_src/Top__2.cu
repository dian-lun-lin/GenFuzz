#include "hip/hip_runtime.h"
// Verilated -*- C++ -*-
// DESCRIPTION: Verilator output: Design implementation internals
// See Top.h for the primary calling header

#include "Top.h"
#include "Top__Syms.h"
// begin of namespace RF =====================================
namespace RF {

    //==========

    __device__
    void Top::_combo__TOP__67(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_combo__TOP__67\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Body
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2475] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x17U] 
                      >> 9U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2474])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2478] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x17U] 
                         >> 8U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2477])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2481] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x17U] 
                      >> 8U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2480])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2484] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x16U] 
                         >> 0x1eU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2483])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2487] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x16U] 
                      >> 0x1eU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2486])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2493] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x16U] 
                      >> 0x1dU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2492])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2700] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x15U] 
                         >> 0x1aU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2699])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2703] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x15U] 
                      >> 0x1aU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2702])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2772] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x15U] 
                         >> 0xeU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2771])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2775] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x15U] 
                      >> 0xeU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2774])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2796] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0xaU] 
                         >> 9U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2795])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2799] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0xaU] 
                      >> 9U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2798])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2976] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[8U] 
                         >> 0x17U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2975])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2979] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[8U] 
                      >> 0x17U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2978])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3126] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[7U] 
                         >> 0x1eU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3125])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3129] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[7U] 
                      >> 0x1eU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3128])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3276] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[7U] 
                         >> 5U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3275])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3279] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[7U] 
                      >> 5U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3278])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3426] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[6U] 
                         >> 0xcU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3425])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3429] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[6U] 
                      >> 0xcU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3428])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3576] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0U] 
                         >> 0x1fU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3575])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3579] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0U] 
                      >> 0x1fU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3578])));
    }

    __device__
    void Top::_sequent__TOP__68(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_sequent__TOP__68\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Body
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2495] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2496])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2495] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2497];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2498] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2499])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2498] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2500];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3107] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3108])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3107] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3109];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3110] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3111])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3110] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3112];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3257] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3258])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3257] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3259];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3260] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3261])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3260] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3262];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3407] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3408])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3407] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3409];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3410] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3411])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3410] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3412];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3557] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3558])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3557] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3559];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3560] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3561])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3560] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3562];
        }
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2497] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2495])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2500] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2498])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3109] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3107])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3112] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3110])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3259] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3257])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3262] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3260])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3409] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3407])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3412] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3410])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3559] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3557])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3562] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3560])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1832] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2495]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2498]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1934] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3107]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3110]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1959] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3257]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3260]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1984] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3407]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3410]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2009] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3557]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3560]));
    }

    __device__
    void Top::_combo__TOP__69(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_combo__TOP__69\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Body
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2490] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x16U] 
                         >> 0x1dU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2489])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2496] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x16U] 
                         >> 0x1cU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2495])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2499] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x16U] 
                      >> 0x1cU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2498])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2505] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x16U] 
                      >> 0x1bU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2504])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2511] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x16U] 
                      >> 0x1aU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2510])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2517] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x16U] 
                      >> 0x19U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2516])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2718] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x15U] 
                         >> 0x17U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2717])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2721] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x15U] 
                      >> 0x17U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2720])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2778] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0xaU] 
                         >> 0xcU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2777])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2781] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0xaU] 
                      >> 0xcU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2780])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2802] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0xaU] 
                         >> 8U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2801])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2805] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0xaU] 
                      >> 8U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2804])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3108] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[8U] 
                         >> 1U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3107])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3111] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[8U] 
                      >> 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3110])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3258] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[7U] 
                         >> 8U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3257])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3261] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[7U] 
                      >> 8U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3260])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3408] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[6U] 
                         >> 0xfU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3407])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3411] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[6U] 
                      >> 0xfU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3410])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3558] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[1U] 
                         >> 2U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3557])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3561] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[1U] 
                      >> 2U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3560])));
    }

    __device__
    void Top::_sequent__TOP__70(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_sequent__TOP__70\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Body
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2735] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2736])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2735] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2737];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2738] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2739])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2738] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2740];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2981] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2982])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2981] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2983];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2984] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2985])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2984] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2986];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3131] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3132])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3131] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3133];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3134] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3135])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3134] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3136];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3281] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3282])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3281] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3283];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3284] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3285])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3284] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3286];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3431] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3432])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3431] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3433];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3434] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3435])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3434] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3436];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3581] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3582])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3581] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3583];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3584] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3585])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3584] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3586];
        }
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2737] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2735])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2740] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2738])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2983] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2981])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2986] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2984])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3133] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3131])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3136] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3134])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3283] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3281])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3286] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3284])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3433] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3431])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3436] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3434])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3583] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3581])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3586] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3584])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1872] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2735]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2738]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1913] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2981]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2984]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1938] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3131]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3134]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1963] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3281]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3284]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1988] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3431]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3434]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2013] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3581]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3584]));
    }

    __device__
    void Top::_combo__TOP__71(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_combo__TOP__71\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Body
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2502] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x16U] 
                         >> 0x1bU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2501])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2508] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x16U] 
                         >> 0x1aU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2507])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2736] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x15U] 
                         >> 0x14U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2735])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2739] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x15U] 
                      >> 0x14U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2738])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2808] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0xaU] 
                         >> 7U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2807])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2811] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0xaU] 
                      >> 7U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2810])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2982] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[8U] 
                         >> 0x16U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2981])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2985] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[8U] 
                      >> 0x16U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2984])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3132] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[7U] 
                         >> 0x1dU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3131])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3135] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[7U] 
                      >> 0x1dU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3134])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3282] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[7U] 
                         >> 4U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3281])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3285] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[7U] 
                      >> 4U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3284])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3432] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[6U] 
                         >> 0xbU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3431])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3435] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[6U] 
                      >> 0xbU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3434])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3582] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0U] 
                         >> 0x18U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3581])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3585] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0U] 
                      >> 0x18U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3584])));
    }

    __device__
    void Top::_sequent__TOP__72(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_sequent__TOP__72\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Body
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2519] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2520])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2519] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2521];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2522] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2523])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2522] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2524];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3113] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3114])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3113] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3115];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3116] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3117])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3116] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3118];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3263] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3264])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3263] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3265];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3266] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3267])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3266] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3268];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3413] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3414])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3413] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3415];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3416] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3417])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3416] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3418];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3563] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3564])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3563] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3565];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3566] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3567])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3566] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3568];
        }
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2521] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2519])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2524] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2522])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3115] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3113])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3118] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3116])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3265] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3263])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3268] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3266])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3415] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3413])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3418] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3416])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3565] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3563])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3568] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3566])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1836] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2519]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2522]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1935] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3113]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3116]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1960] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3263]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3266]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1985] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3413]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3416]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2010] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3563]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3566]));
    }

    __device__
    void Top::_combo__TOP__73(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_combo__TOP__73\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Body
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2514] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x16U] 
                         >> 0x19U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2513])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2520] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x16U] 
                         >> 0x18U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2519])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2523] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x16U] 
                      >> 0x18U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2522])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2529] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x16U] 
                      >> 0x17U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2528])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2535] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x16U] 
                      >> 0x16U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2534])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2541] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x16U] 
                      >> 0x15U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2540])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2742] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x15U] 
                         >> 0x13U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2741])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2745] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x15U] 
                      >> 0x13U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2744])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2784] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0xaU] 
                         >> 0xbU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2783])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2787] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0xaU] 
                      >> 0xbU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2786])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2814] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0xaU] 
                         >> 6U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2813])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2817] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0xaU] 
                      >> 6U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2816])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3114] 
            = (1U & ((~ (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[8U]) 
                     | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3113])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3117] 
            = (1U & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[8U] 
                     | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3116])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3264] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[7U] 
                         >> 7U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3263])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3267] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[7U] 
                      >> 7U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3266])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3414] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[6U] 
                         >> 0xeU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3413])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3417] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[6U] 
                      >> 0xeU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3416])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3564] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[1U] 
                         >> 1U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3563])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3567] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[1U] 
                      >> 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3566])));
    }

    __device__
    void Top::_sequent__TOP__74(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_sequent__TOP__74\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Body
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2747] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2748])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2747] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2749];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2750] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2751])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2750] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2752];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2987] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2988])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2987] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2989];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2990] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2991])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2990] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2992];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3137] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3138])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3137] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3139];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3140] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3141])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3140] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3142];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3287] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3288])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3287] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3289];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3290] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3291])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3290] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3292];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3437] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3438])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3437] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3439];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3440] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3441])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3440] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3442];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3587] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3588])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3587] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3589];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3590] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3591])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3590] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3592];
        }
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2749] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2747])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2752] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2750])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2989] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2987])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2992] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2990])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3139] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3137])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3142] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3140])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3289] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3287])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3292] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3290])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3439] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3437])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3442] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3440])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3589] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3587])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3592] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3590])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1874] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2747]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2750]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1914] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2987]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2990]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1939] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3137]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3140]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1964] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3287]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3290]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1989] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3437]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3440]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2014] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3587]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3590]));
    }

    __device__
    void Top::_combo__TOP__75(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_combo__TOP__75\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Body
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2526] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x16U] 
                         >> 0x17U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2525])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2532] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x16U] 
                         >> 0x16U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2531])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2748] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x15U] 
                         >> 0x12U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2747])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2751] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x15U] 
                      >> 0x12U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2750])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2820] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0xaU] 
                         >> 5U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2819])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2823] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0xaU] 
                      >> 5U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2822])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2988] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[8U] 
                         >> 0x15U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2987])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2991] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[8U] 
                      >> 0x15U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2990])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3138] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[7U] 
                         >> 0x1cU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3137])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3141] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[7U] 
                      >> 0x1cU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3140])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3288] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[7U] 
                         >> 3U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3287])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3291] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[7U] 
                      >> 3U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3290])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3438] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[6U] 
                         >> 0xaU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3437])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3441] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[6U] 
                      >> 0xaU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3440])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3588] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0U] 
                         >> 0x17U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3587])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3591] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0U] 
                      >> 0x17U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3590])));
    }

    __device__
    void Top::_sequent__TOP__76(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_sequent__TOP__76\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Body
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2543] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2544])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2543] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2545];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2546] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2547])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2546] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2548];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3119] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3120])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3119] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3121];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3122] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3123])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3122] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3124];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3269] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3270])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3269] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3271];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3272] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3273])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3272] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3274];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3419] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3420])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3419] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3421];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3422] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3423])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3422] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3424];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3569] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3570])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3569] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3571];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3572] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3573])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3572] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3574];
        }
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2545] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2543])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2548] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2546])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3121] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3119])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3124] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3122])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3271] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3269])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3274] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3272])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3421] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3419])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3424] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3422])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3571] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3569])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3574] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3572])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1840] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2543]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2546]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1936] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3119]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3122]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1961] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3269]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3272]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1986] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3419]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3422]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2011] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3569]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3572]));
    }

    __device__
    void Top::_combo__TOP__77(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_combo__TOP__77\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Body
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2538] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x16U] 
                         >> 0x15U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2537])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2544] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x16U] 
                         >> 0x14U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2543])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2547] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x16U] 
                      >> 0x14U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2546])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2553] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x16U] 
                      >> 0x13U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2552])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2559] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x16U] 
                      >> 0x12U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2558])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2565] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x16U] 
                      >> 0x11U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2564])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2754] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x15U] 
                         >> 0x11U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2753])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2757] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x15U] 
                      >> 0x11U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2756])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2790] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0xaU] 
                         >> 0xaU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2789])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2793] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0xaU] 
                      >> 0xaU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2792])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2826] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0xaU] 
                         >> 4U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2825])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2829] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0xaU] 
                      >> 4U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2828])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3120] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[7U] 
                         >> 0x1fU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3119])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3123] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[7U] 
                      >> 0x1fU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3122])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3270] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[7U] 
                         >> 6U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3269])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3273] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[7U] 
                      >> 6U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3272])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3420] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[6U] 
                         >> 0xdU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3419])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3423] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[6U] 
                      >> 0xdU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3422])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3570] 
            = (1U & ((~ (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[1U]) 
                     | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3569])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3573] 
            = (1U & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[1U] 
                     | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3572])));
    }

    __device__
    void Top::_sequent__TOP__78(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_sequent__TOP__78\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Body
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2765] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2766])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2765] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2767];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2768] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2769])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2768] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2770];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2993] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2994])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2993] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2995];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2996] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2997])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2996] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2998];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3143] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3144])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3143] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3145];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3146] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3147])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3146] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3148];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3293] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3294])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3293] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3295];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3296] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3297])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3296] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3298];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3443] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3444])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3443] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3445];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3446] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3447])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3446] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3448];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3593] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3594])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3593] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3595];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3596] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3597])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3596] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3598];
        }
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2767] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2765])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2770] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2768])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2995] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2993])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2998] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2996])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3145] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3143])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3148] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3146])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3295] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3293])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3298] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3296])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3445] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3443])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3448] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3446])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3595] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3593])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3598] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3596])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1877] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2765]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2768]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1915] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2993]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2996]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1940] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3143]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3146]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1965] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3293]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3296]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1990] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3443]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3446]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2015] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3593]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3596]));
    }

    __device__
    void Top::_combo__TOP__79(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_combo__TOP__79\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Body
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2556] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x16U] 
                         >> 0x12U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2555])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2562] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x16U] 
                         >> 0x11U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2561])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2766] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x15U] 
                         >> 0xfU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2765])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2769] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x15U] 
                      >> 0xfU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2768])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2838] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0xaU] 
                         >> 2U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2837])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2841] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0xaU] 
                      >> 2U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2840])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2994] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[8U] 
                         >> 0x14U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2993])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2997] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[8U] 
                      >> 0x14U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2996])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3144] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[7U] 
                         >> 0x1bU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3143])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3147] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[7U] 
                      >> 0x1bU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3146])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3294] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[7U] 
                         >> 2U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3293])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3297] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[7U] 
                      >> 2U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3296])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3444] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[6U] 
                         >> 9U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3443])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3447] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[6U] 
                      >> 9U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3446])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3594] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0U] 
                         >> 0x13U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3593])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3597] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0U] 
                      >> 0x13U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3596])));
    }

    __device__
    void Top::_sequent__TOP__80(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_sequent__TOP__80\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Body
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2567] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2568])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2567] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2569];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2570] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2571])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2570] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2572];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2999] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3000])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2999] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3001];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3002] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3003])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3002] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3004];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3149] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3150])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3149] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3151];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3152] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3153])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3152] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3154];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3299] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3300])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3299] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3301];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3302] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3303])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3302] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3304];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3449] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3450])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3449] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3451];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3452] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3453])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3452] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3454];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3599] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3600])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3599] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3601];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3602] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3603])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3602] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3604];
        }
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2569] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2567])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2572] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2570])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3001] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2999])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3004] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3002])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3151] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3149])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3154] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3152])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3301] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3299])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3304] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3302])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3451] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3449])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3454] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3452])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3601] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3599])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3604] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3602])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1844] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2567]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2570]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1916] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2999]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3002]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1941] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3149]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3152]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1966] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3299]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3302]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1991] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3449]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3452]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2016] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3599]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3602]));
    }

    __device__
    void Top::_combo__TOP__81(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_combo__TOP__81\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Body
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2568] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x16U] 
                         >> 0x10U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2567])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2571] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x16U] 
                      >> 0x10U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2570])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2574] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x16U] 
                         >> 0xfU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2573])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2577] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x16U] 
                      >> 0xfU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2576])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2844] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0xaU] 
                         >> 1U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2843])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2847] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0xaU] 
                      >> 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2846])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3000] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[8U] 
                         >> 0x13U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2999])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3003] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[8U] 
                      >> 0x13U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3002])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3150] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[7U] 
                         >> 0x1aU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3149])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3153] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[7U] 
                      >> 0x1aU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3152])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3300] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[7U] 
                         >> 1U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3299])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3303] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[7U] 
                      >> 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3302])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3450] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[6U] 
                         >> 8U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3449])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3453] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[6U] 
                      >> 8U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3452])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3600] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0U] 
                         >> 7U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3599])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3603] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0U] 
                      >> 7U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3602])));
    }

    __device__
    void Top::_sequent__TOP__82(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_sequent__TOP__82\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Body
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2579] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2580])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2579] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2581];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2582] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2583])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2582] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2584];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3005] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3006])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3005] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3007];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3008] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3009])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3008] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3010];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3155] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3156])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3155] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3157];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3158] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3159])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3158] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3160];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3305] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3306])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3305] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3307];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3308] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3309])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3308] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3310];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3455] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3456])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3455] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3457];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3458] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3459])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3458] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3460];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3605] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3606])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3605] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3607];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3608] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3609])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3608] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3610];
        }
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2581] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2579])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2584] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2582])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3007] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3005])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3010] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3008])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3157] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3155])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3160] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3158])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3307] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3305])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3310] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3308])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3457] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3455])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3460] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3458])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3607] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3605])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3610] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3608])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1846] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2579]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2582]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1917] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3005]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3008]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1942] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3155]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3158]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1967] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3305]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3308]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1992] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3455]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3458]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2017] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3605]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3608]));
    }

    __device__
    void Top::_combo__TOP__83(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_combo__TOP__83\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Body
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2580] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x16U] 
                         >> 0xeU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2579])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2583] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x16U] 
                      >> 0xeU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2582])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2586] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x16U] 
                         >> 0xdU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2585])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2589] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x16U] 
                      >> 0xdU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2588])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2850] 
            = (1U & ((~ (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0xaU]) 
                     | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2849])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2853] 
            = (1U & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0xaU] 
                     | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2852])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3006] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[8U] 
                         >> 0x12U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3005])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3009] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[8U] 
                      >> 0x12U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3008])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3156] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[7U] 
                         >> 0x19U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3155])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3159] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[7U] 
                      >> 0x19U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3158])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3306] 
            = (1U & ((~ (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[7U]) 
                     | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3305])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3309] 
            = (1U & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[7U] 
                     | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3308])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3456] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[6U] 
                         >> 7U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3455])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3459] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[6U] 
                      >> 7U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3458])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3606] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0U] 
                         >> 6U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3605])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3609] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0U] 
                      >> 6U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3608])));
    }

    __device__
    void Top::_sequent__TOP__84(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_sequent__TOP__84\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Body
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2591] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2592])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2591] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2593];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2594] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2595])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2594] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2596];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3011] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3012])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3011] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3013];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3014] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3015])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3014] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3016];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3161] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3162])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3161] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3163];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3164] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3165])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3164] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3166];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3311] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3312])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3311] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3313];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3314] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3315])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3314] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3316];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3461] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3462])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3461] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3463];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3464] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3465])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3464] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3466];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3611] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3612])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3611] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3613];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3614] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3615])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3614] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3616];
        }
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2593] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2591])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2596] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2594])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3013] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3011])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3016] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3014])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3163] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3161])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3166] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3164])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3313] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3311])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3316] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3314])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3463] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3461])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3466] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3464])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3613] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3611])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3616] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3614])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1848] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2591]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2594]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1918] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3011]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3014]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1943] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3161]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3164]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1968] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3311]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3314]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1993] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3461]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3464]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2018] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3611]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3614]));
    }

    __device__
    void Top::_combo__TOP__85(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_combo__TOP__85\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Body
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2592] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x16U] 
                         >> 0xcU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2591])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2595] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x16U] 
                      >> 0xcU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2594])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2598] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x16U] 
                         >> 0xbU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2597])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2601] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x16U] 
                      >> 0xbU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2600])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2856] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[9U] 
                         >> 0x1fU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2855])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2859] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[9U] 
                      >> 0x1fU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2858])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3012] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[8U] 
                         >> 0x11U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3011])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3015] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[8U] 
                      >> 0x11U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3014])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3162] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[7U] 
                         >> 0x18U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3161])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3165] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[7U] 
                      >> 0x18U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3164])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3312] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[6U] 
                         >> 0x1fU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3311])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3315] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[6U] 
                      >> 0x1fU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3314])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3462] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[6U] 
                         >> 6U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3461])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3465] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[6U] 
                      >> 6U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3464])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3612] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0U] 
                         >> 2U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3611])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3615] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0U] 
                      >> 2U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3614])));
    }

    __device__
    void Top::_sequent__TOP__86(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_sequent__TOP__86\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Body
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2603] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2604])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2603] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2605];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2606] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2607])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2606] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2608];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2609] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2610])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2609] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2611];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2612] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2613])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2612] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2614];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2861] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2862])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2861] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2863];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2864] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2865])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2864] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2866];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3017] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3018])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3017] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3019];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3020] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3021])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3020] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3022];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3167] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3168])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3167] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3169];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3170] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3171])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3170] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3172];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3317] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3318])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3317] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3319];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3320] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3321])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3320] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3322];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3467] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3468])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3467] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3469];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3470] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3471])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3470] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3472];
        }
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2605] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2603])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2608] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2606])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2611] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2609])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2614] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2612])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2863] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2861])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2866] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2864])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3019] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3017])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3022] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3020])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3169] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3167])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3172] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3170])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3319] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3317])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3322] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3320])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3469] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3467])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3472] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3470])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1850] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2603]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2606]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1851] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2609]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2612]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1893] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2861]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2864]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1919] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3017]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3020]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1944] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3167]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3170]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1969] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3317]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3320]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1994] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3467]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3470]));
    }

    __device__
    void Top::_combo__TOP__87(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_combo__TOP__87\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Body
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2604] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x16U] 
                         >> 0xaU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2603])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2607] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x16U] 
                      >> 0xaU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2606])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2610] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x16U] 
                         >> 9U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2609])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2613] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x16U] 
                      >> 9U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2612])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2862] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[9U] 
                         >> 0x1eU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2861])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2865] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[9U] 
                      >> 0x1eU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2864])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3018] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[8U] 
                         >> 0x10U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3017])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3021] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[8U] 
                      >> 0x10U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3020])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3168] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[7U] 
                         >> 0x17U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3167])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3171] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[7U] 
                      >> 0x17U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3170])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3318] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[6U] 
                         >> 0x1eU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3317])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3321] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[6U] 
                      >> 0x1eU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3320])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3468] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[3U] 
                         >> 0x1eU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3467])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3471] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[3U] 
                      >> 0x1eU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3470])));
    }

    __device__
    void Top::_sequent__TOP__88(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_sequent__TOP__88\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Body
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2615] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2616])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2615] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2617];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2618] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2619])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2618] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2620];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2621] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2622])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2621] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2623];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2624] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2625])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2624] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2626];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2867] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2868])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2867] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2869];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2870] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2871])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2870] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2872];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3023] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3024])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3023] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3025];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3026] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3027])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3026] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3028];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3173] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3174])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3173] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3175];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3176] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3177])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3176] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3178];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3323] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3324])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3323] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3325];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3326] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3327])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3326] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3328];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3473] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3474])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3473] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3475];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3476] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3477])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3476] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3478];
        }
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2617] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2615])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2620] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2618])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2623] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2621])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2626] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2624])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2869] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2867])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2872] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2870])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3025] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3023])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3028] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3026])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3175] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3173])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3178] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3176])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3325] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3323])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3328] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3326])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3475] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3473])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3478] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3476])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1852] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2615]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2618]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1853] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2621]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2624]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1894] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2867]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2870]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1920] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3023]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3026]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1945] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3173]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3176]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1970] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3323]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3326]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1995] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3473]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3476]));
    }

    __device__
    void Top::_combo__TOP__89(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_combo__TOP__89\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Body
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2616] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x16U] 
                         >> 8U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2615])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2619] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x16U] 
                      >> 8U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2618])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2622] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x16U] 
                         >> 7U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2621])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2625] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x16U] 
                      >> 7U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2624])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2868] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[9U] 
                         >> 0x1dU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2867])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2871] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[9U] 
                      >> 0x1dU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2870])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3024] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[8U] 
                         >> 0xfU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3023])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3027] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[8U] 
                      >> 0xfU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3026])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3174] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[7U] 
                         >> 0x16U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3173])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3177] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[7U] 
                      >> 0x16U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3176])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3324] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[6U] 
                         >> 0x1dU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3323])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3327] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[6U] 
                      >> 0x1dU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3326])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3474] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[3U] 
                         >> 0x1aU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3473])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3477] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[3U] 
                      >> 0x1aU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3476])));
    }

    __device__
    void Top::_sequent__TOP__90(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_sequent__TOP__90\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Body
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2627] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2628])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2627] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2629];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2630] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2631])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2630] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2632];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2633] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2634])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2633] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2635];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2636] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2637])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2636] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2638];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2873] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2874])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2873] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2875];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2876] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2877])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2876] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2878];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3029] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3030])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3029] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3031];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3032] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3033])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3032] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3034];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3179] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3180])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3179] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3181];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3182] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3183])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3182] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3184];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3329] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3330])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3329] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3331];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3332] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3333])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3332] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3334];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3479] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3480])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3479] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3481];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3482] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3483])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3482] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3484];
        }
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2629] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2627])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2632] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2630])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2635] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2633])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2638] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2636])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2875] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2873])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2878] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2876])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3031] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3029])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3034] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3032])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3181] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3179])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3184] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3182])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3331] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3329])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3334] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3332])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3481] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3479])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3484] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3482])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1854] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2627]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2630]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1855] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2633]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2636]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1895] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2873]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2876]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1921] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3029]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3032]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1946] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3179]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3182]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1971] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3329]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3332]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1996] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3479]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3482]));
    }

    __device__
    void Top::_combo__TOP__91(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_combo__TOP__91\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Body
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2628] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x16U] 
                         >> 6U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2627])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2631] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x16U] 
                      >> 6U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2630])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2634] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x16U] 
                         >> 5U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2633])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2637] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x16U] 
                      >> 5U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2636])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2874] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[9U] 
                         >> 0x1cU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2873])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2877] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[9U] 
                      >> 0x1cU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2876])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3030] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[8U] 
                         >> 0xeU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3029])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3033] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[8U] 
                      >> 0xeU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3032])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3180] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[7U] 
                         >> 0x15U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3179])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3183] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[7U] 
                      >> 0x15U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3182])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3330] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[6U] 
                         >> 0x1cU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3329])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3333] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[6U] 
                      >> 0x1cU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3332])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3480] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[3U] 
                         >> 0xfU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3479])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3483] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[3U] 
                      >> 0xfU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3482])));
    }

    __device__
    void Top::_sequent__TOP__92(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_sequent__TOP__92\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Body
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2639] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2640])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2639] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2641];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2642] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2643])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2642] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2644];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2645] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2646])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2645] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2647];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2648] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2649])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2648] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2650];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2879] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2880])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2879] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2881];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2882] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2883])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2882] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2884];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3035] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3036])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3035] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3037];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3038] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3039])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3038] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3040];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3185] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3186])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3185] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3187];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3188] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3189])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3188] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3190];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3335] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3336])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3335] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3337];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3338] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3339])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3338] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3340];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3485] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3486])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3485] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3487];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3488] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3489])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3488] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3490];
        }
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2641] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2639])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2644] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2642])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2647] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2645])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2650] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2648])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2881] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2879])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2884] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2882])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3037] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3035])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3040] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3038])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3187] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3185])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3190] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3188])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3337] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3335])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3340] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3338])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3487] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3485])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3490] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3488])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1856] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2639]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2642]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1857] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2645]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2648]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1896] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2879]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2882]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1922] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3035]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3038]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1947] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3185]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3188]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1972] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3335]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3338]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1997] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3485]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3488]));
    }

    __device__
    void Top::_combo__TOP__93(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_combo__TOP__93\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Body
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2640] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x16U] 
                         >> 4U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2639])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2643] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x16U] 
                      >> 4U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2642])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2646] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x16U] 
                         >> 3U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2645])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2649] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x16U] 
                      >> 3U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2648])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2880] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[9U] 
                         >> 0x15U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2879])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2883] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[9U] 
                      >> 0x15U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2882])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3036] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[8U] 
                         >> 0xdU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3035])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3039] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[8U] 
                      >> 0xdU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3038])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3186] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[7U] 
                         >> 0x14U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3185])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3189] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[7U] 
                      >> 0x14U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3188])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3336] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[6U] 
                         >> 0x1bU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3335])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3339] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[6U] 
                      >> 0x1bU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3338])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3486] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[1U] 
                         >> 0xfU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3485])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3489] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[1U] 
                      >> 0xfU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3488])));
    }

    __device__
    void Top::_sequent__TOP__94(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_sequent__TOP__94\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Body
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2651] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2652])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2651] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2653];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2654] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2655])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2654] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2656];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2657] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2658])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2657] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2659];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2660] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2661])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2660] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2662];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2885] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2886])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2885] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2887];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2888] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2889])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2888] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2890];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3041] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3042])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3041] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3043];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3044] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3045])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3044] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3046];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3191] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3192])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3191] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3193];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3194] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3195])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3194] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3196];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3341] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3342])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3341] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3343];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3344] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3345])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3344] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3346];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3491] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3492])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3491] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3493];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3494] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3495])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3494] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3496];
        }
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2653] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2651])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2656] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2654])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2659] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2657])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2662] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2660])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2887] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2885])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2890] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2888])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3043] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3041])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3046] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3044])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3193] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3191])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3196] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3194])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3343] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3341])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3346] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3344])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3493] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3491])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3496] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3494])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1858] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2651]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2654]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1859] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2657]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2660]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1897] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2885]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2888]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1923] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3041]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3044]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1948] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3191]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3194]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1973] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3341]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3344]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1998] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3491]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3494]));
    }

    __device__
    void Top::_combo__TOP__95(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_combo__TOP__95\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Body
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2652] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x16U] 
                         >> 2U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2651])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2655] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x16U] 
                      >> 2U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2654])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2658] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x16U] 
                         >> 1U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2657])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2661] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x16U] 
                      >> 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2660])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2886] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[9U] 
                         >> 0x10U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2885])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2889] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[9U] 
                      >> 0x10U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2888])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3042] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[8U] 
                         >> 0xcU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3041])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3045] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[8U] 
                      >> 0xcU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3044])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3192] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[7U] 
                         >> 0x13U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3191])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3195] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[7U] 
                      >> 0x13U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3194])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3342] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[6U] 
                         >> 0x1aU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3341])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3345] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[6U] 
                      >> 0x1aU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3344])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3492] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[1U] 
                         >> 0xeU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3491])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3495] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[1U] 
                      >> 0xeU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3494])));
    }

    __device__
    void Top::_sequent__TOP__96(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_sequent__TOP__96\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Body
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2663] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2664])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2663] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2665];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2666] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2667])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2666] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2668];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2669] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2670])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2669] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2671];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2672] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2673])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2672] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2674];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2891] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2892])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2891] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2893];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2894] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2895])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2894] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2896];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3047] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3048])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3047] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3049];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3050] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3051])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3050] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3052];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3197] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3198])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3197] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3199];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3200] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3201])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3200] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3202];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3347] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3348])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3347] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3349];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3350] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3351])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3350] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3352];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3497] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3498])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3497] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3499];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3500] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3501])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3500] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3502];
        }
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2665] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2663])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2668] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2666])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2671] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2669])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2674] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2672])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2893] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2891])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2896] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2894])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3049] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3047])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3052] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3050])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3199] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3197])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3202] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3200])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3349] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3347])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3352] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3350])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3499] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3497])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3502] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3500])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1860] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2663]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2666]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1861] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2669]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2672]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1898] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2891]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2894]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1924] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3047]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3050]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1949] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3197]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3200]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1974] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3347]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3350]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1999] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3497]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3500]));
    }

    __device__
    void Top::_combo__TOP__97(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_combo__TOP__97\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Body
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2664] 
            = (1U & ((~ (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x16U]) 
                     | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2663])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2667] 
            = (1U & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x16U] 
                     | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2666])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2670] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x15U] 
                         >> 0x1fU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2669])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2673] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x15U] 
                      >> 0x1fU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2672])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2892] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[9U] 
                         >> 0xfU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2891])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2895] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[9U] 
                      >> 0xfU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2894])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3048] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[8U] 
                         >> 0xbU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3047])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3051] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[8U] 
                      >> 0xbU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3050])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3198] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[7U] 
                         >> 0x12U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3197])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3201] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[7U] 
                      >> 0x12U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3200])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3348] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[6U] 
                         >> 0x19U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3347])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3351] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[6U] 
                      >> 0x19U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3350])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3498] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[1U] 
                         >> 0xdU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3497])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3501] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[1U] 
                      >> 0xdU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3500])));
    }

    __device__
    void Top::_sequent__TOP__98(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_sequent__TOP__98\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Body
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2675] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2676])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2675] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2677];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2678] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2679])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2678] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2680];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2681] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2682])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2681] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2683];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2684] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2685])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2684] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2686];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2897] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2898])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2897] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2899];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2900] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2901])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2900] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2902];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3053] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3054])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3053] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3055];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3056] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3057])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3056] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3058];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3203] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3204])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3203] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3205];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3206] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3207])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3206] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3208];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3353] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3354])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3353] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3355];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3356] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3357])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3356] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3358];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3503] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3504])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3503] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3505];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3506] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3507])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3506] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3508];
        }
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2677] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2675])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2680] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2678])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2683] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2681])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2686] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2684])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2899] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2897])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2902] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2900])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3055] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3053])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3058] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3056])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3205] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3203])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3208] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3206])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3355] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3353])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3358] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3356])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3505] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3503])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3508] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3506])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1862] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2675]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2678]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1863] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2681]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2684]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1899] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2897]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2900]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1925] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3053]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3056]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1950] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3203]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3206]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1975] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3353]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3356]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2000] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3503]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3506]));
    }

    __device__
    void Top::_combo__TOP__99(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_combo__TOP__99\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Body
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2676] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x15U] 
                         >> 0x1eU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2675])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2679] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x15U] 
                      >> 0x1eU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2678])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2682] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x15U] 
                         >> 0x1dU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2681])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2685] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x15U] 
                      >> 0x1dU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2684])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2898] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[9U] 
                         >> 0xeU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2897])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2901] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[9U] 
                      >> 0xeU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2900])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3054] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[8U] 
                         >> 0xaU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3053])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3057] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[8U] 
                      >> 0xaU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3056])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3204] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[7U] 
                         >> 0x11U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3203])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3207] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[7U] 
                      >> 0x11U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3206])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3354] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[6U] 
                         >> 0x18U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3353])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3357] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[6U] 
                      >> 0x18U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3356])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3504] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[1U] 
                         >> 0xcU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3503])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3507] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[1U] 
                      >> 0xcU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3506])));
    }

    __device__
    void Top::_sequent__TOP__100(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_sequent__TOP__100\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Body
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2687] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2688])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2687] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2689];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2690] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2691])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2690] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2692];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2693] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2694])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2693] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2695];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2696] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2697])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2696] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2698];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2903] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2904])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2903] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2905];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2906] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2907])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2906] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2908];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3059] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3060])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3059] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3061];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3062] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3063])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3062] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3064];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3209] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3210])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3209] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3211];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3212] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3213])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3212] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3214];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3359] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3360])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3359] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3361];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3362] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3363])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3362] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3364];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3509] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3510])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3509] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3511];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3512] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3513])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3512] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3514];
        }
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2689] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2687])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2692] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2690])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2695] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2693])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2698] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2696])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2905] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2903])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2908] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2906])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3061] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3059])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3064] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3062])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3211] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3209])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3214] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3212])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3361] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3359])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3364] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3362])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3511] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3509])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3514] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3512])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1864] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2687]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2690]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1865] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2693]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2696]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1900] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2903]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2906]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1926] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3059]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3062]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1951] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3209]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3212]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1976] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3359]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3362]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2001] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3509]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3512]));
    }

    __device__
    void Top::_combo__TOP__101(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_combo__TOP__101\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Body
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2688] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x15U] 
                         >> 0x1cU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2687])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2691] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x15U] 
                      >> 0x1cU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2690])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2694] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x15U] 
                         >> 0x1bU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2693])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2697] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x15U] 
                      >> 0x1bU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2696])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2904] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[9U] 
                         >> 0xdU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2903])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2907] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[9U] 
                      >> 0xdU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2906])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3060] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[8U] 
                         >> 9U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3059])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3063] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[8U] 
                      >> 9U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3062])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3210] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[7U] 
                         >> 0x10U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3209])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3213] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[7U] 
                      >> 0x10U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3212])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3360] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[6U] 
                         >> 0x17U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3359])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3363] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[6U] 
                      >> 0x17U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3362])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3510] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[1U] 
                         >> 0xbU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3509])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3513] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[1U] 
                      >> 0xbU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3512])));
    }

    __device__
    void Top::_sequent__TOP__102(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_sequent__TOP__102\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Body
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2705] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2706])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2705] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2707];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2708] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2709])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2708] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2710];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2711] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2712])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2711] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2713];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2714] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2715])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2714] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2716];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2909] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2910])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2909] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2911];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2912] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2913])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2912] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2914];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3065] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3066])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3065] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3067];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3068] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3069])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3068] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3070];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3215] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3216])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3215] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3217];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3218] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3219])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3218] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3220];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3365] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3366])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3365] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3367];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3368] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3369])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3368] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3370];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3515] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3516])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3515] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3517];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3518] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3519])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3518] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3520];
        }
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2707] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2705])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2710] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2708])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2713] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2711])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2716] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2714])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2911] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2909])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2914] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2912])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3067] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3065])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3070] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3068])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3217] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3215])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3220] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3218])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3367] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3365])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3370] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3368])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3517] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3515])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3520] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3518])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1867] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2705]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2708]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1868] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2711]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2714]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1901] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2909]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2912]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1927] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3065]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3068]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1952] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3215]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3218]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1977] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3365]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3368]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2002] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3515]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3518]));
    }

    __device__
    void Top::_combo__TOP__103(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_combo__TOP__103\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Body
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2706] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x15U] 
                         >> 0x19U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2705])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2709] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x15U] 
                      >> 0x19U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2708])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2712] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x15U] 
                         >> 0x18U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2711])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2715] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x15U] 
                      >> 0x18U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2714])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2910] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[9U] 
                         >> 0xcU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2909])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2913] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[9U] 
                      >> 0xcU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2912])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3066] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[8U] 
                         >> 8U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3065])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3069] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[8U] 
                      >> 8U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3068])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3216] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[7U] 
                         >> 0xfU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3215])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3219] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[7U] 
                      >> 0xfU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3218])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3366] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[6U] 
                         >> 0x16U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3365])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3369] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[6U] 
                      >> 0x16U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3368])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3516] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[1U] 
                         >> 0xaU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3515])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3519] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[1U] 
                      >> 0xaU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3518])));
    }

    __device__
    void Top::_sequent__TOP__104(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_sequent__TOP__104\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Body
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2723] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2724])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2723] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2725];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2726] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2727])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2726] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2728];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2729] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2730])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2729] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2731];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2732] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2733])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2732] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2734];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2915] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2916])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2915] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2917];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2918] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2919])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2918] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2920];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3071] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3072])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3071] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3073];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3074] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3075])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3074] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3076];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3221] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3222])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3221] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3223];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3224] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3225])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3224] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3226];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3371] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3372])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3371] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3373];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3374] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3375])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3374] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3376];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3521] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3522])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3521] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3523];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3524] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3525])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3524] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3526];
        }
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2725] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2723])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2728] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2726])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2731] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2729])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2734] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2732])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2917] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2915])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2920] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2918])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3073] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3071])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3076] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3074])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3223] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3221])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3226] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3224])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3373] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3371])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3376] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3374])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3523] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3521])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3526] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3524])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1870] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2723]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2726]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1871] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2729]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2732]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1902] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2915]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2918]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1928] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3071]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3074]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1953] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3221]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3224]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1978] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3371]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3374]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2003] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3521]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3524]));
    }

    __device__
    void Top::_combo__TOP__105(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_combo__TOP__105\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Body
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2724] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x15U] 
                         >> 0x16U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2723])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2727] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x15U] 
                      >> 0x16U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2726])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2730] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x15U] 
                         >> 0x15U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2729])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2733] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[0x15U] 
                      >> 0x15U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2732])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2916] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[9U] 
                         >> 0xbU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2915])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2919] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[9U] 
                      >> 0xbU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2918])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3072] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[8U] 
                         >> 7U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3071])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3075] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[8U] 
                      >> 7U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3074])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3222] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[7U] 
                         >> 0xeU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3221])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3225] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[7U] 
                      >> 0xeU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3224])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3372] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[6U] 
                         >> 0x15U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3371])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3375] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[6U] 
                      >> 0x15U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3374])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3522] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[1U] 
                         >> 9U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3521])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3525] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 24 + BATCH_SIZE * 588)[1U] 
                      >> 9U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3524])));
    }

    __global__
    void _last_assign(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_last_assign\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Body
        // Final
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3633] 
            = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1817];
    }

    __global__
    void _change_request(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_change_request\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Body
        IData __req = false;
        __req |= vlTOPp->_change_request_1(_csignals, _ssignals, _isignals, _qsignals);
        change[blockDim.x * blockIdx.x + threadIdx.x] = __req;
    }

    __device__
    IData Top::_change_request_1(CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_change_request_1\n"); );
        // Body
        // Change detection
        IData __req = false;  // Logically a bool
        return __req;
    }

#ifdef VL_DEBUG
    void Top::_eval_debug_assertions() {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_eval_debug_assertions\n"); );
#pragma omp parallel for
        for(size_t i = 0; i < BATCH_SIZE; ++i) {
            // Body
            if (VL_UNLIKELY((_csignals[i + BATCH_SIZE * 1817] 
                             & 0xfeU))) {
                Verilated::overWidthError("clock");}
            if (VL_UNLIKELY((_csignals[i + BATCH_SIZE * 1818] 
                             & 0xfeU))) {
                Verilated::overWidthError("reset");}
            if (VL_UNLIKELY((_csignals[i + BATCH_SIZE * 2025] 
                             & 0xfeU))) {
                Verilated::overWidthError("io_meta_reset");}
        }
    }
#endif  // VL_DEBUG

    __global__
    void __Vmtask__4(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1817]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3633])))) {
            vlTOPp->_sequent__TOP__4(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__6(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1817]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3633])))) {
            vlTOPp->_sequent__TOP__5(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__7(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1817]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3633])))) {
            vlTOPp->_sequent__TOP__6(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__8(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1817]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3633])))) {
            vlTOPp->_sequent__TOP__7(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__9(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1817]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3633])))) {
            vlTOPp->_sequent__TOP__8(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__10(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1817]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3633])))) {
            vlTOPp->_sequent__TOP__9(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__16(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1817]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3633])))) {
            vlTOPp->_sequent__TOP__10(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__17(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1817]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3633])))) {
            vlTOPp->_sequent__TOP__11(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__18(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1817]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3633])))) {
            vlTOPp->_sequent__TOP__12(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__19(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1817]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3633])))) {
            vlTOPp->_sequent__TOP__13(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__20(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1817]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3633])))) {
            vlTOPp->_sequent__TOP__14(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__21(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1817]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3633])))) {
            vlTOPp->_sequent__TOP__15(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__22(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1817]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3633])))) {
            vlTOPp->_sequent__TOP__16(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__23(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1817]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3633])))) {
            vlTOPp->_sequent__TOP__17(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__25(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1817]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3633])))) {
            vlTOPp->_sequent__TOP__18(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__26(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1817]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3633])))) {
            vlTOPp->_sequent__TOP__19(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__27(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1817]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3633])))) {
            vlTOPp->_sequent__TOP__20(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__29(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1817]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3633])))) {
            vlTOPp->_sequent__TOP__21(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__30(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1817]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3633])))) {
            vlTOPp->_sequent__TOP__22(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__31(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1817]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3633])))) {
            vlTOPp->_sequent__TOP__23(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__32(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1817]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3633])))) {
            vlTOPp->_sequent__TOP__24(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__1(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1817]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3633])))) {
            vlTOPp->_sequent__TOP__25(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__2(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1817]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3633])))) {
            vlTOPp->_sequent__TOP__26(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__24(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1817]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3633])))) {
            vlTOPp->_sequent__TOP__27(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__28(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1817]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3633])))) {
            vlTOPp->_sequent__TOP__28(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__36(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1817]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3633])))) {
            vlTOPp->_sequent__TOP__30(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__37(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1817]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3633])))) {
            vlTOPp->_sequent__TOP__31(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__32(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1817]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3633])))) {
            vlTOPp->_sequent__TOP__33(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__38(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1817]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3633])))) {
            vlTOPp->_sequent__TOP__34(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__40(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1817]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3633])))) {
            vlTOPp->_sequent__TOP__35(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__41(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1817]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3633])))) {
            vlTOPp->_sequent__TOP__36(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__42(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1817]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3633])))) {
            vlTOPp->_sequent__TOP__37(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__43(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1817]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3633])))) {
            vlTOPp->_sequent__TOP__38(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__44(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1817]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3633])))) {
            vlTOPp->_sequent__TOP__39(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__45(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1817]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3633])))) {
            vlTOPp->_sequent__TOP__40(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__46(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1817]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3633])))) {
            vlTOPp->_sequent__TOP__41(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__47(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1817]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3633])))) {
            vlTOPp->_sequent__TOP__42(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__48(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1817]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3633])))) {
            vlTOPp->_sequent__TOP__43(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__49(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1817]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3633])))) {
            vlTOPp->_sequent__TOP__44(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__50(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1817]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3633])))) {
            vlTOPp->_sequent__TOP__45(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__34(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1817]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3633])))) {
            vlTOPp->_sequent__TOP__46(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__35(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1817]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3633])))) {
            vlTOPp->_sequent__TOP__47(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__39(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1817]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3633])))) {
            vlTOPp->_sequent__TOP__48(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__52(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1817]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3633])))) {
            vlTOPp->_sequent__TOP__49(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__50(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1817]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3633])))) {
            vlTOPp->_sequent__TOP__51(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__52(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__5(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1817]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3633])))) {
            vlTOPp->_sequent__TOP__53(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__54(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__11(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1817]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3633])))) {
            vlTOPp->_sequent__TOP__55(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__56(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__12(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1817]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3633])))) {
            vlTOPp->_sequent__TOP__57(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__58(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__13(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1817]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3633])))) {
            vlTOPp->_sequent__TOP__59(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__60(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__14(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1817]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3633])))) {
            vlTOPp->_sequent__TOP__61(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__62(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__15(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1817]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3633])))) {
            vlTOPp->_sequent__TOP__63(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__64(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__51(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1817]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3633])))) {
            vlTOPp->_sequent__TOP__65(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__53(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1817]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3633])))) {
            vlTOPp->_sequent__TOP__66(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__67(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__54(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1817]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3633])))) {
            vlTOPp->_sequent__TOP__68(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__69(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__55(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1817]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3633])))) {
            vlTOPp->_sequent__TOP__70(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__71(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__56(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1817]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3633])))) {
            vlTOPp->_sequent__TOP__72(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__73(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__57(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1817]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3633])))) {
            vlTOPp->_sequent__TOP__74(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__75(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__58(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1817]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3633])))) {
            vlTOPp->_sequent__TOP__76(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__77(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__59(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1817]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3633])))) {
            vlTOPp->_sequent__TOP__78(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__79(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__60(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1817]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3633])))) {
            vlTOPp->_sequent__TOP__80(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__81(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__61(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1817]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3633])))) {
            vlTOPp->_sequent__TOP__82(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__83(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__62(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1817]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3633])))) {
            vlTOPp->_sequent__TOP__84(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__85(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__63(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1817]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3633])))) {
            vlTOPp->_sequent__TOP__86(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__87(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__64(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1817]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3633])))) {
            vlTOPp->_sequent__TOP__88(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__89(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__65(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1817]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3633])))) {
            vlTOPp->_sequent__TOP__90(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__91(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__66(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1817]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3633])))) {
            vlTOPp->_sequent__TOP__92(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__93(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__67(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1817]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3633])))) {
            vlTOPp->_sequent__TOP__94(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__95(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__68(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1817]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3633])))) {
            vlTOPp->_sequent__TOP__96(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__97(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__69(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1817]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3633])))) {
            vlTOPp->_sequent__TOP__98(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__99(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__70(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1817]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3633])))) {
            vlTOPp->_sequent__TOP__100(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__101(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__71(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1817]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3633])))) {
            vlTOPp->_sequent__TOP__102(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__103(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__72(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1817]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3633])))) {
            vlTOPp->_sequent__TOP__104(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__105(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }
} // end of namespace RF ========================================
