#include "hip/hip_runtime.h"
// Verilated -*- C++ -*-
// DESCRIPTION: Verilator output: Design implementation internals
// See Top.h for the primary calling header

#include "Top.h"
#include "Top__Syms.h"
// begin of namespace RF =====================================
namespace RF {

    //==========

    __device__
    void Top::_combo__TOP__256(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_combo__TOP__256\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Body
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 25091] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x86U] 
                         >> 0x1aU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 25090])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 25094] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x86U] 
                      >> 0x1aU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 25093])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 25097] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x86U] 
                         >> 0x18U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 25096])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 25100] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x86U] 
                      >> 0x18U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 25099])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 25247] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x85U] 
                         >> 4U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 25246])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 25250] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x85U] 
                      >> 4U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 25249])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 25397] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x83U] 
                         >> 9U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 25396])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 25400] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x83U] 
                      >> 9U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 25399])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 25547] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x82U] 
                         >> 0xfU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 25546])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 25550] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x82U] 
                      >> 0xfU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 25549])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 25697] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x81U] 
                         >> 0x13U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 25696])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 25700] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x81U] 
                      >> 0x13U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 25699])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 25847] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x80U] 
                         >> 1U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 25846])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 25850] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x80U] 
                      >> 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 25849])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 25997] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x7dU] 
                         >> 8U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 25996])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 26000] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x7dU] 
                      >> 8U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 25999])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 26147] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x7cU] 
                         >> 6U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 26146])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 26150] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x7cU] 
                      >> 6U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 26149])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 26297] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x78U] 
                         >> 0x1eU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 26296])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 26300] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x78U] 
                      >> 0x1eU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 26299])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 26381] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x76U] 
                         >> 3U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 26380])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 26384] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x76U] 
                      >> 3U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 26383])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 26537] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x75U] 
                         >> 5U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 26536])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 26540] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x75U] 
                      >> 5U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 26539])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 26747] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x73U] 
                         >> 0x13U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 26746])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 26750] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x73U] 
                      >> 0x13U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 26749])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 26897] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x71U] 
                         >> 0x17U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 26896])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 26900] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x71U] 
                      >> 0x17U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 26899])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 26987] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x6eU] 
                         >> 0x1aU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 26986])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 26990] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x6eU] 
                      >> 0x1aU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 26989])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 27173] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x6bU] 
                         >> 0x1cU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 27172])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 27176] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x6bU] 
                      >> 0x1cU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 27175])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 27407] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x69U] 
                         >> 0x10U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 27406])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 27410] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x69U] 
                      >> 0x10U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 27409])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 27497] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x68U] 
                         >> 0x19U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 27496])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 27500] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x68U] 
                      >> 0x19U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 27499])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 27623] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x67U] 
                         >> 6U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 27622])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 27626] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x67U] 
                      >> 6U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 27625])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 27773] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x65U] 
                         >> 0xbU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 27772])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 27776] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x65U] 
                      >> 0xbU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 27775])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 27923] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x64U] 
                         >> 0xeU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 27922])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 27926] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x64U] 
                      >> 0xeU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 27925])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 28073] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x63U] 
                         >> 0x14U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 28072])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 28076] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x63U] 
                      >> 0x14U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 28075])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 28223] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x62U] 
                         >> 0x1aU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 28222])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 28226] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x62U] 
                      >> 0x1aU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 28225])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 28373] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x60U] 
                         >> 7U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 28372])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 28376] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x60U] 
                      >> 7U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 28375])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 28541] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x5eU] 
                         >> 0xdU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 28540])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 28544] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x5eU] 
                      >> 0xdU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 28543])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 28817] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x52U] 
                         >> 5U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 28816])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 28820] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x52U] 
                      >> 5U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 28819])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 28823] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x52U] 
                         >> 3U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 28822])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 28826] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x52U] 
                      >> 3U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 28825])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 28973] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x51U] 
                         >> 9U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 28972])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 28976] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x51U] 
                      >> 9U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 28975])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 29123] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x50U] 
                         >> 0x10U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 29122])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 29126] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x50U] 
                      >> 0x10U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 29125])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 29417] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x4eU] 
                         >> 0x1fU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 29416])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 29420] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x4eU] 
                      >> 0x1fU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 29419])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 29423] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x4eU] 
                         >> 0x1eU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 29422])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 29426] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x4eU] 
                      >> 0x1eU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 29425])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 29669] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x4dU] 
                         >> 0x15U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 29668])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 29672] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x4dU] 
                      >> 0x15U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 29671])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 29975] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x48U] 
                         >> 0x19U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 29974])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 29978] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x48U] 
                      >> 0x19U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 29977])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 29981] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x48U] 
                         >> 0x17U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 29980])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 29984] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x48U] 
                      >> 0x17U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 29983])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 30203] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x3fU] 
                         >> 0xdU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 30202])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 30206] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x3fU] 
                      >> 0xdU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 30205])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 30209] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x3fU] 
                         >> 7U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 30208])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 30212] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x3fU] 
                      >> 7U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 30211])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 30455] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x3cU] 
                         >> 0x15U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 30454])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 30458] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x3cU] 
                      >> 0x15U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 30457])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 30755] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x39U] 
                         >> 0xaU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 30754])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 30758] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x39U] 
                      >> 0xaU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 30757])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 30761] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x38U] 
                         >> 0x19U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 30760])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 30764] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x38U] 
                      >> 0x19U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 30763])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 31055] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x36U] 
                         >> 0x1bU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 31054])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 31058] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x36U] 
                      >> 0x1bU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 31057])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 31061] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x36U] 
                         >> 0x19U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 31060])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 31064] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x36U] 
                      >> 0x19U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 31063])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 31367] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x32U] 
                         >> 0x19U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 31366])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 31370] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x32U] 
                      >> 0x19U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 31369])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 31373] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x32U] 
                         >> 0x18U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 31372])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 31376] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x32U] 
                      >> 0x18U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 31375])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 31739] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x26U] 
                         >> 0xcU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 31738])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 31742] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x26U] 
                      >> 0xcU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 31741])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 31745] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x26U] 
                         >> 0xbU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 31744])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 31748] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x26U] 
                      >> 0xbU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 31747])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 31751] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x26U] 
                         >> 0xaU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 31750])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 31754] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x26U] 
                      >> 0xaU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 31753])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 32057] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x21U] 
                         >> 0xfU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 32056])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 32060] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x21U] 
                      >> 0xfU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 32059])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 32063] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x21U] 
                         >> 0xeU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 32062])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 32066] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x21U] 
                      >> 0xeU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 32065])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 32375] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x16U] 
                         >> 0x1bU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 32374])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 32378] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x16U] 
                      >> 0x1bU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 32377])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 32381] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x16U] 
                         >> 0x1aU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 32380])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 32384] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[0x16U] 
                      >> 0x1aU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 32383])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 32831] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[6U] 
                         >> 0x12U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 32830])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 32834] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[6U] 
                      >> 0x12U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 32833])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 32837] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[6U] 
                         >> 0x11U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 32836])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 32840] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[6U] 
                      >> 0x11U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 32839])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 32843] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[6U] 
                         >> 0x10U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 32842])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 32846] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 142 + BATCH_SIZE * 11037)[6U] 
                      >> 0x10U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 32845])));
    }

    __global__
    void _last_assign(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_last_assign\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Body
        // Final
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113] 
            = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557];
    }

    __global__
    void _change_request(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_change_request\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Body
        IData __req = false;
        __req |= vlTOPp->_change_request_1(_csignals, _ssignals, _isignals, _qsignals);
        change[blockDim.x * blockIdx.x + threadIdx.x] = __req;
    }

    __device__
    IData Top::_change_request_1(CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_change_request_1\n"); );
        // Body
        // Change detection
        IData __req = false;  // Logically a bool
        return __req;
    }

#ifdef VL_DEBUG
    void Top::_eval_debug_assertions() {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_eval_debug_assertions\n"); );
#pragma omp parallel for
        for(size_t i = 0; i < BATCH_SIZE; ++i) {
            // Body
            if (VL_UNLIKELY((_csignals[i + BATCH_SIZE * 16557] 
                             & 0xfeU))) {
                Verilated::overWidthError("clock");}
            if (VL_UNLIKELY((_csignals[i + BATCH_SIZE * 16558] 
                             & 0xfeU))) {
                Verilated::overWidthError("reset");}
            if (VL_UNLIKELY((_csignals[i + BATCH_SIZE * 18429] 
                             & 0xfeU))) {
                Verilated::overWidthError("io_meta_reset");}
        }
    }
#endif  // VL_DEBUG

    __global__
    void __Vmtask__1(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__10(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__5(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__11(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__6(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__12(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__8(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__13(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__20(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__14(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__22(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__15(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__23(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__16(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__49(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__17(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__54(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__18(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__55(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__19(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__56(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__20(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__61(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__21(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__7(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__22(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__24(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__23(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__26(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__24(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__29(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__25(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__31(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__26(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__39(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__27(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__40(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__28(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__41(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__29(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__42(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__30(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__43(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__31(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__44(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__32(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__45(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__33(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__46(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__34(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__47(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__35(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__48(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__36(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__50(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__37(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__66(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__38(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__25(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__39(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__27(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__40(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__30(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__41(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__34(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__42(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__35(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__43(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__36(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__44(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__37(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__45(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__38(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__46(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__53(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__47(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__62(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__48(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__64(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__49(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__65(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__50(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__67(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__51(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__73(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__52(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__74(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__53(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__3(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__54(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__4(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__55(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__21(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__56(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__28(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__57(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__33(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__58(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__60(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__59(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__82(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__60(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__98(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__61(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__2(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__62(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__32(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__63(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__64(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__65(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__58(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__66(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__68(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__67(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__68(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__69(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__70(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__71(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__71(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__72(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__81(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__73(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__84(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__74(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__95(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__75(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__76(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__77(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__78(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__79(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__51(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__80(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__59(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__81(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__82(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__83(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__84(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__85(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__72(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__86(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__75(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__87(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__88(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__89(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__90(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__91(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__92(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__93(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__94(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__95(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__76(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__96(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__78(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__97(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__79(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__98(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__85(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__99(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__86(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__100(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__88(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__101(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__91(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__102(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__94(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__103(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__70(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__104(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__77(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__105(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__106(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__107(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__108(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__109(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__80(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__110(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__87(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__111(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__89(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__112(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__113(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__114(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__115(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__116(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__90(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        vlTOPp->_combo__TOP__117(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__118(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__119(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__120(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__121(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__122(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__123(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__124(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__125(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__126(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__127(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__128(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__92(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__129(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__130(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__131(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__93(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__132(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__133(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__134(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__135(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__136(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__137(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__138(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__97(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__139(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__57(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__140(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__141(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__142(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__143(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__144(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__145(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__63(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__146(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__69(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__147(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__148(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__149(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__150(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__151(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__83(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__152(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__153(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__154(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__155(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__156(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__157(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__158(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__159(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__160(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__161(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__162(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__163(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__99(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__164(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__165(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__166(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__167(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__168(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__169(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__170(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__171(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__172(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__173(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__174(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__175(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__176(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__177(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__100(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__178(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__179(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__101(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__180(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__181(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__102(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__182(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__183(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__52(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__184(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__185(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__186(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__187(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__188(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__189(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__190(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__96(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__191(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__192(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__193(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__194(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__195(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__196(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__197(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__198(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__199(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__200(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__201(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__202(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__203(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__204(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__103(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__205(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__206(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__104(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__207(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__208(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__105(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__209(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__210(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__106(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__211(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__212(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__107(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__213(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__214(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__108(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__215(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__216(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__109(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__217(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__218(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__110(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__219(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__220(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__111(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__221(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__222(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__112(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__223(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__224(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__113(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__225(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__226(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__114(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__227(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__228(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__115(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__229(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__230(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__116(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__231(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__232(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__117(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__233(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__234(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__9(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__235(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__236(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__10(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__237(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__238(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__11(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__239(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__240(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__12(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__241(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__242(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__13(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__243(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__244(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__14(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__245(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__246(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__15(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__247(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__248(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__16(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__249(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__250(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__17(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__251(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__252(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__18(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__253(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__254(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__19(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 16557]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 33113])))) {
            vlTOPp->_sequent__TOP__255(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__256(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }
} // end of namespace RF ========================================
