#include "hip/hip_runtime.h"
// Verilated -*- C++ -*-
// DESCRIPTION: Verilator output: Design implementation internals
// See Top.h for the primary calling header

#include "Top.h"
#include "Top__Syms.h"
// begin of namespace RF =====================================
namespace RF {

    //==========

    __device__
    void Top::_combo__TOP__80(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_combo__TOP__80\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Body
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2155] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[9U] 
                         >> 0xaU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2154])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2158] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[9U] 
                      >> 0xaU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2157])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2161] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[9U] 
                         >> 9U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2160])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2164] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[9U] 
                      >> 9U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2163])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2407] 
            = (1U & ((~ (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[8U]) 
                     | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2406])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2410] 
            = (1U & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[8U] 
                     | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2409])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2563] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[6U] 
                         >> 0x11U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2562])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2566] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[6U] 
                      >> 0x11U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2565])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2713] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[2U] 
                         >> 0x1aU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2712])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2716] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[2U] 
                      >> 0x1aU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2715])));
    }

    __device__
    void Top::_sequent__TOP__81(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_sequent__TOP__81\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Body
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2166] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2167])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2166] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2168];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2169] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2170])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2169] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2171];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2172] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2173])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2172] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2174];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2175] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2176])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2175] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2177];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2412] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2413])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2412] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2414];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2415] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2416])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2415] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2417];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2568] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2569])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2568] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2570];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2571] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2572])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2571] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2573];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2718] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2719])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2718] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2720];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2721] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2722])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2721] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2723];
        }
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2168] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2166])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2171] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2169])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2174] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2172])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2177] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2175])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2414] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2412])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2417] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2415])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2570] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2568])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2573] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2571])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2720] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2718])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2723] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2721])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1523] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2166]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2169]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1524] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2172]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2175]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1564] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2412]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2415]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1590] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2568]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2571]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1615] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2718]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2721]));
    }

    __device__
    void Top::_combo__TOP__82(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_combo__TOP__82\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Body
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2167] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[9U] 
                         >> 8U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2166])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2170] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[9U] 
                      >> 8U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2169])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2173] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[9U] 
                         >> 7U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2172])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2176] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[9U] 
                      >> 7U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2175])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2413] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[7U] 
                         >> 0x1fU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2412])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2416] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[7U] 
                      >> 0x1fU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2415])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2569] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[6U] 
                         >> 0xfU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2568])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2572] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[6U] 
                      >> 0xfU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2571])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2719] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[2U] 
                         >> 0x19U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2718])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2722] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[2U] 
                      >> 0x19U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2721])));
    }

    __device__
    void Top::_sequent__TOP__83(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_sequent__TOP__83\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Body
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2178] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2179])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2178] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2180];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2181] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2182])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2181] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2183];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2184] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2185])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2184] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2186];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2187] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2188])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2187] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2189];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2418] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2419])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2418] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2420];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2421] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2422])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2421] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2423];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2574] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2575])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2574] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2576];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2577] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2578])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2577] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2579];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2724] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2725])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2724] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2726];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2727] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2728])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2727] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2729];
        }
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2180] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2178])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2183] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2181])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2186] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2184])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2189] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2187])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2420] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2418])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2423] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2421])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2576] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2574])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2579] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2577])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2726] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2724])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2729] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2727])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1525] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2178]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2181]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1526] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2184]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2187]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1565] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2418]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2421]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1591] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2574]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2577]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1616] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2724]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2727]));
    }

    __device__
    void Top::_combo__TOP__84(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_combo__TOP__84\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Body
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2179] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[9U] 
                         >> 6U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2178])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2182] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[9U] 
                      >> 6U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2181])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2185] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[9U] 
                         >> 5U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2184])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2188] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[9U] 
                      >> 5U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2187])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2419] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[7U] 
                         >> 0x1eU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2418])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2422] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[7U] 
                      >> 0x1eU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2421])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2575] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[6U] 
                         >> 0xcU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2574])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2578] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[6U] 
                      >> 0xcU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2577])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2725] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[2U] 
                         >> 0x14U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2724])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2728] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[2U] 
                      >> 0x14U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2727])));
    }

    __device__
    void Top::_sequent__TOP__85(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_sequent__TOP__85\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Body
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2190] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2191])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2190] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2192];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2193] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2194])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2193] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2195];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2196] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2197])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2196] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2198];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2199] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2200])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2199] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2201];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2424] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2425])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2424] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2426];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2427] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2428])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2427] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2429];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2580] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2581])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2580] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2582];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2583] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2584])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2583] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2585];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2730] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2731])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2730] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2732];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2733] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2734])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2733] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2735];
        }
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2192] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2190])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2195] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2193])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2198] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2196])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2201] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2199])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2426] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2424])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2429] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2427])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2582] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2580])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2585] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2583])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2732] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2730])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2735] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2733])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1527] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2190]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2193]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1528] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2196]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2199]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1566] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2424]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2427]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1592] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2580]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2583]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1617] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2730]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2733]));
    }

    __device__
    void Top::_combo__TOP__86(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_combo__TOP__86\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Body
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2191] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[9U] 
                         >> 4U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2190])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2194] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[9U] 
                      >> 4U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2193])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2197] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[9U] 
                         >> 3U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2196])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2200] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[9U] 
                      >> 3U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2199])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2425] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[7U] 
                         >> 0x1dU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2424])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2428] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[7U] 
                      >> 0x1dU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2427])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2581] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[6U] 
                         >> 0xaU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2580])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2584] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[6U] 
                      >> 0xaU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2583])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2731] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[2U] 
                         >> 0x13U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2730])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2734] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[2U] 
                      >> 0x13U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2733])));
    }

    __device__
    void Top::_sequent__TOP__87(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_sequent__TOP__87\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Body
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2202] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2203])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2202] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2204];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2205] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2206])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2205] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2207];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2208] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2209])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2208] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2210];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2211] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2212])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2211] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2213];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2430] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2431])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2430] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2432];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2433] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2434])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2433] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2435];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2586] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2587])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2586] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2588];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2589] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2590])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2589] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2591];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2736] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2737])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2736] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2738];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2739] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2740])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2739] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2741];
        }
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2204] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2202])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2207] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2205])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2210] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2208])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2213] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2211])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2432] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2430])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2435] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2433])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2588] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2586])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2591] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2589])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2738] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2736])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2741] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2739])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1529] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2202]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2205]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1530] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2208]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2211]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1567] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2430]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2433]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1593] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2586]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2589]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1618] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2736]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2739]));
    }

    __device__
    void Top::_combo__TOP__88(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_combo__TOP__88\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Body
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2203] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[9U] 
                         >> 2U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2202])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2206] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[9U] 
                      >> 2U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2205])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2209] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[9U] 
                         >> 1U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2208])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2212] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[9U] 
                      >> 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2211])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2431] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[7U] 
                         >> 0x1cU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2430])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2434] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[7U] 
                      >> 0x1cU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2433])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2587] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[6U] 
                         >> 4U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2586])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2590] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[6U] 
                      >> 4U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2589])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2737] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[2U] 
                         >> 0x12U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2736])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2740] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[2U] 
                      >> 0x12U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2739])));
    }

    __device__
    void Top::_sequent__TOP__89(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_sequent__TOP__89\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Body
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2214] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2215])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2214] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2216];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2217] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2218])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2217] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2219];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2220] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2221])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2220] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2222];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2223] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2224])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2223] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2225];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2436] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2437])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2436] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2438];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2439] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2440])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2439] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2441];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2592] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2593])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2592] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2594];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2595] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2596])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2595] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2597];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2742] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2743])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2742] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2744];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2745] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2746])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2745] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2747];
        }
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2216] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2214])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2219] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2217])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2222] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2220])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2225] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2223])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2438] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2436])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2441] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2439])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2594] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2592])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2597] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2595])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2744] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2742])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2747] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2745])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1531] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2214]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2217]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1532] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2220]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2223]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1568] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2436]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2439]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1594] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2592]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2595]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1619] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2742]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2745]));
    }

    __device__
    void Top::_combo__TOP__90(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_combo__TOP__90\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Body
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2215] 
            = (1U & ((~ (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[9U]) 
                     | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2214])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2218] 
            = (1U & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[9U] 
                     | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2217])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2221] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[8U] 
                         >> 0x1fU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2220])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2224] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[8U] 
                      >> 0x1fU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2223])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2437] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[7U] 
                         >> 0x1bU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2436])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2440] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[7U] 
                      >> 0x1bU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2439])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2593] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[5U] 
                         >> 0x1eU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2592])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2596] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[5U] 
                      >> 0x1eU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2595])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2743] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[2U] 
                         >> 0x11U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2742])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2746] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[2U] 
                      >> 0x11U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2745])));
    }

    __device__
    void Top::_sequent__TOP__91(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_sequent__TOP__91\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Body
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2226] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2227])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2226] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2228];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2229] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2230])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2229] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2231];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2232] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2233])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2232] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2234];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2235] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2236])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2235] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2237];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2442] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2443])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2442] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2444];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2445] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2446])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2445] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2447];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2598] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2599])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2598] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2600];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2601] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2602])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2601] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2603];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2748] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2749])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2748] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2750];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2751] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2752])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2751] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2753];
        }
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2228] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2226])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2231] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2229])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2234] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2232])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2237] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2235])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2444] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2442])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2447] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2445])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2600] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2598])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2603] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2601])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2750] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2748])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2753] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2751])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1533] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2226]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2229]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1534] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2232]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2235]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1569] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2442]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2445]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1595] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2598]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2601]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1620] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2748]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2751]));
    }

    __device__
    void Top::_combo__TOP__92(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_combo__TOP__92\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Body
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2227] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[8U] 
                         >> 0x1eU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2226])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2230] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[8U] 
                      >> 0x1eU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2229])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2233] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[8U] 
                         >> 0x1dU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2232])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2236] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[8U] 
                      >> 0x1dU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2235])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2443] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[7U] 
                         >> 0x1aU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2442])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2446] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[7U] 
                      >> 0x1aU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2445])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2599] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[5U] 
                         >> 0x1dU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2598])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2602] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[5U] 
                      >> 0x1dU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2601])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2749] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[2U] 
                         >> 0x10U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2748])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2752] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[2U] 
                      >> 0x10U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2751])));
    }

    __device__
    void Top::_sequent__TOP__93(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_sequent__TOP__93\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Body
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2244] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2245])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2244] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2246];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2247] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2248])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2247] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2249];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2250] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2251])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2250] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2252];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2253] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2254])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2253] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2255];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2448] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2449])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2448] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2450];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2451] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2452])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2451] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2453];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2604] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2605])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2604] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2606];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2607] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2608])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2607] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2609];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2754] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2755])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2754] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2756];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2757] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2758])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2757] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2759];
        }
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2246] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2244])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2249] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2247])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2252] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2250])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2255] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2253])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2450] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2448])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2453] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2451])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2606] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2604])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2609] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2607])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2756] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2754])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2759] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2757])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1536] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2244]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2247]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1537] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2250]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2253]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1570] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2448]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2451]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1596] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2604]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2607]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1621] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2754]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2757]));
    }

    __device__
    void Top::_combo__TOP__94(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_combo__TOP__94\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Body
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2245] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[8U] 
                         >> 0x1bU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2244])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2248] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[8U] 
                      >> 0x1bU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2247])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2251] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[8U] 
                         >> 0x1aU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2250])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2254] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[8U] 
                      >> 0x1aU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2253])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2449] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[7U] 
                         >> 0x19U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2448])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2452] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[7U] 
                      >> 0x19U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2451])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2605] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[5U] 
                         >> 0x1cU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2604])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2608] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[5U] 
                      >> 0x1cU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2607])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2755] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[2U] 
                         >> 0xfU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2754])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2758] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[2U] 
                      >> 0xfU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2757])));
    }

    __device__
    void Top::_sequent__TOP__95(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_sequent__TOP__95\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Body
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2262] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2263])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2262] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2264];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2265] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2266])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2265] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2267];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2268] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2269])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2268] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2270];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2271] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2272])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2271] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2273];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2454] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2455])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2454] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2456];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2457] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2458])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2457] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2459];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2610] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2611])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2610] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2612];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2613] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2614])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2613] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2615];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2760] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2761])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2760] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2762];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2763] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2764])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2763] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2765];
        }
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2264] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2262])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2267] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2265])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2270] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2268])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2273] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2271])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2456] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2454])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2459] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2457])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2612] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2610])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2615] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2613])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2762] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2760])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2765] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2763])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1539] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2262]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2265]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1540] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2268]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2271]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1571] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2454]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2457]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1597] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2610]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2613]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1622] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2760]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2763]));
    }

    __device__
    void Top::_combo__TOP__96(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_combo__TOP__96\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Body
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2263] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[8U] 
                         >> 0x18U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2262])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2266] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[8U] 
                      >> 0x18U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2265])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2269] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[8U] 
                         >> 0x17U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2268])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2272] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[8U] 
                      >> 0x17U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2271])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2455] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[7U] 
                         >> 0x18U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2454])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2458] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[7U] 
                      >> 0x18U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2457])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2611] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[5U] 
                         >> 0x1bU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2610])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2614] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[5U] 
                      >> 0x1bU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2613])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2761] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[2U] 
                         >> 0xdU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2760])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2764] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[2U] 
                      >> 0xdU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2763])));
    }

    __device__
    void Top::_sequent__TOP__97(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_sequent__TOP__97\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Body
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2460] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2461])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2460] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2462];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2463] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2464])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2463] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2465];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2616] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2617])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2616] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2618];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2619] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2620])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2619] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2621];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2766] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2767])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2766] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2768];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2769] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2770])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2769] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2771];
        }
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2462] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2460])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2465] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2463])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2618] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2616])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2621] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2619])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2768] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2766])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2771] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2769])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1572] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2460]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2463]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1598] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2616]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2619]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1623] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2766]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2769]));
    }

    __device__
    void Top::_combo__TOP__98(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_combo__TOP__98\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Body
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2011] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[0xaU] 
                         >> 2U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2010])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2089] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[9U] 
                         >> 0x15U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2088])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2299] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[8U] 
                         >> 0x12U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2298])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2302] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[8U] 
                      >> 0x12U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2301])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2371] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[8U] 
                         >> 6U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2370])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2374] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[8U] 
                      >> 6U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2373])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2461] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[7U] 
                         >> 0x17U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2460])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2464] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[7U] 
                      >> 0x17U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2463])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2617] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[5U] 
                         >> 0x1aU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2616])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2620] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[5U] 
                      >> 0x1aU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2619])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2767] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[2U] 
                         >> 0xcU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2766])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2770] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[2U] 
                      >> 0xcU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2769])));
    }

    __device__
    void Top::_sequent__TOP__99(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_sequent__TOP__99\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Body
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2466] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2467])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2466] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2468];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2469] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2470])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2469] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2471];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2472] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2473])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2472] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2474];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2475] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2476])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2475] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2477];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2622] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2623])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2622] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2624];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2625] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2626])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2625] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2627];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2772] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2773])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2772] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2774];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2775] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2776])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2775] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2777];
        }
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2468] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2466])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2471] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2469])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2474] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2472])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2477] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2475])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2624] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2622])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2627] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2625])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2774] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2772])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2777] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2775])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1573] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2466]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2469]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1574] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2472]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2475]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1599] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2622]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2625]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1624] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2772]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2775]));
    }

    __device__
    void Top::_combo__TOP__100(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_combo__TOP__100\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Body
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2467] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[7U] 
                         >> 0x16U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2466])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2470] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[7U] 
                      >> 0x16U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2469])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2473] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[7U] 
                         >> 0x15U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2472])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2476] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[7U] 
                      >> 0x15U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2475])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2623] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[5U] 
                         >> 0x15U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2622])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2626] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[5U] 
                      >> 0x15U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2625])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2773] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[2U] 
                         >> 0xbU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2772])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2776] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[2U] 
                      >> 0xbU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2775])));
    }

    __device__
    void Top::_sequent__TOP__101(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_sequent__TOP__101\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Body
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2478] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2479])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2478] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2480];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2481] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2482])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2481] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2483];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2484] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2485])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2484] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2486];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2487] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2488])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2487] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2489];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2628] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2629])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2628] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2630];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2631] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2632])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2631] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2633];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2778] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2779])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2778] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2780];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2781] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2782])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2781] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2783];
        }
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2480] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2478])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2483] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2481])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2486] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2484])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2489] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2487])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2630] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2628])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2633] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2631])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2780] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2778])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2783] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2781])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1575] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2478]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2481]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1576] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2484]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2487]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1600] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2628]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2631]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1625] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2778]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2781]));
    }

    __device__
    void Top::_combo__TOP__102(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_combo__TOP__102\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Body
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2479] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[7U] 
                         >> 0x14U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2478])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2482] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[7U] 
                      >> 0x14U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2481])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2485] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[7U] 
                         >> 0x13U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2484])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2488] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[7U] 
                      >> 0x13U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2487])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2629] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[5U] 
                         >> 0x14U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2628])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2632] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[5U] 
                      >> 0x14U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2631])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2779] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[2U] 
                         >> 0xaU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2778])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2782] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[2U] 
                      >> 0xaU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2781])));
    }

    __device__
    void Top::_sequent__TOP__103(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_sequent__TOP__103\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Body
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2490] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2491])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2490] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2492];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2493] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2494])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2493] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2495];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2496] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2497])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2496] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2498];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2499] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2500])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2499] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2501];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2634] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2635])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2634] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2636];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2637] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2638])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2637] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2639];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2784] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2785])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2784] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2786];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2787] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2788])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2787] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2789];
        }
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2492] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2490])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2495] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2493])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2498] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2496])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2501] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2499])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2636] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2634])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2639] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2637])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2786] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2784])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2789] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2787])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1577] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2490]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2493]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1578] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2496]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2499]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1601] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2634]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2637]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1626] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2784]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2787]));
    }

    __device__
    void Top::_combo__TOP__104(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_combo__TOP__104\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Body
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2491] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[7U] 
                         >> 0x12U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2490])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2494] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[7U] 
                      >> 0x12U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2493])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2497] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[7U] 
                         >> 0x11U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2496])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2500] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[7U] 
                      >> 0x11U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2499])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2635] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[5U] 
                         >> 0x13U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2634])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2638] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[5U] 
                      >> 0x13U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2637])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2785] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[2U] 
                         >> 5U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2784])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2788] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[2U] 
                      >> 5U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2787])));
    }

    __device__
    void Top::_sequent__TOP__105(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_sequent__TOP__105\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Body
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2502] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2503])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2502] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2504];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2505] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2506])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2505] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2507];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2508] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2509])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2508] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2510];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2511] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2512])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2511] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2513];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2640] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2641])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2640] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2642];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2643] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2644])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2643] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2645];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2790] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2791])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2790] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2792];
        }
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421]) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2793] = 0U;
        } else if ((1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2794])))) {
            _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2793] 
                = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2795];
        }
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2504] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2502])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2507] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2505])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2510] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2508])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2513] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2511])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2642] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2640])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2645] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2643])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2792] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2790])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2795] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2793])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1579] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2502]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2505]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1580] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2508]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2511]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1602] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2640]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2643]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1627] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2790]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2793]));
    }

    __device__
    void Top::_combo__TOP__106(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_combo__TOP__106\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Body
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2503] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[7U] 
                         >> 0x10U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2502])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2506] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[7U] 
                      >> 0x10U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2505])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2509] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[7U] 
                         >> 0xfU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2508])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2512] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[7U] 
                      >> 0xfU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2511])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2641] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[5U] 
                         >> 8U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2640])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2644] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[5U] 
                      >> 8U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2643])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2791] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[2U] 
                         >> 4U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2790])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2794] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 11 + BATCH_SIZE * 88)[2U] 
                      >> 4U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2793])));
    }

    __global__
    void _last_assign(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_last_assign\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Body
        // Final
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2839] 
            = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1420];
    }

    __global__
    void _change_request(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_change_request\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Body
        IData __req = false;
        __req |= vlTOPp->_change_request_1(_csignals, _ssignals, _isignals, _qsignals);
        change[blockDim.x * blockIdx.x + threadIdx.x] = __req;
    }

    __device__
    IData Top::_change_request_1(CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_change_request_1\n"); );
        // Body
        // Change detection
        IData __req = false;  // Logically a bool
        return __req;
    }

#ifdef VL_DEBUG
    void Top::_eval_debug_assertions() {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_eval_debug_assertions\n"); );
#pragma omp parallel for
        for(size_t i = 0; i < BATCH_SIZE; ++i) {
            // Body
            if (VL_UNLIKELY((_csignals[i + BATCH_SIZE * 1420] 
                             & 0xfeU))) {
                Verilated::overWidthError("clock");}
            if (VL_UNLIKELY((_csignals[i + BATCH_SIZE * 1421] 
                             & 0xfeU))) {
                Verilated::overWidthError("reset");}
            if (VL_UNLIKELY((_csignals[i + BATCH_SIZE * 1636] 
                             & 0xfeU))) {
                Verilated::overWidthError("io_meta_reset");}
        }
    }
#endif  // VL_DEBUG

    __global__
    void __Vmtask__1(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1420]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2839])))) {
            vlTOPp->_sequent__TOP__3(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__3(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1420]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2839])))) {
            vlTOPp->_sequent__TOP__4(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__4(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1420]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2839])))) {
            vlTOPp->_sequent__TOP__5(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__5(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1420]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2839])))) {
            vlTOPp->_sequent__TOP__6(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__6(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1420]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2839])))) {
            vlTOPp->_sequent__TOP__7(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__7(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1420]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2839])))) {
            vlTOPp->_sequent__TOP__8(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__8(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1420]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2839])))) {
            vlTOPp->_sequent__TOP__9(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__9(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1420]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2839])))) {
            vlTOPp->_sequent__TOP__10(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__10(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1420]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2839])))) {
            vlTOPp->_sequent__TOP__11(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__16(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1420]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2839])))) {
            vlTOPp->_sequent__TOP__12(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__17(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1420]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2839])))) {
            vlTOPp->_sequent__TOP__13(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__18(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1420]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2839])))) {
            vlTOPp->_sequent__TOP__14(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__20(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1420]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2839])))) {
            vlTOPp->_sequent__TOP__15(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__22(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1420]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2839])))) {
            vlTOPp->_sequent__TOP__16(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__23(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1420]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2839])))) {
            vlTOPp->_sequent__TOP__17(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__24(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1420]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2839])))) {
            vlTOPp->_sequent__TOP__18(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__25(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1420]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2839])))) {
            vlTOPp->_sequent__TOP__19(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__26(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1420]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2839])))) {
            vlTOPp->_sequent__TOP__20(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__31(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1420]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2839])))) {
            vlTOPp->_sequent__TOP__21(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__19(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1420]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2839])))) {
            vlTOPp->_sequent__TOP__22(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__28(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1420]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2839])))) {
            vlTOPp->_sequent__TOP__23(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__24(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__30(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1420]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2839])))) {
            vlTOPp->_sequent__TOP__25(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__32(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1420]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2839])))) {
            vlTOPp->_sequent__TOP__26(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__33(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1420]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2839])))) {
            vlTOPp->_sequent__TOP__27(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__34(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1420]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2839])))) {
            vlTOPp->_sequent__TOP__28(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__29(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__35(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        vlTOPp->_combo__TOP__30(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__36(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        vlTOPp->_combo__TOP__31(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__37(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        vlTOPp->_combo__TOP__32(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__38(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        vlTOPp->_combo__TOP__34(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__2(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1420]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2839])))) {
            vlTOPp->_sequent__TOP__35(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__36(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__21(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1420]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2839])))) {
            vlTOPp->_sequent__TOP__37(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__38(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__27(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1420]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2839])))) {
            vlTOPp->_sequent__TOP__39(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__29(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1420]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2839])))) {
            vlTOPp->_sequent__TOP__40(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__41(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__39(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        vlTOPp->_combo__TOP__42(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__40(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1420]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2839])))) {
            vlTOPp->_sequent__TOP__43(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__44(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__41(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1420]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2839])))) {
            vlTOPp->_sequent__TOP__45(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__46(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__42(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1420]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2839])))) {
            vlTOPp->_sequent__TOP__47(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__48(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__43(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1420]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2839])))) {
            vlTOPp->_sequent__TOP__49(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__50(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__44(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1420]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2839])))) {
            vlTOPp->_sequent__TOP__51(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__52(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__45(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1420]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2839])))) {
            vlTOPp->_sequent__TOP__53(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__54(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__46(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1420]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2839])))) {
            vlTOPp->_sequent__TOP__55(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__56(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__47(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1420]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2839])))) {
            vlTOPp->_sequent__TOP__57(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__58(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__48(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1420]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2839])))) {
            vlTOPp->_sequent__TOP__59(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__60(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__49(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1420]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2839])))) {
            vlTOPp->_sequent__TOP__61(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__62(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__50(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1420]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2839])))) {
            vlTOPp->_sequent__TOP__63(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__64(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__51(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1420]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2839])))) {
            vlTOPp->_sequent__TOP__65(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__66(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__52(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1420]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2839])))) {
            vlTOPp->_sequent__TOP__67(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__68(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__53(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1420]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2839])))) {
            vlTOPp->_sequent__TOP__69(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__70(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__54(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1420]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2839])))) {
            vlTOPp->_sequent__TOP__71(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__72(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__55(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1420]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2839])))) {
            vlTOPp->_sequent__TOP__73(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__74(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__56(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1420]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2839])))) {
            vlTOPp->_sequent__TOP__75(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__76(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__57(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1420]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2839])))) {
            vlTOPp->_sequent__TOP__77(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__78(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__58(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1420]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2839])))) {
            vlTOPp->_sequent__TOP__79(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__80(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__59(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1420]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2839])))) {
            vlTOPp->_sequent__TOP__81(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__82(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__60(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1420]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2839])))) {
            vlTOPp->_sequent__TOP__83(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__84(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__61(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1420]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2839])))) {
            vlTOPp->_sequent__TOP__85(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__86(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__62(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1420]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2839])))) {
            vlTOPp->_sequent__TOP__87(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__88(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__63(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1420]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2839])))) {
            vlTOPp->_sequent__TOP__89(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__90(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__64(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1420]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2839])))) {
            vlTOPp->_sequent__TOP__91(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__92(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__65(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1420]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2839])))) {
            vlTOPp->_sequent__TOP__93(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__94(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__66(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1420]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2839])))) {
            vlTOPp->_sequent__TOP__95(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__96(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__11(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1420]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2839])))) {
            vlTOPp->_sequent__TOP__97(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__98(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__12(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1420]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2839])))) {
            vlTOPp->_sequent__TOP__99(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__100(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__13(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1420]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2839])))) {
            vlTOPp->_sequent__TOP__101(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__102(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__14(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1420]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2839])))) {
            vlTOPp->_sequent__TOP__103(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__104(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__15(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1420]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2839])))) {
            vlTOPp->_sequent__TOP__105(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__106(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }
} // end of namespace RF ========================================
