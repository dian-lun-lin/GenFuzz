#include "hip/hip_runtime.h"
// Verilated -*- C++ -*-
// DESCRIPTION: Verilator output: Design implementation internals
// See Top.h for the primary calling header

#include "Top.h"
#include "Top__Syms.h"
// begin of namespace RF =====================================
namespace RF {

    //==========

    Top::Top(VerilatedContext* _vcontextp__, const char* _vcname__)
      : VerilatedModule{_vcname__}
     {
        Top__Syms* __restrict vlSymsp = __VlSymsp = new Top__Syms(_vcontextp__, this, name());
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Reset internal values

    }

    void Top::__Vconfigure(Top__Syms* vlSymsp, bool first) {
        if (false && first) {}  // Prevent unused
        this->__VlSymsp = vlSymsp;
        if (false && this->__VlSymsp) {}  // Prevent unused
        vlSymsp->_vm_contextp__->timeunit(-12);
        vlSymsp->_vm_contextp__->timeprecision(-12);
    }

    Top::~Top() {
        VL_DO_CLEAR(delete __VlSymsp, __VlSymsp = nullptr);
    }

    void Top::_initial__TOP__1(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_initial__TOP__1\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
#pragma omp parallel for
        for(size_t i = 0; i < BATCH_SIZE; ++i) {
            // Body
            _csignals[i + BATCH_SIZE * 1153] = 0U;
            _csignals[i + BATCH_SIZE * 1154] = 0U;
            _csignals[i + BATCH_SIZE * 1155] = 0U;
        }
    }

    __device__
    void Top::_settle__TOP__2(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_settle__TOP__2\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Variables
        // Begin mtask footprint all: 
        RfWide<3>/*95:0*/ __Vtemp1;
        RfWide<3>/*95:0*/ __Vtemp2;
        RfWide<3>/*95:0*/ __Vtemp4;
        RfWide<4>/*127:0*/ __Vtemp5;
        RfWide<4>/*127:0*/ __Vtemp6;
        RfWide<3>/*95:0*/ __Vtemp9;
        RfWide<4>/*127:0*/ __Vtemp11;
        RfWide<4>/*127:0*/ __Vtemp12;
        RfWide<4>/*127:0*/ __Vtemp17;
        RfWide<4>/*127:0*/ __Vtemp18;
        RfWide<4>/*127:0*/ __Vtemp19;
        RfWide<3>/*95:0*/ __Vtemp20;
        RfWide<3>/*95:0*/ __Vtemp21;
        RfWide<4>/*127:0*/ __Vtemp22;
        RfWide<5>/*159:0*/ __Vtemp24;
        RfWide<6>/*191:0*/ __Vtemp26;
        RfWide<7>/*223:0*/ __Vtemp28;
        RfWide<7>/*223:0*/ __Vtemp29;
        RfWide<8>/*255:0*/ __Vtemp30;
        RfWide<9>/*287:0*/ __Vtemp31;
        RfWide<10>/*319:0*/ __Vtemp33;
        // Body
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1014] 
            = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1292];
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1015] 
            = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1295];
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1016] 
            = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1298];
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1017] 
            = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1301];
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1018] 
            = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1304];
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1019] 
            = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1307];
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1020] 
            = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1310];
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1021] 
            = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1313];
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1022] 
            = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1316];
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1023] 
            = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1319];
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1024] 
            = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1322];
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1025] 
            = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1325];
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1026] 
            = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1328];
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1027] 
            = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1331];
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1028] 
            = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1334];
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1029] 
            = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1337];
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1030] 
            = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1340];
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1031] 
            = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1343];
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1032] 
            = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1346];
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1033] 
            = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1349];
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1034] 
            = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1352];
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1035] 
            = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1355];
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1036] 
            = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1358];
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1037] 
            = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1361];
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1038] 
            = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1364];
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1039] 
            = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1367];
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1040] 
            = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1370];
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1041] 
            = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1373];
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1042] 
            = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1376];
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1043] 
            = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1379];
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1044] 
            = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1382];
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1045] 
            = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1385];
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1046] 
            = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1388];
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1047] 
            = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1391];
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1048] 
            = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1394];
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1049] 
            = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1397];
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1050] 
            = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1400];
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1051] 
            = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1403];
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1052] 
            = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1406];
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1053] 
            = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1409];
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1054] 
            = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1412];
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1055] 
            = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1415];
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1056] 
            = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1418];
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1057] 
            = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421];
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1058] 
            = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1424];
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1059] 
            = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1427];
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1060] 
            = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1430];
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1061] 
            = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1433];
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1062] 
            = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1436];
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1063] 
            = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1439];
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1064] 
            = _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1442];
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1197] 
            = (0U == (IData)(_ssignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 7]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1231] 
            = (1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1196])));
        _ssignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10] 
            = (0x3fffU & ((IData)(_ssignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 7]) 
                          - (IData)(1U)));
        _ssignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 11] 
            = (0xffffU & ((IData)(_ssignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 8]) 
                          - (IData)(1U)));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1269] 
            = (1U & ((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1247]) 
                     - (IData)(1U)));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1270] 
            = (1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1247])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1274] 
            = (1U & ((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1252]) 
                     - (IData)(1U)));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1275] 
            = (1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1252])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1278] 
            = (1U & ((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1256]) 
                     - (IData)(1U)));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1279] 
            = (1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1256])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1280] 
            = (1U & ((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1257]) 
                     - (IData)(1U)));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1281] 
            = (1U & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1257])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1294] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1292])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1297] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1295])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1300] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1298])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1303] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1301])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1306] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1304])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1309] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1307])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1312] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1310])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1315] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1313])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1318] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1316])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1321] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1319])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1324] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1322])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1327] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1325])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1330] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1328])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1333] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1331])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1336] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1334])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1339] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1337])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1342] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1340])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1345] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1343])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1348] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1346])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1351] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1349])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1354] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1352])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1357] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1355])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1360] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1358])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1363] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1361])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1366] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1364])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1369] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1367])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1372] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1370])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1375] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1373])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1378] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1376])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1381] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1379])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1384] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1382])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1387] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1385])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1390] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1388])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1393] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1391])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1396] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1394])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1399] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1397])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1402] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1400])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1405] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1403])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1408] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1406])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1411] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1409])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1414] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1412])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1417] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1415])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1420] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1418])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1423] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1426] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1424])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1429] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1427])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1432] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1430])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1435] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1433])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1438] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1436])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1441] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1439])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1444] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1442])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1447] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1445])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1450] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1448])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1453] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1451])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1456] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1454])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1459] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1457])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1462] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1460])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1465] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1463])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1468] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1466])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1471] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1469])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1474] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1472])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1477] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1475])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1480] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1478])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1483] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1481])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1486] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1484])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1489] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1487])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1492] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1490])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1495] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1493])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1498] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1496])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1501] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1499])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1504] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1502])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1507] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1505])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1510] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1508])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1513] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1511])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1516] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1514])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1519] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1517])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1522] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1520])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1525] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1523])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1528] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1526])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1531] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1529])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1534] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1532])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1537] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1535])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1540] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1538])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1543] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1541])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1546] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1544])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1549] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1547])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1552] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1550])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1555] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1553])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1558] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1556])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1561] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1559])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1564] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1562])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1567] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1565])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1570] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1568])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1573] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1571])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1576] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1574])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1579] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1577])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1582] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1580])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1585] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1583])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1588] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1586])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1591] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1589])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1594] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1592])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1597] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1595])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1600] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1598])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1603] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1601])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1606] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1604])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1609] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1607])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1612] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1610])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1615] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1613])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1618] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1616])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1621] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1619])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1624] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1622])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1627] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1625])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1630] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1628])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1633] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1631])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1636] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1634])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1639] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1637])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1642] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1640])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1645] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1643])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1648] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1646])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1651] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1649])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1654] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1652])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1657] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1655])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1660] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1658])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1663] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1661])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1666] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1664])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1669] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1667])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1672] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1670])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1675] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1673])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1678] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1676])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1681] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1679])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1684] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1682])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1687] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1685])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1690] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1688])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1693] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1691])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1696] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1694])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1699] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1697])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1702] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1700])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1705] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1703])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1708] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1706])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1711] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1709])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1714] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1712])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1717] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1715])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1720] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1718])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1723] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1721])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1726] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1724])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1729] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1727])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1732] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1730])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1735] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1733])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1738] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1736])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1741] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1739])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1744] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1742])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1747] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1745])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1750] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1748])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1753] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1751])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1756] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1754])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1759] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1757])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1762] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1760])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1765] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1763])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1768] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1766])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1771] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1769])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1774] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1772])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1777] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1775])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1780] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1778])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1783] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1781])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1786] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1784])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1789] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1787])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1792] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1790])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1795] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1793])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1798] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1796])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1801] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1799])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1804] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1802])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1807] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1805])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1810] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1808])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1813] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1811])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1816] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1814])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1819] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1817])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1822] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1820])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1825] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1823])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1828] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1826])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1831] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1829])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1834] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1832])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1837] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1835])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1840] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1838])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1843] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1841])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1846] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1844])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1849] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1847])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1852] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1850])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1855] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1853])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1858] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1856])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1861] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1859])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1864] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1862])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1867] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1865])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1870] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1868])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1873] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1871])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1876] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1874])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1879] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1877])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1882] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1880])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1885] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1883])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1888] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1886])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1891] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1889])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1894] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1892])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1897] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1895])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1900] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1898])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1903] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1901])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1906] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1904])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1909] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1907])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1912] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1910])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1915] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1913])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1918] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1916])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1921] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1919])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1924] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1922])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1927] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1925])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1930] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1928])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1933] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1931])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1936] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1934])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1939] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1937])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1942] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1940])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1945] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1943])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1948] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1946])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1951] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1949])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1954] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1952])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1957] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1955])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1960] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1958])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1963] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1961])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1966] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1964])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1969] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1967])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1972] 
            = (1U & ((IData)(1U) + (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1970])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1065] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1445]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1448]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1066] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1451]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1454]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1067] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1457]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1460]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1068] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1463]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1466]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1069] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1469]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1472]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1070] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1475]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1478]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1071] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1481]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1484]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1072] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1487]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1490]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1073] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1493]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1496]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1074] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1499]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1502]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1075] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1505]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1508]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1076] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1511]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1514]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1077] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1517]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1520]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1078] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1523]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1526]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1079] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1529]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1532]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1080] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1535]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1538]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1081] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1541]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1544]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1082] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1547]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1550]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1083] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1553]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1556]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1084] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1559]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1562]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1085] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1565]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1568]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1086] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1571]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1574]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1087] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1577]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1580]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1088] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1583]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1586]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1089] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1589]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1592]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1090] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1595]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1598]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1091] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1601]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1604]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1092] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1607]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1610]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1093] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1613]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1616]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1094] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1619]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1622]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1095] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1625]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1628]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1096] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1631]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1634]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1097] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1637]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1640]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1098] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1643]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1646]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1099] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1649]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1652]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1100] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1655]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1658]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1101] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1661]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1664]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1102] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1667]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1670]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1103] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1673]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1676]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1104] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1679]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1682]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1105] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1685]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1688]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1106] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1691]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1694]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1107] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1697]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1700]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1108] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1703]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1706]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1109] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1709]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1712]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1110] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1715]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1718]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1111] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1721]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1724]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1112] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1727]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1730]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1113] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1733]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1736]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1114] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1739]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1742]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1115] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1745]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1748]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1116] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1751]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1754]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1117] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1757]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1760]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1118] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1763]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1766]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1119] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1769]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1772]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1120] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1775]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1778]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1121] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1781]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1784]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1122] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1787]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1790]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1123] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1793]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1796]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1124] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1799]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1802]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1125] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1805]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1808]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1126] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1811]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1814]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1127] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1817]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1820]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1128] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1823]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1826]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1129] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1829]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1832]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1130] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1835]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1838]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1131] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1841]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1844]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1132] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1847]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1850]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1133] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1853]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1856]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1134] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1859]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1862]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1135] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1865]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1868]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1136] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1871]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1874]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1137] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1877]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1880]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1138] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1883]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1886]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1139] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1889]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1892]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1140] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1895]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1898]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1141] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1901]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1904]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1142] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1907]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1910]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1143] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1913]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1916]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1144] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1919]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1922]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1145] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1925]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1928]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1146] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1931]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1934]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1147] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1937]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1940]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1148] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1943]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1946]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1149] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1949]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1952]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1150] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1955]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1958]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1151] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1961]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1964]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1152] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1967]) 
                << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1970]));
        _ssignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 9] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1159]) 
                << 8U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1160]));
        __Vtemp1[2U] = ((0xffffU & ((IData)((((QData)((IData)(
                                                              (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 990]) 
                                                                << 0x18U) 
                                                               | (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 991]) 
                                                                   << 0x10U) 
                                                                  | (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 992]) 
                                                                      << 8U) 
                                                                     | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 993])))))) 
                                              << 0x10U) 
                                             | (QData)((IData)(
                                                               (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 994]) 
                                                                 << 8U) 
                                                                | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 995])))))) 
                                    >> 0x10U)) | (0xffff0000U 
                                                  & ((IData)(
                                                             ((((QData)((IData)(
                                                                                (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 990]) 
                                                                                << 0x18U) 
                                                                                | (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 991]) 
                                                                                << 0x10U) 
                                                                                | (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 992]) 
                                                                                << 8U) 
                                                                                | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 993])))))) 
                                                                << 0x10U) 
                                                               | (QData)((IData)(
                                                                                (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 994]) 
                                                                                << 8U) 
                                                                                | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 995]))))) 
                                                              >> 0x20U)) 
                                                     << 0x10U)));
        __Vtemp2[2U] = ((0xffffU & ((IData)((((QData)((IData)(
                                                              (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1002]) 
                                                                << 0x18U) 
                                                               | (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1003]) 
                                                                   << 0x10U) 
                                                                  | (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1004]) 
                                                                      << 8U) 
                                                                     | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1005])))))) 
                                              << 0x10U) 
                                             | (QData)((IData)(
                                                               (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1006]) 
                                                                 << 8U) 
                                                                | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1007])))))) 
                                    >> 0x10U)) | (0xffff0000U 
                                                  & ((IData)(
                                                             ((((QData)((IData)(
                                                                                (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1002]) 
                                                                                << 0x18U) 
                                                                                | (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1003]) 
                                                                                << 0x10U) 
                                                                                | (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1004]) 
                                                                                << 8U) 
                                                                                | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1005])))))) 
                                                                << 0x10U) 
                                                               | (QData)((IData)(
                                                                                (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1006]) 
                                                                                << 8U) 
                                                                                | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1007]))))) 
                                                              >> 0x20U)) 
                                                     << 0x10U)));
        (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[0U] 
            = (IData)((((QData)((IData)((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1008]) 
                                          << 0x18U) 
                                         | (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1009]) 
                                             << 0x10U) 
                                            | (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1010]) 
                                                << 8U) 
                                               | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1011])))))) 
                        << 0x10U) | (QData)((IData)(
                                                    (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1012]) 
                                                      << 8U) 
                                                     | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1013]))))));
        (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[1U] 
            = ((0xffff0000U & ((IData)((((QData)((IData)(
                                                         (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1002]) 
                                                           << 0x18U) 
                                                          | (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1003]) 
                                                              << 0x10U) 
                                                             | (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1004]) 
                                                                 << 8U) 
                                                                | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1005])))))) 
                                         << 0x10U) 
                                        | (QData)((IData)(
                                                          (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1006]) 
                                                            << 8U) 
                                                           | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1007])))))) 
                               << 0x10U)) | (IData)(
                                                    ((((QData)((IData)(
                                                                       (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1008]) 
                                                                         << 0x18U) 
                                                                        | (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1009]) 
                                                                            << 0x10U) 
                                                                           | (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1010]) 
                                                                               << 8U) 
                                                                              | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1011])))))) 
                                                       << 0x10U) 
                                                      | (QData)((IData)(
                                                                        (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1012]) 
                                                                          << 8U) 
                                                                         | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1013]))))) 
                                                     >> 0x20U)));
        (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[2U] 
            = __Vtemp2[2U];
        (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[3U] 
            = (IData)((((QData)((IData)((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 996]) 
                                          << 0x18U) 
                                         | (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 997]) 
                                             << 0x10U) 
                                            | (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 998]) 
                                                << 8U) 
                                               | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 999])))))) 
                        << 0x10U) | (QData)((IData)(
                                                    (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1000]) 
                                                      << 8U) 
                                                     | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1001]))))));
        (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[4U] 
            = ((0xffff0000U & ((IData)((((QData)((IData)(
                                                         (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 990]) 
                                                           << 0x18U) 
                                                          | (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 991]) 
                                                              << 0x10U) 
                                                             | (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 992]) 
                                                                 << 8U) 
                                                                | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 993])))))) 
                                         << 0x10U) 
                                        | (QData)((IData)(
                                                          (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 994]) 
                                                            << 8U) 
                                                           | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 995])))))) 
                               << 0x10U)) | (IData)(
                                                    ((((QData)((IData)(
                                                                       (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 996]) 
                                                                         << 0x18U) 
                                                                        | (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 997]) 
                                                                            << 0x10U) 
                                                                           | (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 998]) 
                                                                               << 8U) 
                                                                              | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 999])))))) 
                                                       << 0x10U) 
                                                      | (QData)((IData)(
                                                                        (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1000]) 
                                                                          << 8U) 
                                                                         | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1001]))))) 
                                                     >> 0x20U)));
        (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[5U] 
            = __Vtemp1[2U];
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1199] 
            = (8U == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1186]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1200] 
            = (4U == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1186]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1201] 
            = (1U == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1186]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1202] 
            = (0x11U == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1203] 
            = (0x10U == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1204] 
            = (0xfU == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1205] 
            = (0xeU == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1206] 
            = (0xdU == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1207] 
            = (0xcU == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1208] 
            = (0xbU == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1209] 
            = (0xaU == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1210] 
            = (9U == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1211] 
            = (8U == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1212] 
            = (7U == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1213] 
            = (6U == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1214] 
            = (5U == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1215] 
            = (4U == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1216] 
            = (3U == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1217] 
            = (2U == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1218] 
            = (1U == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1219] 
            = (0U == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1222] 
            = (5U == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1195]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1223] 
            = (4U == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1195]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1224] 
            = (2U == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1195]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1225] 
            = (3U == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1195]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1226] 
            = (1U == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1195]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1227] 
            = (0U == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1195]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1243] 
            = (2U == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1186]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1244] 
            = (7U & ((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1194]) 
                     - (IData)(1U)));
        _ssignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 13] 
            = ((0x1fcU & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[2U] 
                           << 3U) | (4U & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[1U] 
                                           >> 0x1dU)))) 
               | (3U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[2U] 
                         << 0x1aU) | ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[1U] 
                                      >> 6U))));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1288] 
            = (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[1U] 
                >> 3U) & (1U == (7U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[2U] 
                                        << 0xfU) | 
                                       ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[1U] 
                                        >> 0x11U)))));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1289] 
            = (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[1U] 
                >> 3U) & (2U == (7U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[2U] 
                                        << 0xfU) | 
                                       ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[1U] 
                                        >> 0x11U)))));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1290] 
            = (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[1U] 
                >> 3U) & (3U == (7U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[2U] 
                                        << 0xfU) | 
                                       ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[1U] 
                                        >> 0x11U)))));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1291] 
            = (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[1U] 
                >> 3U) & (5U == (7U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[2U] 
                                        << 0xfU) | 
                                       ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[1U] 
                                        >> 0x11U)))));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1271] 
            = (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[1U] 
                >> 3U) & (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1247]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1284] 
            = (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[1U] 
                >> 3U) & (6U == (7U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[2U] 
                                        << 0xfU) | 
                                       ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[1U] 
                                        >> 0x11U)))));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1285] 
            = (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[1U] 
                >> 3U) & (7U == (7U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[2U] 
                                        << 0xfU) | 
                                       ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[1U] 
                                        >> 0x11U)))));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1286] 
            = (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[1U] 
                >> 3U) & (4U == (7U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[2U] 
                                        << 0xfU) | 
                                       ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[1U] 
                                        >> 0x11U)))));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1261] 
            = ((2U <= (3U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[2U] 
                              << 0x1aU) | ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[1U] 
                                           >> 6U)))) 
               | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 989]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1263] 
            = ((2U >= (7U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[2U] 
                              << 0x12U) | ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[1U] 
                                           >> 0xeU)))) 
               | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 989]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1264] 
            = ((0U == (0xfU & (~ (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[2U] 
                                   << 0xcU) | ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[1U] 
                                               >> 0x14U))))) 
               | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 989]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1287] 
            = (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[1U] 
                >> 3U) & (0U == (7U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[2U] 
                                        << 0xfU) | 
                                       ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[1U] 
                                        >> 0x11U)))));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1262] 
            = ((0U == (3U & ((((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[4U] 
                               << 0x1dU) | ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[3U] 
                                            >> 3U)) 
                             & (~ (0x1fU & ((IData)(3U) 
                                            << (3U 
                                                & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[2U] 
                                                    << 0x1aU) 
                                                   | ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[1U] 
                                                      >> 6U))))))))) 
               | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 989]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1265] 
            = (((2U >= (3U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[2U] 
                               << 0x1aU) | ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[1U] 
                                            >> 6U)))) 
                & (0U == (0x1ffff000U & (0x10016000U 
                                         ^ (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[4U] 
                                             << 0x1dU) 
                                            | ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[3U] 
                                               >> 3U)))))) 
               | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 989]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1266] 
            = ((0U == (7U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[2U] 
                              << 0x12U) | ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[1U] 
                                           >> 0xeU)))) 
               | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 989]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1238] 
            = ((7U == (7U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[4U] 
                              << 0x1bU) | ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[3U] 
                                           >> 5U)))) 
               | ((6U == (7U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[4U] 
                                 << 0x1bU) | ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[3U] 
                                              >> 5U)))) 
                  | ((5U == (7U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[4U] 
                                    << 0x1bU) | ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[3U] 
                                                 >> 5U)))) 
                     | ((4U == (7U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[4U] 
                                       << 0x1bU) | 
                                      ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[3U] 
                                       >> 5U)))) ? 
                        (0U == (0x3f8U & (4U ^ (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[4U] 
                                                 << 0x1bU) 
                                                | ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[3U] 
                                                   >> 5U)))))
                         : ((3U == (7U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[4U] 
                                           << 0x1bU) 
                                          | ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[3U] 
                                             >> 5U))))
                             ? (0U == (0x3f8U & (3U 
                                                 ^ 
                                                 (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[4U] 
                                                   << 0x1bU) 
                                                  | ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[3U] 
                                                     >> 5U)))))
                             : ((2U == (7U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[4U] 
                                               << 0x1bU) 
                                              | ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[3U] 
                                                 >> 5U))))
                                 ? (0U == (0x3f8U & 
                                           (2U ^ ((
                                                   (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[4U] 
                                                   << 0x1bU) 
                                                  | ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[3U] 
                                                     >> 5U)))))
                                 : ((1U == (7U & ((
                                                   (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[4U] 
                                                   << 0x1bU) 
                                                  | ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[3U] 
                                                     >> 5U))))
                                     ? (0U == (0x3f8U 
                                               & (1U 
                                                  ^ 
                                                  (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[4U] 
                                                    << 0x1bU) 
                                                   | ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[3U] 
                                                      >> 5U)))))
                                     : (0U == (0x3f8U 
                                               & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[4U] 
                                                   << 0x1bU) 
                                                  | ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[3U] 
                                                     >> 5U)))))))))));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1258] 
            = (1U & ((2U <= (3U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[2U] 
                                    << 0x1aU) | ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[1U] 
                                                 >> 6U)))) 
                     | (1U & ((((IData)(1U) << (1U 
                                                & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[1U] 
                                                   >> 6U))) 
                               >> 1U) & (~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[3U] 
                                            >> 4U))))));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1259] 
            = (1U & ((2U <= (3U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[2U] 
                                    << 0x1aU) | ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[1U] 
                                                 >> 6U)))) 
                     | (1U & ((((IData)(1U) << (1U 
                                                & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[1U] 
                                                   >> 6U))) 
                               >> 1U) & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[3U] 
                                         >> 4U)))));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1228] 
            = ((((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[1U] 
                 >> 3U) & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[0U] 
                           >> 0x1fU)) & (4U != (7U 
                                                & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[2U] 
                                                    << 0xfU) 
                                                   | ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[1U] 
                                                      >> 0x11U)))));
        _isignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 114] 
            = ((((0x800000U & (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[1U])
                  ? 0xffU : 0U) << 0x18U) | ((((0x400000U 
                                                & (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[1U])
                                                ? 0xffU
                                                : 0U) 
                                              << 0x10U) 
                                             | ((((0x200000U 
                                                   & (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[1U])
                                                   ? 0xffU
                                                   : 0U) 
                                                 << 8U) 
                                                | ((0x100000U 
                                                    & (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[1U])
                                                    ? 0xffU
                                                    : 0U))));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1230] 
            = ((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1193]) 
               | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1190]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1232] 
            = (0xffU & ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1228]) 
                          & ((IData)(1U) << (7U & (
                                                   ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[4U] 
                                                    << 0x1bU) 
                                                   | ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[3U] 
                                                      >> 5U))))) 
                         & (0U == (0x3f8U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[4U] 
                                              << 0x1bU) 
                                             | ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[3U] 
                                                >> 5U))))) 
                        & (0U == (0xffU & (~ _isignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 114])))));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1233] 
            = (0x7fU & ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1228]) 
                          & (((IData)(1U) << (7U & 
                                              (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[4U] 
                                                << 0x1bU) 
                                               | ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[3U] 
                                                  >> 5U)))) 
                             >> 1U)) & (0U == (0x3f8U 
                                               & (1U 
                                                  ^ 
                                                  (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[4U] 
                                                    << 0x1bU) 
                                                   | ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[3U] 
                                                      >> 5U)))))) 
                        & (0U == (0xffU & (~ _isignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 114])))));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1237] 
            = (0x1fU & ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1228]) 
                          & (((IData)(1U) << (7U & 
                                              (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[4U] 
                                                << 0x1bU) 
                                               | ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[3U] 
                                                  >> 5U)))) 
                             >> 3U)) & (0U == (0x3f8U 
                                               & (3U 
                                                  ^ 
                                                  (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[4U] 
                                                    << 0x1bU) 
                                                   | ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[3U] 
                                                      >> 5U)))))) 
                        & (0U == (0xffU & (~ _isignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 114])))));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1229] 
            = (0xfU & ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1228]) 
                         & (((IData)(1U) << (7U & (
                                                   ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[4U] 
                                                    << 0x1bU) 
                                                   | ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[3U] 
                                                      >> 5U)))) 
                            >> 4U)) & (0U == (0x3f8U 
                                              & (4U 
                                                 ^ 
                                                 (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[4U] 
                                                   << 0x1bU) 
                                                  | ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[3U] 
                                                     >> 5U)))))) 
                       & (0U == (0xffU & (~ _isignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 114])))));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1260] 
            = (((8U & (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1259]) 
                        << 3U) | (0x7ffffff8U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[3U] 
                                                  >> 1U) 
                                                 & (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[3U])))) 
                | (4U & (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1259]) 
                          << 2U) | (0x3ffffffcU & (
                                                   ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[3U] 
                                                    >> 2U) 
                                                   & ((~ 
                                                       ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[3U] 
                                                        >> 3U)) 
                                                      << 2U)))))) 
               | ((2U & (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1258]) 
                          << 1U) | (0x3ffffffeU & (
                                                   ((~ 
                                                     ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[3U] 
                                                      >> 4U)) 
                                                    << 1U) 
                                                   & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[3U] 
                                                      >> 2U))))) 
                  | (1U & ((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1258]) 
                           | ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[3U] 
                                  >> 4U)) & (~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[3U] 
                                                >> 3U)))))));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1234] 
            = (0x3fU & (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1228]) 
                         & (((IData)(1U) << (7U & (
                                                   ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[4U] 
                                                    << 0x1bU) 
                                                   | ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[3U] 
                                                      >> 5U)))) 
                            >> 2U)) & (0U == (0x3f8U 
                                              & (2U 
                                                 ^ 
                                                 (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[4U] 
                                                   << 0x1bU) 
                                                  | ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[3U] 
                                                     >> 5U)))))));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1246] 
            = ((4U != (7U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[2U] 
                              << 0xfU) | ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[1U] 
                                          >> 0x11U)))) 
               | ((7U == (7U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[4U] 
                                 << 0x1bU) | ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[3U] 
                                              >> 5U)))) 
                  | ((6U == (7U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[4U] 
                                    << 0x1bU) | ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[3U] 
                                                 >> 5U)))) 
                     | ((5U == (7U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[4U] 
                                       << 0x1bU) | 
                                      ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[3U] 
                                       >> 5U)))) | 
                        ((4U != (7U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[4U] 
                                        << 0x1bU) | 
                                       ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[3U] 
                                        >> 5U)))) | 
                         (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1196]) 
                           | (0U == (0xffU & _isignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 114]))) 
                          | (0U != (0x3f8U & (4U ^ 
                                              (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[4U] 
                                                << 0x1bU) 
                                               | ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[3U] 
                                                  >> 5U)))))))))));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1242] 
            = (1U & ((((~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1181])) 
                       & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1176]))) 
                      & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1172]))) 
                     | ((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1179]) 
                        & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1172])))));
        _ssignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 12] 
            = ((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1191])
                ? (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1162])
                : ((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1192])
                    ? (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1163]) 
                        << 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1185]))
                    : (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1163])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1245] 
            = (0xffU & ((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1229])
                         ? (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[5U]
                         : (0xfeU & (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1164]) 
                                      << 7U) | (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1165]) 
                                                 << 6U) 
                                                | (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1166]) 
                                                    << 5U) 
                                                   | (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1167]) 
                                                       << 4U) 
                                                      | (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1168]) 
                                                          << 3U) 
                                                         | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1169])))))))));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1267] 
            = (((0xfU & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[2U] 
                          << 0xcU) | ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[1U] 
                                      >> 0x14U))) == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1260])) 
               | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 989]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1239] 
            = ((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1172]) 
               & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1174])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1235] 
            = ((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1234]) 
               & (_isignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 114] 
                  >> 6U));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1236] 
            = ((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1234]) 
               & (_isignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 114] 
                  >> 7U));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1157] 
            = (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[0U] 
                >> 0x1fU) & (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1246]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1158] 
            = (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[1U] 
                >> 3U) & (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1246]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1220] 
            = ((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1166]) 
               | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1167]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1198] 
            = (1U & (((0U == (IData)(_ssignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 8])) 
                      | (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1161]))) 
                     | (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1174]) 
                         & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1172]))) 
                        & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1181])))));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1276] 
            = ((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1158]) 
               & (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1252]));
        _isignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 88] 
            = ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1284]) 
                 & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 989]))) 
                << 0x17U) | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1284]) 
                               & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 989]))) 
                              << 0x16U) | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1284]) 
                                             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1261]))) 
                                            << 0x14U) 
                                           | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1284]) 
                                                & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1262]))) 
                                               << 0x13U) 
                                              | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1284]) 
                                                   & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1263]))) 
                                                  << 0x12U) 
                                                 | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1284]) 
                                                      & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1264]))) 
                                                     << 0x11U) 
                                                    | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1285]) 
                                                         & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 989]))) 
                                                        << 0x10U) 
                                                       | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1285]) 
                                                            & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 989]))) 
                                                           << 0xfU) 
                                                          | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1285]) 
                                                               & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1261]))) 
                                                              << 0xdU) 
                                                             | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1285]) 
                                                                  & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1262]))) 
                                                                 << 0xcU) 
                                                                | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1285]) 
                                                                     & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1263]))) 
                                                                    << 0xbU) 
                                                                   | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1285]) 
                                                                        & (~ 
                                                                           ((0U 
                                                                             != 
                                                                             (7U 
                                                                              & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[2U] 
                                                                                << 0x12U) 
                                                                                | ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[1U] 
                                                                                >> 0xeU)))) 
                                                                            | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 989])))) 
                                                                       << 0xaU) 
                                                                      | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1285]) 
                                                                           & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1264]))) 
                                                                          << 9U) 
                                                                         | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1286]) 
                                                                              & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1265]))) 
                                                                             << 8U) 
                                                                            | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1286]) 
                                                                                & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1262]))) 
                                                                                << 6U) 
                                                                               | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1286]) 
                                                                                & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1266]))) 
                                                                                << 5U) 
                                                                                | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1286]) 
                                                                                & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1267]))) 
                                                                                << 4U) 
                                                                                | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1287]) 
                                                                                & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1265]))) 
                                                                                << 3U) 
                                                                                | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1287]) 
                                                                                & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1262]))) 
                                                                                << 1U) 
                                                                                | ((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1287]) 
                                                                                & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1266]))))))))))))))))))))));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1268] 
            = ((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1157]) 
               & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[1U] 
                  >> 3U));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1273] 
            = (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[0U] 
                >> 0x1fU) & (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1158]));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1221] 
            = (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1220]) 
                | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1165])) 
               & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1193])));
        _isignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 95] 
            = (((0U == (IData)(_ssignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 7])) 
                << 0x1bU) | ((0x4000000U & ((~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1161])) 
                                            << 0x1aU)) 
                             | (((0U == (IData)(_ssignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 7])) 
                                 << 0x19U) | (((0U 
                                                == (IData)(_ssignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 7])) 
                                               << 0x18U) 
                                              | (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1179]) 
                                                  << 0x17U) 
                                                 | (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1198]) 
                                                     << 0x16U) 
                                                    | (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1180]) 
                                                        << 0x15U) 
                                                       | (((8U 
                                                            == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1186])) 
                                                           << 0x14U) 
                                                          | (((4U 
                                                               == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1186])) 
                                                              << 0x13U) 
                                                             | (((2U 
                                                                  == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1186])) 
                                                                 << 0x12U) 
                                                                | (((1U 
                                                                     == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1186])) 
                                                                    << 0x11U) 
                                                                   | (((0x11U 
                                                                        == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                                       << 0x10U) 
                                                                      | (((0x11U 
                                                                           == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                                          << 0xfU) 
                                                                         | (((0x11U 
                                                                              == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                                             << 0xeU) 
                                                                            | (((0x11U 
                                                                                == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                                                << 0xdU) 
                                                                               | (((0x10U 
                                                                                == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                                                << 0xcU) 
                                                                                | (((0x10U 
                                                                                == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                                                << 0xbU) 
                                                                                | (((0x10U 
                                                                                == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                                                << 0xaU) 
                                                                                | (((0x10U 
                                                                                == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                                                << 9U) 
                                                                                | (((0x10U 
                                                                                == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                                                << 8U) 
                                                                                | (((0xfU 
                                                                                == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                                                << 7U) 
                                                                                | (((0xfU 
                                                                                == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                                                << 6U) 
                                                                                | (((0xfU 
                                                                                == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                                                << 5U) 
                                                                                | (((0xfU 
                                                                                == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                                                << 4U) 
                                                                                | (((0xfU 
                                                                                == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                                                << 3U) 
                                                                                | (((0xeU 
                                                                                == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                                                << 2U) 
                                                                                | (((0xeU 
                                                                                == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                                                << 1U) 
                                                                                | (0xeU 
                                                                                == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])))))))))))))))))))))))))))));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1272] 
            = ((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1268]) 
               & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1247])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1277] 
            = ((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1273]) 
               & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1252])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1240] 
            = ((0U == (IData)(_ssignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 7]))
                ? (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1170]) 
                    << 1U) | (1U & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[0U] 
                                    >> 0x1cU))) : (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1170]));
        __Vtemp4[0U] = ((0xffffff00U & ((IData)((((QData)((IData)(
                                                                  ((_isignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 88] 
                                                                    << 7U) 
                                                                   | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1287]) 
                                                                        & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1267]))) 
                                                                       << 6U) 
                                                                      | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1288]) 
                                                                           & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1265]))) 
                                                                          << 5U) 
                                                                         | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1288]) 
                                                                              & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1262]))) 
                                                                             << 3U) 
                                                                            | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1288]) 
                                                                                & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1266]))) 
                                                                                << 2U) 
                                                                               | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1288]) 
                                                                                & (~ 
                                                                                ((0U 
                                                                                == 
                                                                                (0xfU 
                                                                                & ((((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[2U] 
                                                                                << 0xcU) 
                                                                                | ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[1U] 
                                                                                >> 0x14U)) 
                                                                                & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1260]))))) 
                                                                                | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 989])))) 
                                                                                << 1U) 
                                                                                | ((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1289]) 
                                                                                & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 989]))))))))))) 
                                                  << 0x20U) 
                                                 | (QData)((IData)(
                                                                   ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1289]) 
                                                                      & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1262]))) 
                                                                     << 0x1eU) 
                                                                    | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1289]) 
                                                                         & (~ 
                                                                            ((4U 
                                                                              >= 
                                                                              (7U 
                                                                               & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[2U] 
                                                                                << 0x12U) 
                                                                                | ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[1U] 
                                                                                >> 0xeU)))) 
                                                                             | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 989])))) 
                                                                        << 0x1dU) 
                                                                       | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1289]) 
                                                                            & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1267]))) 
                                                                           << 0x1cU) 
                                                                          | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1290]) 
                                                                               & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 989]))) 
                                                                              << 0x1bU) 
                                                                             | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1290]) 
                                                                                & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1262]))) 
                                                                                << 0x19U) 
                                                                                | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1290]) 
                                                                                & (~ 
                                                                                ((3U 
                                                                                >= 
                                                                                (7U 
                                                                                & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[2U] 
                                                                                << 0x12U) 
                                                                                | ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[1U] 
                                                                                >> 0xeU)))) 
                                                                                | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 989])))) 
                                                                                << 0x18U) 
                                                                                | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1290]) 
                                                                                & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1267]))) 
                                                                                << 0x17U) 
                                                                                | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1291]) 
                                                                                & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 989]))) 
                                                                                << 0x16U) 
                                                                                | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1291]) 
                                                                                & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1262]))) 
                                                                                << 0x14U) 
                                                                                | (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1291]) 
                                                                                & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1267]))) 
                                                                                << 0x13U)))))))))))))) 
                                        << 8U)) | (
                                                   (8U 
                                                    & ((~ 
                                                        ((~ 
                                                          ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[1U] 
                                                           >> 1U)) 
                                                         | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 989]))) 
                                                       << 3U)) 
                                                   | ((4U 
                                                       & ((~ 
                                                           ((~ 
                                                             ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[0U] 
                                                              >> 0x1eU)) 
                                                            | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 989]))) 
                                                          << 2U)) 
                                                      | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1271]) 
                                                           & (~ 
                                                              (((7U 
                                                                 & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[2U] 
                                                                     << 0xfU) 
                                                                    | ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[1U] 
                                                                       >> 0x11U))) 
                                                                == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1248])) 
                                                               | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 989])))) 
                                                          << 1U) 
                                                         | ((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1271]) 
                                                            & (~ 
                                                               (((7U 
                                                                  & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[2U] 
                                                                      << 0x12U) 
                                                                     | ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[1U] 
                                                                        >> 0xeU))) 
                                                                 == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1249])) 
                                                                | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 989]))))))));
        __Vtemp4[1U] = ((0xffU & ((IData)((((QData)((IData)(
                                                            ((_isignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 88] 
                                                              << 7U) 
                                                             | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1287]) 
                                                                  & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1267]))) 
                                                                 << 6U) 
                                                                | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1288]) 
                                                                     & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1265]))) 
                                                                    << 5U) 
                                                                   | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1288]) 
                                                                        & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1262]))) 
                                                                       << 3U) 
                                                                      | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1288]) 
                                                                           & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1266]))) 
                                                                          << 2U) 
                                                                         | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1288]) 
                                                                              & (~ 
                                                                                ((0U 
                                                                                == 
                                                                                (0xfU 
                                                                                & ((((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[2U] 
                                                                                << 0xcU) 
                                                                                | ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[1U] 
                                                                                >> 0x14U)) 
                                                                                & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1260]))))) 
                                                                                | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 989])))) 
                                                                             << 1U) 
                                                                            | ((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1289]) 
                                                                               & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 989]))))))))))) 
                                            << 0x20U) 
                                           | (QData)((IData)(
                                                             ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1289]) 
                                                                & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1262]))) 
                                                               << 0x1eU) 
                                                              | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1289]) 
                                                                   & (~ 
                                                                      ((4U 
                                                                        >= 
                                                                        (7U 
                                                                         & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[2U] 
                                                                             << 0x12U) 
                                                                            | ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[1U] 
                                                                               >> 0xeU)))) 
                                                                       | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 989])))) 
                                                                  << 0x1dU) 
                                                                 | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1289]) 
                                                                      & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1267]))) 
                                                                     << 0x1cU) 
                                                                    | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1290]) 
                                                                         & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 989]))) 
                                                                        << 0x1bU) 
                                                                       | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1290]) 
                                                                            & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1262]))) 
                                                                           << 0x19U) 
                                                                          | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1290]) 
                                                                               & (~ 
                                                                                ((3U 
                                                                                >= 
                                                                                (7U 
                                                                                & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[2U] 
                                                                                << 0x12U) 
                                                                                | ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[1U] 
                                                                                >> 0xeU)))) 
                                                                                | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 989])))) 
                                                                              << 0x18U) 
                                                                             | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1290]) 
                                                                                & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1267]))) 
                                                                                << 0x17U) 
                                                                                | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1291]) 
                                                                                & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 989]))) 
                                                                                << 0x16U) 
                                                                                | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1291]) 
                                                                                & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1262]))) 
                                                                                << 0x14U) 
                                                                                | (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1291]) 
                                                                                & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1267]))) 
                                                                                << 0x13U)))))))))))))) 
                                  >> 0x18U)) | (0xffffff00U 
                                                & ((IData)(
                                                           ((((QData)((IData)(
                                                                              ((_isignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 88] 
                                                                                << 7U) 
                                                                               | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1287]) 
                                                                                & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1267]))) 
                                                                                << 6U) 
                                                                                | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1288]) 
                                                                                & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1265]))) 
                                                                                << 5U) 
                                                                                | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1288]) 
                                                                                & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1262]))) 
                                                                                << 3U) 
                                                                                | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1288]) 
                                                                                & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1266]))) 
                                                                                << 2U) 
                                                                                | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1288]) 
                                                                                & (~ 
                                                                                ((0U 
                                                                                == 
                                                                                (0xfU 
                                                                                & ((((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[2U] 
                                                                                << 0xcU) 
                                                                                | ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[1U] 
                                                                                >> 0x14U)) 
                                                                                & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1260]))))) 
                                                                                | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 989])))) 
                                                                                << 1U) 
                                                                                | ((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1289]) 
                                                                                & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 989]))))))))))) 
                                                              << 0x20U) 
                                                             | (QData)((IData)(
                                                                               ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1289]) 
                                                                                & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1262]))) 
                                                                                << 0x1eU) 
                                                                                | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1289]) 
                                                                                & (~ 
                                                                                ((4U 
                                                                                >= 
                                                                                (7U 
                                                                                & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[2U] 
                                                                                << 0x12U) 
                                                                                | ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[1U] 
                                                                                >> 0xeU)))) 
                                                                                | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 989])))) 
                                                                                << 0x1dU) 
                                                                                | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1289]) 
                                                                                & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1267]))) 
                                                                                << 0x1cU) 
                                                                                | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1290]) 
                                                                                & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 989]))) 
                                                                                << 0x1bU) 
                                                                                | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1290]) 
                                                                                & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1262]))) 
                                                                                << 0x19U) 
                                                                                | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1290]) 
                                                                                & (~ 
                                                                                ((3U 
                                                                                >= 
                                                                                (7U 
                                                                                & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[2U] 
                                                                                << 0x12U) 
                                                                                | ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[1U] 
                                                                                >> 0xeU)))) 
                                                                                | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 989])))) 
                                                                                << 0x18U) 
                                                                                | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1290]) 
                                                                                & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1267]))) 
                                                                                << 0x17U) 
                                                                                | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1291]) 
                                                                                & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 989]))) 
                                                                                << 0x16U) 
                                                                                | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1291]) 
                                                                                & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1262]))) 
                                                                                << 0x14U) 
                                                                                | (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1291]) 
                                                                                & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1267]))) 
                                                                                << 0x13U))))))))))))) 
                                                            >> 0x20U)) 
                                                   << 8U)));
        __Vtemp4[2U] = (0xffU & ((IData)(((((QData)((IData)(
                                                            ((_isignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 88] 
                                                              << 7U) 
                                                             | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1287]) 
                                                                  & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1267]))) 
                                                                 << 6U) 
                                                                | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1288]) 
                                                                     & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1265]))) 
                                                                    << 5U) 
                                                                   | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1288]) 
                                                                        & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1262]))) 
                                                                       << 3U) 
                                                                      | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1288]) 
                                                                           & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1266]))) 
                                                                          << 2U) 
                                                                         | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1288]) 
                                                                              & (~ 
                                                                                ((0U 
                                                                                == 
                                                                                (0xfU 
                                                                                & ((((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[2U] 
                                                                                << 0xcU) 
                                                                                | ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[1U] 
                                                                                >> 0x14U)) 
                                                                                & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1260]))))) 
                                                                                | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 989])))) 
                                                                             << 1U) 
                                                                            | ((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1289]) 
                                                                               & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 989]))))))))))) 
                                            << 0x20U) 
                                           | (QData)((IData)(
                                                             ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1289]) 
                                                                & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1262]))) 
                                                               << 0x1eU) 
                                                              | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1289]) 
                                                                   & (~ 
                                                                      ((4U 
                                                                        >= 
                                                                        (7U 
                                                                         & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[2U] 
                                                                             << 0x12U) 
                                                                            | ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[1U] 
                                                                               >> 0xeU)))) 
                                                                       | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 989])))) 
                                                                  << 0x1dU) 
                                                                 | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1289]) 
                                                                      & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1267]))) 
                                                                     << 0x1cU) 
                                                                    | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1290]) 
                                                                         & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 989]))) 
                                                                        << 0x1bU) 
                                                                       | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1290]) 
                                                                            & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1262]))) 
                                                                           << 0x19U) 
                                                                          | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1290]) 
                                                                               & (~ 
                                                                                ((3U 
                                                                                >= 
                                                                                (7U 
                                                                                & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[2U] 
                                                                                << 0x12U) 
                                                                                | ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[1U] 
                                                                                >> 0xeU)))) 
                                                                                | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 989])))) 
                                                                              << 0x18U) 
                                                                             | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1290]) 
                                                                                & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1267]))) 
                                                                                << 0x17U) 
                                                                                | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1291]) 
                                                                                & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 989]))) 
                                                                                << 0x16U) 
                                                                                | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1291]) 
                                                                                & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1262]))) 
                                                                                << 0x14U) 
                                                                                | (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1291]) 
                                                                                & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1267]))) 
                                                                                << 0x13U))))))))))))) 
                                          >> 0x20U)) 
                                 >> 0x18U));
        (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 3 + BATCH_SIZE * 89)[0U] 
            = __Vtemp4[0U];
        (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 3 + BATCH_SIZE * 89)[1U] 
            = __Vtemp4[1U];
        (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 3 + BATCH_SIZE * 89)[2U] 
            = __Vtemp4[2U];
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1282] 
            = ((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1268]) 
               & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1256])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1283] 
            = ((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1273]) 
               & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1257])));
        _qsignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1] 
            = (((QData)((IData)(((_isignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 95] 
                                  << 4U) | (((0xeU 
                                              == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                             << 3U) 
                                            | (((0xeU 
                                                 == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                << 2U) 
                                               | (((0xdU 
                                                    == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                   << 1U) 
                                                  | (0xdU 
                                                     == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])))))))) 
                << 0x16U) | (QData)((IData)((((0xdU 
                                               == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                              << 0x15U) 
                                             | (((0xdU 
                                                  == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                 << 0x14U) 
                                                | (((0xdU 
                                                     == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                    << 0x13U) 
                                                   | (((0xcU 
                                                        == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                       << 0x12U) 
                                                      | (((0xcU 
                                                           == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                          << 0x11U) 
                                                         | (((0xcU 
                                                              == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                             << 0x10U) 
                                                            | (((0xcU 
                                                                 == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                                << 0xfU) 
                                                               | (((0xcU 
                                                                    == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                                   << 0xeU) 
                                                                  | (((0xbU 
                                                                       == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                                      << 0xdU) 
                                                                     | (((0xbU 
                                                                          == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                                         << 0xcU) 
                                                                        | (((0xbU 
                                                                             == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                                            << 0xbU) 
                                                                           | (((0xbU 
                                                                                == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                                               << 0xaU) 
                                                                              | (((0xbU 
                                                                                == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                                                << 9U) 
                                                                                | (((0xaU 
                                                                                == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                                                << 8U) 
                                                                                | (((0xaU 
                                                                                == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                                                << 7U) 
                                                                                | (((0xaU 
                                                                                == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                                                << 6U) 
                                                                                | (((0xaU 
                                                                                == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                                                << 5U) 
                                                                                | (((0xaU 
                                                                                == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                                                << 4U) 
                                                                                | (((9U 
                                                                                == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                                                << 3U) 
                                                                                | (((9U 
                                                                                == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                                                << 2U) 
                                                                                | (((9U 
                                                                                == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                                                << 1U) 
                                                                                | (9U 
                                                                                == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189]))))))))))))))))))))))))));
        __Vtemp5[0U] = 1U;
        __Vtemp5[1U] = 0U;
        __Vtemp5[2U] = 0U;
        __Vtemp5[3U] = 0U;
        VL_SHIFTL_WWI(128,128,7, __Vtemp6, __Vtemp5, 
                      (0x7fU & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[2U] 
                                 << 1U) | ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[1U] 
                                           >> 0x1fU))));
        if (_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1282]) {
            (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 4 + BATCH_SIZE * 136)[0U] 
                = __Vtemp6[0U];
            (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 4 + BATCH_SIZE * 136)[1U] 
                = __Vtemp6[1U];
            (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 4 + BATCH_SIZE * 136)[2U] 
                = __Vtemp6[2U];
            (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 4 + BATCH_SIZE * 136)[3U] 
                = __Vtemp6[3U];
        } else {
            (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 4 + BATCH_SIZE * 136)[0U] = 0U;
            (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 4 + BATCH_SIZE * 136)[1U] = 0U;
            (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 4 + BATCH_SIZE * 136)[2U] = 0U;
            (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 4 + BATCH_SIZE * 136)[3U] = 0U;
        }
        __Vtemp9[0U] = ((0xffff0000U & ((IData)(((_qsignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1] 
                                                  << 0xaU) 
                                                 | (QData)((IData)(
                                                                   (((9U 
                                                                      == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                                     << 9U) 
                                                                    | (((8U 
                                                                         == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                                        << 8U) 
                                                                       | (((8U 
                                                                            == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                                           << 7U) 
                                                                          | (((8U 
                                                                               == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                                              << 6U) 
                                                                             | (((8U 
                                                                                == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                                                << 5U) 
                                                                                | (((8U 
                                                                                == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                                                << 4U) 
                                                                                | (((7U 
                                                                                == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                                                << 3U) 
                                                                                | (((7U 
                                                                                == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                                                << 2U) 
                                                                                | (((7U 
                                                                                == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                                                << 1U) 
                                                                                | (7U 
                                                                                == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189]))))))))))))))) 
                                        << 0x10U)) 
                        | (((7U == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                            << 0xfU) | (((6U == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                         << 0xeU) | 
                                        (((6U == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                          << 0xdU) 
                                         | (((6U == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                             << 0xcU) 
                                            | (((6U 
                                                 == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                << 0xbU) 
                                               | (((6U 
                                                    == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                   << 0xaU) 
                                                  | (((5U 
                                                       == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                      << 9U) 
                                                     | (((5U 
                                                          == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                         << 8U) 
                                                        | (((5U 
                                                             == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                            << 7U) 
                                                           | (((5U 
                                                                == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                               << 6U) 
                                                              | (((5U 
                                                                   == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                                  << 5U) 
                                                                 | (((4U 
                                                                      == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                                     << 4U) 
                                                                    | (((4U 
                                                                         == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                                        << 3U) 
                                                                       | (((4U 
                                                                            == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                                           << 2U) 
                                                                          | (((4U 
                                                                               == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                                              << 1U) 
                                                                             | (4U 
                                                                                == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189]))))))))))))))))));
        __Vtemp9[1U] = ((0xffffU & ((IData)(((_qsignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1] 
                                              << 0xaU) 
                                             | (QData)((IData)(
                                                               (((9U 
                                                                  == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                                 << 9U) 
                                                                | (((8U 
                                                                     == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                                    << 8U) 
                                                                   | (((8U 
                                                                        == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                                       << 7U) 
                                                                      | (((8U 
                                                                           == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                                          << 6U) 
                                                                         | (((8U 
                                                                              == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                                             << 5U) 
                                                                            | (((8U 
                                                                                == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                                                << 4U) 
                                                                               | (((7U 
                                                                                == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                                                << 3U) 
                                                                                | (((7U 
                                                                                == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                                                << 2U) 
                                                                                | (((7U 
                                                                                == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                                                << 1U) 
                                                                                | (7U 
                                                                                == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189]))))))))))))))) 
                                    >> 0x10U)) | (0xffff0000U 
                                                  & ((IData)(
                                                             (((_qsignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1] 
                                                                << 0xaU) 
                                                               | (QData)((IData)(
                                                                                (((9U 
                                                                                == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                                                << 9U) 
                                                                                | (((8U 
                                                                                == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                                                << 8U) 
                                                                                | (((8U 
                                                                                == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                                                << 7U) 
                                                                                | (((8U 
                                                                                == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                                                << 6U) 
                                                                                | (((8U 
                                                                                == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                                                << 5U) 
                                                                                | (((8U 
                                                                                == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                                                << 4U) 
                                                                                | (((7U 
                                                                                == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                                                << 3U) 
                                                                                | (((7U 
                                                                                == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                                                << 2U) 
                                                                                | (((7U 
                                                                                == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                                                << 1U) 
                                                                                | (7U 
                                                                                == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])))))))))))))) 
                                                              >> 0x20U)) 
                                                     << 0x10U)));
        __Vtemp9[2U] = (0xffffU & ((IData)((((_qsignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1] 
                                              << 0xaU) 
                                             | (QData)((IData)(
                                                               (((9U 
                                                                  == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                                 << 9U) 
                                                                | (((8U 
                                                                     == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                                    << 8U) 
                                                                   | (((8U 
                                                                        == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                                       << 7U) 
                                                                      | (((8U 
                                                                           == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                                          << 6U) 
                                                                         | (((8U 
                                                                              == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                                             << 5U) 
                                                                            | (((8U 
                                                                                == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                                                << 4U) 
                                                                               | (((7U 
                                                                                == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                                                << 3U) 
                                                                                | (((7U 
                                                                                == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                                                << 2U) 
                                                                                | (((7U 
                                                                                == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                                                << 1U) 
                                                                                | (7U 
                                                                                == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])))))))))))))) 
                                            >> 0x20U)) 
                                   >> 0x10U));
        (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 3 + BATCH_SIZE * 96)[0U] 
            = __Vtemp9[0U];
        (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 3 + BATCH_SIZE * 96)[1U] 
            = __Vtemp9[1U];
        (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 3 + BATCH_SIZE * 96)[2U] 
            = __Vtemp9[2U];
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1241] 
            = ((0U == (IData)(_ssignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 7]))
                ? (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1171]) 
                    << 1U) | (1U & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[0U] 
                                    >> 0x1bU))) : (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1171]));
        __Vtemp11[0U] = 1U;
        __Vtemp11[1U] = 0U;
        __Vtemp11[2U] = 0U;
        __Vtemp11[3U] = 0U;
        VL_SHIFTL_WWI(128,128,7, __Vtemp12, __Vtemp11, 
                      (0x7fU & ((IData)(_ssignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 13]) 
                                >> 2U)));
        (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 4 + BATCH_SIZE * 140)[0U] 
            = (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 4 + BATCH_SIZE * 132)[0U] 
                | (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 4 + BATCH_SIZE * 136)[0U]) 
               & (~ ((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1283])
                      ? __Vtemp12[0U] : 0U)));
        (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 4 + BATCH_SIZE * 140)[1U] 
            = (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 4 + BATCH_SIZE * 132)[1U] 
                | (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 4 + BATCH_SIZE * 136)[1U]) 
               & (~ ((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1283])
                      ? __Vtemp12[1U] : 0U)));
        (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 4 + BATCH_SIZE * 140)[2U] 
            = (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 4 + BATCH_SIZE * 132)[2U] 
                | (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 4 + BATCH_SIZE * 136)[2U]) 
               & (~ ((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1283])
                      ? __Vtemp12[2U] : 0U)));
        (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 4 + BATCH_SIZE * 140)[3U] 
            = (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 4 + BATCH_SIZE * 132)[3U] 
                | (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 4 + BATCH_SIZE * 136)[3U]) 
               & (~ ((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1283])
                      ? __Vtemp12[3U] : 0U)));
        VL_SHIFTR_WWI(128,128,7, __Vtemp17, _isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 4 + BATCH_SIZE * 132, 
                      (0x7fU & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[2U] 
                                 << 1U) | ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[1U] 
                                           >> 0x1fU))));
        __Vtemp18[0U] = ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 4 + BATCH_SIZE * 136)[0U] 
                         | (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 4 + BATCH_SIZE * 132)[0U]);
        __Vtemp18[1U] = ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 4 + BATCH_SIZE * 136)[1U] 
                         | (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 4 + BATCH_SIZE * 132)[1U]);
        __Vtemp18[2U] = ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 4 + BATCH_SIZE * 136)[2U] 
                         | (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 4 + BATCH_SIZE * 132)[2U]);
        __Vtemp18[3U] = ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 4 + BATCH_SIZE * 136)[3U] 
                         | (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 4 + BATCH_SIZE * 132)[3U]);
        VL_SHIFTR_WWI(128,128,7, __Vtemp19, __Vtemp18, 
                      (0x7fU & ((IData)(_ssignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 13]) 
                                >> 2U)));
        __Vtemp20[0U] = ((0xfffff800U & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 3 + BATCH_SIZE * 89)[0U] 
                                         << 0xbU)) 
                         | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1271]) 
                              & (~ (((3U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[2U] 
                                             << 0x1aU) 
                                            | ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[1U] 
                                               >> 6U))) 
                                     == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1250])) 
                                    | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 989])))) 
                             << 0xaU) | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1271]) 
                                           & (~ (((0x7fU 
                                                   & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[2U] 
                                                       << 1U) 
                                                      | ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[1U] 
                                                         >> 0x1fU))) 
                                                  == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1251])) 
                                                 | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 989])))) 
                                          << 9U) | 
                                         ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1271]) 
                                            & (~ ((
                                                   (0x1fffffffU 
                                                    & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[4U] 
                                                        << 0x1dU) 
                                                       | ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[3U] 
                                                          >> 3U))) 
                                                   == _isignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 131]) 
                                                  | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 989])))) 
                                           << 8U) | 
                                          ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1276]) 
                                             & (~ (
                                                   ((4U 
                                                     == 
                                                     (7U 
                                                      & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[2U] 
                                                          << 0xfU) 
                                                         | ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[1U] 
                                                            >> 0x11U)))) 
                                                    == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1253])) 
                                                   | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 989])))) 
                                            << 7U) 
                                           | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1276]) 
                                                & (~ 
                                                   (((3U 
                                                      & (IData)(_ssignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 13])) 
                                                     == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1254])) 
                                                    | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 989])))) 
                                               << 5U) 
                                              | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1276]) 
                                                   & (~ 
                                                      (((0x7fU 
                                                         & ((IData)(_ssignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 13]) 
                                                            >> 2U)) 
                                                        == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1255])) 
                                                       | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 989])))) 
                                                  << 4U) 
                                                 | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1282]) 
                                                      & (~ 
                                                         ((~ 
                                                           __Vtemp17[0U]) 
                                                          | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 989])))) 
                                                     << 1U) 
                                                    | ((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1283]) 
                                                       & (~ 
                                                          (__Vtemp19[0U] 
                                                           | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 989]))))))))))));
        (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 3 + BATCH_SIZE * 92)[0U] 
            = __Vtemp20[0U];
        (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 3 + BATCH_SIZE * 92)[1U] 
            = ((0x7ffU & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 3 + BATCH_SIZE * 89)[0U] 
                          >> 0x15U)) | (0xfffff800U 
                                        & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 3 + BATCH_SIZE * 89)[1U] 
                                           << 0xbU)));
        (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 3 + BATCH_SIZE * 92)[2U] 
            = ((0x7ffU & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 3 + BATCH_SIZE * 89)[1U] 
                          >> 0x15U)) | (0xfffff800U 
                                        & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 3 + BATCH_SIZE * 89)[2U] 
                                           << 0xbU)));
        __Vtemp21[0U] = ((0xffff0000U & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 3 + BATCH_SIZE * 96)[0U] 
                                         << 0x10U)) 
                         | (((3U == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                             << 0xfU) | (((3U == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                          << 0xeU) 
                                         | (((3U == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                             << 0xdU) 
                                            | (((3U 
                                                 == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                << 0xcU) 
                                               | (((3U 
                                                    == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                   << 0xbU) 
                                                  | (((2U 
                                                       == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                      << 0xaU) 
                                                     | (((2U 
                                                          == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                         << 9U) 
                                                        | (((2U 
                                                             == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                            << 8U) 
                                                           | (((2U 
                                                                == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                               << 7U) 
                                                              | (((2U 
                                                                   == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                                  << 6U) 
                                                                 | (((1U 
                                                                      == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                                     << 5U) 
                                                                    | (((1U 
                                                                         == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                                        << 4U) 
                                                                       | (((1U 
                                                                            == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                                           << 3U) 
                                                                          | (((1U 
                                                                               == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                                              << 2U) 
                                                                             | (((1U 
                                                                                == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                                                << 1U) 
                                                                                | (0U 
                                                                                == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189]))))))))))))))))));
        __Vtemp22[0U] = ((0xffe00000U & (__Vtemp21[0U] 
                                         << 0x15U)) 
                         | (((0U == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                             << 0x14U) | (((0U == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                           << 0x13U) 
                                          | (((0U == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                              << 0x12U) 
                                             | (((0U 
                                                  == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1189])) 
                                                 << 0x11U) 
                                                | ((0x1f000U 
                                                    & ((- (IData)((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1180]))) 
                                                       << 0xcU)) 
                                                   | ((0xf80U 
                                                       & ((- (IData)((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1190]))) 
                                                          << 7U)) 
                                                      | (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1192]) 
                                                          << 6U) 
                                                         | (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1191]) 
                                                             << 5U) 
                                                            | (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1192]) 
                                                                << 4U) 
                                                               | (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1191]) 
                                                                   << 3U) 
                                                                  | ((6U 
                                                                      & ((- (IData)((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1167]))) 
                                                                         << 1U)) 
                                                                     | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1166])))))))))))));
        (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 4 + BATCH_SIZE * 99)[0U] 
            = __Vtemp22[0U];
        (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 4 + BATCH_SIZE * 99)[1U] 
            = ((0x1fffffU & (__Vtemp21[0U] >> 0xbU)) 
               | (0xffe00000U & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 3 + BATCH_SIZE * 96)[0U] 
                                 << 5U)));
        (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 4 + BATCH_SIZE * 99)[2U] 
            = ((0x1fffffU & ((0x1fU & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 3 + BATCH_SIZE * 96)[0U] 
                                       >> 0x1bU)) | 
                             (0x1fffe0U & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 3 + BATCH_SIZE * 96)[1U] 
                                           << 5U)))) 
               | (0xffe00000U & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 3 + BATCH_SIZE * 96)[1U] 
                                 << 5U)));
        (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 4 + BATCH_SIZE * 99)[3U] 
            = (0x1fffffU & ((0x1fU & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 3 + BATCH_SIZE * 96)[1U] 
                                      >> 0x1bU)) | 
                            (0x1fffe0U & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 3 + BATCH_SIZE * 96)[2U] 
                                          << 5U))));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1293] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 3 + BATCH_SIZE * 92)[2U] 
                         >> 0x12U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1292])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1296] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 3 + BATCH_SIZE * 92)[2U] 
                         >> 0x11U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1295])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1299] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 3 + BATCH_SIZE * 92)[2U] 
                         >> 0xeU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1298])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1302] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 3 + BATCH_SIZE * 92)[2U] 
                         >> 0xdU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1301])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1305] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 3 + BATCH_SIZE * 92)[2U] 
                         >> 0xcU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1304])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1308] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 3 + BATCH_SIZE * 92)[2U] 
                         >> 0xbU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1307])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1311] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 3 + BATCH_SIZE * 92)[2U] 
                         >> 0xaU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1310])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1314] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 3 + BATCH_SIZE * 92)[2U] 
                         >> 7U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1313])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1317] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 3 + BATCH_SIZE * 92)[2U] 
                         >> 6U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1316])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1320] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 3 + BATCH_SIZE * 92)[2U] 
                         >> 5U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1319])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1323] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 3 + BATCH_SIZE * 92)[2U] 
                         >> 4U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1322])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1326] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 3 + BATCH_SIZE * 92)[2U] 
                         >> 3U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1325])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1329] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 3 + BATCH_SIZE * 92)[2U] 
                         >> 2U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1328])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1332] 
            = (1U & ((~ (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 3 + BATCH_SIZE * 92)[2U]) 
                     | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1331])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1335] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 3 + BATCH_SIZE * 92)[1U] 
                         >> 0x1fU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1334])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1338] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 3 + BATCH_SIZE * 92)[1U] 
                         >> 0x1eU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1337])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1341] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 3 + BATCH_SIZE * 92)[1U] 
                         >> 0x1dU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1340])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1344] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 3 + BATCH_SIZE * 92)[1U] 
                         >> 0x1bU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1343])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1347] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 3 + BATCH_SIZE * 92)[1U] 
                         >> 0x1aU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1346])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1350] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 3 + BATCH_SIZE * 92)[1U] 
                         >> 0x19U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1349])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1353] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 3 + BATCH_SIZE * 92)[1U] 
                         >> 0x18U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1352])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1356] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 3 + BATCH_SIZE * 92)[1U] 
                         >> 0x16U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1355])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1359] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 3 + BATCH_SIZE * 92)[1U] 
                         >> 0x15U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1358])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1362] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 3 + BATCH_SIZE * 92)[1U] 
                         >> 0x14U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1361])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1365] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 3 + BATCH_SIZE * 92)[1U] 
                         >> 0x13U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1364])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1368] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 3 + BATCH_SIZE * 92)[1U] 
                         >> 0x11U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1367])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1371] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 3 + BATCH_SIZE * 92)[1U] 
                         >> 0x10U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1370])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1374] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 3 + BATCH_SIZE * 92)[1U] 
                         >> 0xfU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1373])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1377] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 3 + BATCH_SIZE * 92)[1U] 
                         >> 0xeU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1376])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1380] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 3 + BATCH_SIZE * 92)[1U] 
                         >> 0xcU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1379])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1383] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 3 + BATCH_SIZE * 92)[1U] 
                         >> 0xbU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1382])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1386] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 3 + BATCH_SIZE * 92)[1U] 
                         >> 0xaU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1385])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1389] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 3 + BATCH_SIZE * 92)[1U] 
                         >> 9U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1388])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1392] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 3 + BATCH_SIZE * 92)[1U] 
                         >> 7U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1391])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1395] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 3 + BATCH_SIZE * 92)[1U] 
                         >> 6U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1394])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1398] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 3 + BATCH_SIZE * 92)[1U] 
                         >> 5U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1397])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1401] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 3 + BATCH_SIZE * 92)[1U] 
                         >> 2U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1400])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1404] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 3 + BATCH_SIZE * 92)[0U] 
                         >> 0x1eU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1403])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1407] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 3 + BATCH_SIZE * 92)[0U] 
                         >> 0x1aU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1406])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1410] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 3 + BATCH_SIZE * 92)[0U] 
                         >> 0xeU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1409])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1413] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 3 + BATCH_SIZE * 92)[0U] 
                         >> 0xdU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1412])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1416] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 3 + BATCH_SIZE * 92)[0U] 
                         >> 0xcU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1415])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1419] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 3 + BATCH_SIZE * 92)[0U] 
                         >> 0xbU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1418])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1422] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 3 + BATCH_SIZE * 92)[0U] 
                         >> 0xaU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1421])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1425] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 3 + BATCH_SIZE * 92)[0U] 
                         >> 9U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1424])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1428] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 3 + BATCH_SIZE * 92)[0U] 
                         >> 8U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1427])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1431] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 3 + BATCH_SIZE * 92)[0U] 
                         >> 7U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1430])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1434] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 3 + BATCH_SIZE * 92)[0U] 
                         >> 5U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1433])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1437] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 3 + BATCH_SIZE * 92)[0U] 
                         >> 4U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1436])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1440] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 3 + BATCH_SIZE * 92)[0U] 
                         >> 1U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1439])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1443] 
            = (1U & ((~ (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 3 + BATCH_SIZE * 92)[0U]) 
                     | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1442])));
        __Vtemp24[0U] = ((0x80000000U & ((0U == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1194])) 
                                         << 0x1fU)) 
                         | (((0U == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1194])) 
                             << 0x1eU) | ((0x38000000U 
                                           & ((- (IData)((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1188]))) 
                                              << 0x1bU)) 
                                          | (((0U == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1194])) 
                                              << 0x1aU) 
                                             | (((0U 
                                                  == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1194])) 
                                                 << 0x19U) 
                                                | ((0x1c00000U 
                                                    & ((- (IData)((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1188]))) 
                                                       << 0x16U)) 
                                                   | ((0x380000U 
                                                       & ((- (IData)((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1165]))) 
                                                          << 0x13U)) 
                                                      | ((0x7f000U 
                                                          & ((- (IData)((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1188]))) 
                                                             << 0xcU)) 
                                                         | (((5U 
                                                              == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1195])) 
                                                             << 0xbU) 
                                                            | (((5U 
                                                                 == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1195])) 
                                                                << 0xaU) 
                                                               | (((5U 
                                                                    == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1195])) 
                                                                   << 9U) 
                                                                  | (((4U 
                                                                       == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1195])) 
                                                                      << 8U) 
                                                                     | (((4U 
                                                                          == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1195])) 
                                                                         << 7U) 
                                                                        | (((4U 
                                                                             == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1195])) 
                                                                            << 6U) 
                                                                           | (((4U 
                                                                                == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1195])) 
                                                                               << 5U) 
                                                                              | (((4U 
                                                                                == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1195])) 
                                                                                << 4U) 
                                                                                | (((2U 
                                                                                == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1195])) 
                                                                                << 3U) 
                                                                                | (((2U 
                                                                                == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1195])) 
                                                                                << 2U) 
                                                                                | (((2U 
                                                                                == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1195])) 
                                                                                << 1U) 
                                                                                | (2U 
                                                                                == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1195])))))))))))))))))))));
        __Vtemp24[1U] = ((0x7fffffffU & ((0x7ffffc00U 
                                          & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 4 + BATCH_SIZE * 99)[0U] 
                                             << 0xaU)) 
                                         | ((0x7ffffe00U 
                                             & ((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1166]) 
                                                << 9U)) 
                                            | ((0x180U 
                                                & ((- (IData)((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1164]))) 
                                                   << 7U)) 
                                               | ((0x60U 
                                                   & ((- (IData)((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1221]))) 
                                                      << 5U)) 
                                                  | ((0x18U 
                                                      & ((- (IData)((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1166]))) 
                                                         << 3U)) 
                                                     | ((6U 
                                                         & ((- (IData)((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1188]))) 
                                                            << 1U)) 
                                                        | ((0x7fffffffU 
                                                            & (0U 
                                                               == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1194]))) 
                                                           | ((0U 
                                                               == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1194])) 
                                                              >> 1U))))))))) 
                         | (0x80000000U & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 4 + BATCH_SIZE * 99)[0U] 
                                           << 0xaU)));
        (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 5 + BATCH_SIZE * 103)[0U] 
            = __Vtemp24[0U];
        (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 5 + BATCH_SIZE * 103)[1U] 
            = __Vtemp24[1U];
        (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 5 + BATCH_SIZE * 103)[2U] 
            = ((0x7fffffffU & ((0x3ffU & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 4 + BATCH_SIZE * 99)[0U] 
                                          >> 0x16U)) 
                               | (0x7ffffc00U & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 4 + BATCH_SIZE * 99)[1U] 
                                                 << 0xaU)))) 
               | (0x80000000U & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 4 + BATCH_SIZE * 99)[1U] 
                                 << 0xaU)));
        (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 5 + BATCH_SIZE * 103)[3U] 
            = ((0x7fffffffU & ((0x3ffU & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 4 + BATCH_SIZE * 99)[1U] 
                                          >> 0x16U)) 
                               | (0x7ffffc00U & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 4 + BATCH_SIZE * 99)[2U] 
                                                 << 0xaU)))) 
               | (0x80000000U & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 4 + BATCH_SIZE * 99)[2U] 
                                 << 0xaU)));
        (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 5 + BATCH_SIZE * 103)[4U] 
            = (0x7fffffffU & ((0x3ffU & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 4 + BATCH_SIZE * 99)[2U] 
                                         >> 0x16U)) 
                              | (0x7ffffc00U & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 4 + BATCH_SIZE * 99)[3U] 
                                                << 0xaU))));
        __Vtemp26[0U] = ((0xc0000000U & ((0x80000000U 
                                          & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 5 + BATCH_SIZE * 103)[0U] 
                                             << 0x1fU)) 
                                         | ((2U == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1195])) 
                                            << 0x1eU))) 
                         | (((2U == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1195])) 
                             << 0x1dU) | (((3U == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1195])) 
                                           << 0x1cU) 
                                          | (((3U == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1195])) 
                                              << 0x1bU) 
                                             | (((3U 
                                                  == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1195])) 
                                                 << 0x1aU) 
                                                | (((3U 
                                                     == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1195])) 
                                                    << 0x19U) 
                                                   | (((3U 
                                                        == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1195])) 
                                                       << 0x18U) 
                                                      | (((3U 
                                                           == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1195])) 
                                                          << 0x17U) 
                                                         | (((1U 
                                                              == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1195])) 
                                                             << 0x16U) 
                                                            | (((1U 
                                                                 == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1195])) 
                                                                << 0x15U) 
                                                               | (((1U 
                                                                    == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1195])) 
                                                                   << 0x14U) 
                                                                  | (((1U 
                                                                       == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1195])) 
                                                                      << 0x13U) 
                                                                     | (((1U 
                                                                          == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1195])) 
                                                                         << 0x12U) 
                                                                        | (((1U 
                                                                             == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1195])) 
                                                                            << 0x11U) 
                                                                           | (((1U 
                                                                                == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1195])) 
                                                                               << 0x10U) 
                                                                              | (((0U 
                                                                                == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1195])) 
                                                                                << 0xfU) 
                                                                                | (((0U 
                                                                                == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1195])) 
                                                                                << 0xeU) 
                                                                                | (((0U 
                                                                                == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1195])) 
                                                                                << 0xdU) 
                                                                                | (((0U 
                                                                                == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1195])) 
                                                                                << 0xcU) 
                                                                                | (((0U 
                                                                                == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1195])) 
                                                                                << 0xbU) 
                                                                                | (((0U 
                                                                                == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1195])) 
                                                                                << 0xaU) 
                                                                                | (((0U 
                                                                                == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1195])) 
                                                                                << 9U) 
                                                                                | ((0x1fcU 
                                                                                & ((- (IData)((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1190]))) 
                                                                                << 2U)) 
                                                                                | (3U 
                                                                                & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[2U] 
                                                                                << 0xaU) 
                                                                                | ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[1U] 
                                                                                >> 0x16U))))))))))))))))))))))))));
        (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 108)[0U] 
            = __Vtemp26[0U];
        (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 108)[1U] 
            = ((0x3fffffffU & ((0x3fffffffU & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 5 + BATCH_SIZE * 103)[0U] 
                                               >> 1U)) 
                               | ((2U == (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1195])) 
                                  >> 2U))) | (0xc0000000U 
                                              & ((0x40000000U 
                                                  & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 5 + BATCH_SIZE * 103)[0U] 
                                                     >> 1U)) 
                                                 | (0x80000000U 
                                                    & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 5 + BATCH_SIZE * 103)[1U] 
                                                       << 0x1fU)))));
        (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 108)[2U] 
            = ((0x3fffffffU & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 5 + BATCH_SIZE * 103)[1U] 
                               >> 1U)) | (0xc0000000U 
                                          & ((0x40000000U 
                                              & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 5 + BATCH_SIZE * 103)[1U] 
                                                 >> 1U)) 
                                             | (0x80000000U 
                                                & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 5 + BATCH_SIZE * 103)[2U] 
                                                   << 0x1fU)))));
        (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 108)[3U] 
            = ((0x3fffffffU & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 5 + BATCH_SIZE * 103)[2U] 
                               >> 1U)) | (0xc0000000U 
                                          & ((0x40000000U 
                                              & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 5 + BATCH_SIZE * 103)[2U] 
                                                 >> 1U)) 
                                             | (0x80000000U 
                                                & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 5 + BATCH_SIZE * 103)[3U] 
                                                   << 0x1fU)))));
        (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 108)[4U] 
            = ((0x3fffffffU & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 5 + BATCH_SIZE * 103)[3U] 
                               >> 1U)) | (0xc0000000U 
                                          & ((0x40000000U 
                                              & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 5 + BATCH_SIZE * 103)[3U] 
                                                 >> 1U)) 
                                             | (0x80000000U 
                                                & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 5 + BATCH_SIZE * 103)[4U] 
                                                   << 0x1fU)))));
        (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 108)[5U] 
            = (0x3fffffffU & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 5 + BATCH_SIZE * 103)[4U] 
                              >> 1U));
        __Vtemp28[0U] = ((0xf8000000U & ((0xe0000000U 
                                          & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 108)[0U] 
                                             << 0x1dU)) 
                                         | (0x18000000U 
                                            & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[1U] 
                                               << 7U)))) 
                         | (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1229]) 
                             << 0x1aU) | ((0x3c00000U 
                                           & ((- (IData)((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1230]))) 
                                              << 0x16U)) 
                                          | (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1177]) 
                                              << 0x15U) 
                                             | (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1178]) 
                                                 << 0x14U) 
                                                | (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1164]) 
                                                    << 0x13U) 
                                                   | (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1190]) 
                                                       << 0x12U) 
                                                      | ((0x20000U 
                                                          & ((~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1196])) 
                                                             << 0x11U)) 
                                                         | (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1230]) 
                                                             << 0x10U) 
                                                            | (((4U 
                                                                 == 
                                                                 (7U 
                                                                  & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[4U] 
                                                                      << 0x1bU) 
                                                                     | ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[3U] 
                                                                        >> 5U)))) 
                                                                << 0xfU) 
                                                               | (((5U 
                                                                    == 
                                                                    (7U 
                                                                     & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[4U] 
                                                                         << 0x1bU) 
                                                                        | ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[3U] 
                                                                           >> 5U)))) 
                                                                   << 0xeU) 
                                                                  | (((6U 
                                                                       == 
                                                                       (7U 
                                                                        & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[4U] 
                                                                            << 0x1bU) 
                                                                           | ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[3U] 
                                                                              >> 5U)))) 
                                                                      << 0xdU) 
                                                                     | (((7U 
                                                                          == 
                                                                          (7U 
                                                                           & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[4U] 
                                                                               << 0x1bU) 
                                                                              | ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[3U] 
                                                                                >> 5U)))) 
                                                                         << 0xcU) 
                                                                        | (((4U 
                                                                             == 
                                                                             (7U 
                                                                              & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[2U] 
                                                                                << 0xfU) 
                                                                                | ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[1U] 
                                                                                >> 0x11U)))) 
                                                                            << 0xbU) 
                                                                           | (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1232]) 
                                                                               << 0xaU) 
                                                                              | (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1233]) 
                                                                                << 9U) 
                                                                                | (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1235]) 
                                                                                << 8U) 
                                                                                | (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1236]) 
                                                                                << 7U) 
                                                                                | (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1237]) 
                                                                                << 6U) 
                                                                                | (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1229]) 
                                                                                << 5U) 
                                                                                | (((1U 
                                                                                == 
                                                                                (7U 
                                                                                & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[4U] 
                                                                                << 0x1bU) 
                                                                                | ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[3U] 
                                                                                >> 5U)))) 
                                                                                << 4U) 
                                                                                | (((2U 
                                                                                == 
                                                                                (7U 
                                                                                & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[4U] 
                                                                                << 0x1bU) 
                                                                                | ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[3U] 
                                                                                >> 5U)))) 
                                                                                << 3U) 
                                                                                | (((3U 
                                                                                == 
                                                                                (7U 
                                                                                & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[4U] 
                                                                                << 0x1bU) 
                                                                                | ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[3U] 
                                                                                >> 5U)))) 
                                                                                << 2U) 
                                                                                | (((4U 
                                                                                == 
                                                                                (7U 
                                                                                & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[4U] 
                                                                                << 0x1bU) 
                                                                                | ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[3U] 
                                                                                >> 5U)))) 
                                                                                << 1U) 
                                                                                | (5U 
                                                                                == 
                                                                                (7U 
                                                                                & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[4U] 
                                                                                << 0x1bU) 
                                                                                | ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[3U] 
                                                                                >> 5U))))))))))))))))))))))))))));
        (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 7 + BATCH_SIZE * 115)[0U] 
            = __Vtemp28[0U];
        (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 7 + BATCH_SIZE * 115)[1U] 
            = ((0x7ffffffU & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 108)[0U] 
                              >> 3U)) | (0xf8000000U 
                                         & ((0x18000000U 
                                             & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 108)[0U] 
                                                >> 3U)) 
                                            | (0xe0000000U 
                                               & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 108)[1U] 
                                                  << 0x1dU)))));
        (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 7 + BATCH_SIZE * 115)[2U] 
            = ((0x7ffffffU & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 108)[1U] 
                              >> 3U)) | (0xf8000000U 
                                         & ((0x18000000U 
                                             & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 108)[1U] 
                                                >> 3U)) 
                                            | (0xe0000000U 
                                               & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 108)[2U] 
                                                  << 0x1dU)))));
        (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 7 + BATCH_SIZE * 115)[3U] 
            = ((0x7ffffffU & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 108)[2U] 
                              >> 3U)) | (0xf8000000U 
                                         & ((0x18000000U 
                                             & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 108)[2U] 
                                                >> 3U)) 
                                            | (0xe0000000U 
                                               & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 108)[3U] 
                                                  << 0x1dU)))));
        (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 7 + BATCH_SIZE * 115)[4U] 
            = ((0x7ffffffU & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 108)[3U] 
                              >> 3U)) | (0xf8000000U 
                                         & ((0x18000000U 
                                             & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 108)[3U] 
                                                >> 3U)) 
                                            | (0xe0000000U 
                                               & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 108)[4U] 
                                                  << 0x1dU)))));
        (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 7 + BATCH_SIZE * 115)[5U] 
            = ((0x7ffffffU & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 108)[4U] 
                              >> 3U)) | (0xf8000000U 
                                         & ((0x18000000U 
                                             & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 108)[4U] 
                                                >> 3U)) 
                                            | (0xe0000000U 
                                               & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 108)[5U] 
                                                  << 0x1dU)))));
        (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 7 + BATCH_SIZE * 115)[6U] 
            = (0x7ffffffU & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 108)[5U] 
                             >> 3U));
        __Vtemp29[0U] = ((0xffffffe0U & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 7 + BATCH_SIZE * 115)[0U] 
                                         << 5U)) | 
                         (((6U == (7U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[4U] 
                                          << 0x1bU) 
                                         | ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[3U] 
                                            >> 5U)))) 
                           << 4U) | (((7U == (7U & 
                                              (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[4U] 
                                                << 0x1bU) 
                                               | ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[3U] 
                                                  >> 5U)))) 
                                      << 3U) | (((1U 
                                                  == 
                                                  (7U 
                                                   & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[4U] 
                                                       << 0x1bU) 
                                                      | ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[3U] 
                                                         >> 5U)))) 
                                                 << 2U) 
                                                | (((2U 
                                                     == 
                                                     (7U 
                                                      & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[4U] 
                                                          << 0x1bU) 
                                                         | ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[3U] 
                                                            >> 5U)))) 
                                                    << 1U) 
                                                   | (3U 
                                                      == 
                                                      (7U 
                                                       & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[4U] 
                                                           << 0x1bU) 
                                                          | ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[3U] 
                                                             >> 5U)))))))));
        __Vtemp30[0U] = (((4U == (7U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[4U] 
                                         << 0x1bU) 
                                        | ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[3U] 
                                           >> 5U)))) 
                          << 0x1fU) | (((5U == (7U 
                                                & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[4U] 
                                                    << 0x1bU) 
                                                   | ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[3U] 
                                                      >> 5U)))) 
                                        << 0x1eU) | 
                                       (((6U == (7U 
                                                 & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[4U] 
                                                     << 0x1bU) 
                                                    | ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[3U] 
                                                       >> 5U)))) 
                                         << 0x1dU) 
                                        | (((7U == 
                                             (7U & 
                                              (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[4U] 
                                                << 0x1bU) 
                                               | ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[3U] 
                                                  >> 5U)))) 
                                            << 0x1cU) 
                                           | (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1238]) 
                                               << 0x1bU) 
                                              | (0x7ffffffU 
                                                 & (- (IData)((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 989])))))))));
        __Vtemp31[0U] = ((0xf8000000U & (__Vtemp30[0U] 
                                         << 0x1bU)) 
                         | ((0x7fc0000U & ((- (IData)((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 989]))) 
                                           << 0x12U)) 
                            | (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1239]) 
                                << 0x11U) | ((0x1ffe0U 
                                              & ((- (IData)((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 989]))) 
                                                 << 5U)) 
                                             | ((0x10U 
                                                 & ((~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1247])) 
                                                    << 4U)) 
                                                | ((0xfffffff8U 
                                                    & (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1157]) 
                                                        << 3U) 
                                                       & (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[1U])) 
                                                   | ((4U 
                                                       & ((~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1252])) 
                                                          << 2U)) 
                                                      | ((2U 
                                                          & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[0U] 
                                                              >> 0x1eU) 
                                                             & ((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1158]) 
                                                                << 1U))) 
                                                         | (1U 
                                                            & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1256])))))))))));
        (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 9 + BATCH_SIZE * 122)[0U] 
            = __Vtemp31[0U];
        (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 9 + BATCH_SIZE * 122)[1U] 
            = ((0x7ffffffU & (__Vtemp30[0U] >> 5U)) 
               | (0xf8000000U & (__Vtemp29[0U] << 0x1bU)));
        (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 9 + BATCH_SIZE * 122)[2U] 
            = ((0x7ffffffU & (__Vtemp29[0U] >> 5U)) 
               | (0xf8000000U & (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 7 + BATCH_SIZE * 115)[0U]));
        (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 9 + BATCH_SIZE * 122)[3U] 
            = ((0x7ffffffU & (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 7 + BATCH_SIZE * 115)[1U]) 
               | (0xf8000000U & (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 7 + BATCH_SIZE * 115)[1U]));
        (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 9 + BATCH_SIZE * 122)[4U] 
            = ((0x7ffffffU & (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 7 + BATCH_SIZE * 115)[2U]) 
               | (0xf8000000U & (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 7 + BATCH_SIZE * 115)[2U]));
        (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 9 + BATCH_SIZE * 122)[5U] 
            = ((0x7ffffffU & (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 7 + BATCH_SIZE * 115)[3U]) 
               | (0xf8000000U & (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 7 + BATCH_SIZE * 115)[3U]));
        (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 9 + BATCH_SIZE * 122)[6U] 
            = ((0x7ffffffU & (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 7 + BATCH_SIZE * 115)[4U]) 
               | (0xf8000000U & (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 7 + BATCH_SIZE * 115)[4U]));
        (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 9 + BATCH_SIZE * 122)[7U] 
            = ((0x7ffffffU & (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 7 + BATCH_SIZE * 115)[5U]) 
               | (0xf8000000U & (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 7 + BATCH_SIZE * 115)[5U]));
        (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 9 + BATCH_SIZE * 122)[8U] 
            = (0x7ffffffU & (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 7 + BATCH_SIZE * 115)[6U]);
        __Vtemp33[0U] = ((0xffffe000U & ((0xfffc0000U 
                                          & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 9 + BATCH_SIZE * 122)[0U] 
                                             << 0x12U)) 
                                         | ((0xfffe0000U 
                                             & (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1157]) 
                                                 << 0x11U) 
                                                & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[1U] 
                                                   << 0xeU))) 
                                            | ((0x10000U 
                                                & ((~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1257])) 
                                                   << 0x10U)) 
                                               | ((0x8000U 
                                                   & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[0U] 
                                                       >> 0x10U) 
                                                      & ((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1158]) 
                                                         << 0xfU))) 
                                                  | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1268]) 
                                                       & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1256]))) 
                                                      << 0xeU) 
                                                     | ((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1283]) 
                                                        << 0xdU))))))) 
                         | (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 989]) 
                             << 0xcU) | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1268]) 
                                           & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1247]))) 
                                          << 0xbU) 
                                         | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1268]) 
                                              & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1247]))) 
                                             << 0xaU) 
                                            | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1268]) 
                                                 & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1247]))) 
                                                << 9U) 
                                               | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1268]) 
                                                    & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1247]))) 
                                                   << 8U) 
                                                  | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1268]) 
                                                       & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1247]))) 
                                                      << 7U) 
                                                     | (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 989]) 
                                                         << 6U) 
                                                        | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1273]) 
                                                             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1252]))) 
                                                            << 5U) 
                                                           | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1273]) 
                                                                & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1252]))) 
                                                               << 4U) 
                                                              | ((((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1273]) 
                                                                   & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1252]))) 
                                                                  << 3U) 
                                                                 | (7U 
                                                                    & (- (IData)((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 989])))))))))))))));
        __Vtemp33[1U] = ((0x1fffU & ((0x1fffU & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 9 + BATCH_SIZE * 122)[0U] 
                                                 >> 0xeU)) 
                                     | ((0x1fffU & 
                                         (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1157]) 
                                           >> 0xfU) 
                                          & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 6 + BATCH_SIZE * 72)[1U] 
                                             >> 0x12U))) 
                                        | ((0x1fffU 
                                            & (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1268]) 
                                                & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1256]))) 
                                               >> 0x12U)) 
                                           | ((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1283]) 
                                              >> 0x13U))))) 
                         | (0xffffe000U & ((0x3e000U 
                                            & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 9 + BATCH_SIZE * 122)[0U] 
                                               >> 0xeU)) 
                                           | (0xfffc0000U 
                                              & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 9 + BATCH_SIZE * 122)[1U] 
                                                 << 0x12U)))));
        (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[0U] 
            = __Vtemp33[0U];
        (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[1U] 
            = __Vtemp33[1U];
        (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[2U] 
            = ((0x1fffU & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 9 + BATCH_SIZE * 122)[1U] 
                           >> 0xeU)) | (0xffffe000U 
                                        & ((0x3e000U 
                                            & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 9 + BATCH_SIZE * 122)[1U] 
                                               >> 0xeU)) 
                                           | (0xfffc0000U 
                                              & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 9 + BATCH_SIZE * 122)[2U] 
                                                 << 0x12U)))));
        (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[3U] 
            = ((0x1fffU & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 9 + BATCH_SIZE * 122)[2U] 
                           >> 0xeU)) | (0xffffe000U 
                                        & ((0x3e000U 
                                            & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 9 + BATCH_SIZE * 122)[2U] 
                                               >> 0xeU)) 
                                           | (0xfffc0000U 
                                              & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 9 + BATCH_SIZE * 122)[3U] 
                                                 << 0x12U)))));
        (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[4U] 
            = ((0x1fffU & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 9 + BATCH_SIZE * 122)[3U] 
                           >> 0xeU)) | (0xffffe000U 
                                        & ((0x3e000U 
                                            & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 9 + BATCH_SIZE * 122)[3U] 
                                               >> 0xeU)) 
                                           | (0xfffc0000U 
                                              & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 9 + BATCH_SIZE * 122)[4U] 
                                                 << 0x12U)))));
        (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[5U] 
            = ((0x1fffU & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 9 + BATCH_SIZE * 122)[4U] 
                           >> 0xeU)) | (0xffffe000U 
                                        & ((0x3e000U 
                                            & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 9 + BATCH_SIZE * 122)[4U] 
                                               >> 0xeU)) 
                                           | (0xfffc0000U 
                                              & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 9 + BATCH_SIZE * 122)[5U] 
                                                 << 0x12U)))));
        (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[6U] 
            = ((0x1fffU & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 9 + BATCH_SIZE * 122)[5U] 
                           >> 0xeU)) | (0xffffe000U 
                                        & ((0x3e000U 
                                            & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 9 + BATCH_SIZE * 122)[5U] 
                                               >> 0xeU)) 
                                           | (0xfffc0000U 
                                              & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 9 + BATCH_SIZE * 122)[6U] 
                                                 << 0x12U)))));
        (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[7U] 
            = ((0x1fffU & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 9 + BATCH_SIZE * 122)[6U] 
                           >> 0xeU)) | (0xffffe000U 
                                        & ((0x3e000U 
                                            & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 9 + BATCH_SIZE * 122)[6U] 
                                               >> 0xeU)) 
                                           | (0xfffc0000U 
                                              & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 9 + BATCH_SIZE * 122)[7U] 
                                                 << 0x12U)))));
        (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[8U] 
            = ((0x1fffU & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 9 + BATCH_SIZE * 122)[7U] 
                           >> 0xeU)) | (0xffffe000U 
                                        & ((0x3e000U 
                                            & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 9 + BATCH_SIZE * 122)[7U] 
                                               >> 0xeU)) 
                                           | (0xfffc0000U 
                                              & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 9 + BATCH_SIZE * 122)[8U] 
                                                 << 0x12U)))));
        (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[9U] 
            = (0x1fffU & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 9 + BATCH_SIZE * 122)[8U] 
                          >> 0xeU));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1446] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[9U] 
                         >> 0xcU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1445])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1449] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[9U] 
                      >> 0xcU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1448])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1452] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[9U] 
                         >> 0xbU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1451])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1455] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[9U] 
                      >> 0xbU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1454])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1458] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[9U] 
                         >> 8U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1457])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1461] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[9U] 
                      >> 8U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1460])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1464] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[9U] 
                         >> 7U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1463])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1467] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[9U] 
                      >> 7U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1466])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1470] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[9U] 
                         >> 6U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1469])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1473] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[9U] 
                      >> 6U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1472])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1476] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[9U] 
                         >> 5U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1475])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1479] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[9U] 
                      >> 5U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1478])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1482] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[9U] 
                         >> 4U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1481])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1485] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[9U] 
                      >> 4U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1484])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1488] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[9U] 
                         >> 3U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1487])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1491] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[9U] 
                      >> 3U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1490])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1494] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[9U] 
                         >> 2U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1493])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1497] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[9U] 
                      >> 2U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1496])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1500] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[9U] 
                         >> 1U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1499])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1503] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[9U] 
                      >> 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1502])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1506] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[8U] 
                         >> 0x1dU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1505])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1509] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[8U] 
                      >> 0x1dU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1508])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1512] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[8U] 
                         >> 0x18U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1511])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1515] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[8U] 
                      >> 0x18U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1514])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1518] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[8U] 
                         >> 0x13U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1517])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1521] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[8U] 
                      >> 0x13U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1520])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1524] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[8U] 
                         >> 0xeU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1523])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1527] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[8U] 
                      >> 0xeU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1526])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1530] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[8U] 
                         >> 9U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1529])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1533] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[8U] 
                      >> 9U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1532])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1536] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[8U] 
                         >> 4U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1535])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1539] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[8U] 
                      >> 4U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1538])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1542] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[7U] 
                         >> 0x1fU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1541])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1545] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[7U] 
                      >> 0x1fU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1544])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1548] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[7U] 
                         >> 0x1aU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1547])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1551] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[7U] 
                      >> 0x1aU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1550])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1554] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[7U] 
                         >> 0x15U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1553])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1557] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[7U] 
                      >> 0x15U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1556])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1560] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[7U] 
                         >> 0x10U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1559])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1563] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[7U] 
                      >> 0x10U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1562])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1566] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[7U] 
                         >> 0xbU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1565])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1569] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[7U] 
                      >> 0xbU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1568])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1572] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[7U] 
                         >> 6U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1571])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1575] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[7U] 
                      >> 6U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1574])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1578] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[7U] 
                         >> 1U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1577])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1581] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[7U] 
                      >> 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1580])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1584] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[6U] 
                         >> 0x1cU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1583])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1587] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[6U] 
                      >> 0x1cU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1586])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1590] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[6U] 
                         >> 0x17U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1589])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1593] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[6U] 
                      >> 0x17U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1592])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1596] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[6U] 
                         >> 0x12U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1595])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1599] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[6U] 
                      >> 0x12U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1598])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1602] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[6U] 
                         >> 0xdU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1601])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1605] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[6U] 
                      >> 0xdU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1604])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1608] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[6U] 
                         >> 3U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1607])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1611] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[6U] 
                      >> 3U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1610])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1614] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[5U] 
                         >> 0x1eU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1613])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1617] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[5U] 
                      >> 0x1eU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1616])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1620] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[5U] 
                         >> 0x1dU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1619])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1623] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[5U] 
                      >> 0x1dU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1622])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1626] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[5U] 
                         >> 0x1aU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1625])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1629] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[5U] 
                      >> 0x1aU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1628])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1632] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[5U] 
                         >> 0x18U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1631])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1635] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[5U] 
                      >> 0x18U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1634])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1638] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[5U] 
                         >> 0x16U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1637])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1641] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[5U] 
                      >> 0x16U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1640])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1644] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[5U] 
                         >> 0x14U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1643])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1647] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[5U] 
                      >> 0x14U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1646])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1650] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[5U] 
                         >> 0x10U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1649])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1653] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[5U] 
                      >> 0x10U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1652])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1656] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[5U] 
                         >> 0xeU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1655])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1659] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[5U] 
                      >> 0xeU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1658])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1662] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[5U] 
                         >> 3U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1661])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1665] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[5U] 
                      >> 3U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1664])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1668] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[4U] 
                         >> 0x19U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1667])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1671] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[4U] 
                      >> 0x19U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1670])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1674] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[4U] 
                         >> 0x16U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1673])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1677] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[4U] 
                      >> 0x16U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1676])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1680] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[4U] 
                         >> 0x11U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1679])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1683] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[4U] 
                      >> 0x11U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1682])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1686] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[4U] 
                         >> 0xbU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1685])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1689] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[4U] 
                      >> 0xbU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1688])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1692] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[4U] 
                         >> 5U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1691])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1695] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[4U] 
                      >> 5U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1694])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1698] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[3U] 
                         >> 0x1eU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1697])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1701] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[3U] 
                      >> 0x1eU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1700])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1704] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[3U] 
                         >> 0x10U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1703])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1707] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[3U] 
                      >> 0x10U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1706])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1710] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[3U] 
                         >> 0xfU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1709])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1713] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[3U] 
                      >> 0xfU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1712])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1716] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[3U] 
                         >> 0xeU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1715])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1719] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[3U] 
                      >> 0xeU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1718])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1722] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[3U] 
                         >> 0xdU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1721])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1725] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[3U] 
                      >> 0xdU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1724])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1728] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[3U] 
                         >> 0xcU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1727])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1731] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[3U] 
                      >> 0xcU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1730])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1734] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[3U] 
                         >> 0xbU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1733])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1737] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[3U] 
                      >> 0xbU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1736])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1740] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[3U] 
                         >> 7U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1739])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1743] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[3U] 
                      >> 7U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1742])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1746] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[3U] 
                         >> 6U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1745])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1749] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[3U] 
                      >> 6U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1748])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1752] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[3U] 
                         >> 3U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1751])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1755] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[3U] 
                      >> 3U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1754])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1758] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[3U] 
                         >> 1U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1757])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1761] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[3U] 
                      >> 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1760])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1764] 
            = (1U & ((~ (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[3U]) 
                     | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1763])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1767] 
            = (1U & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[3U] 
                     | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1766])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1770] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[2U] 
                         >> 0x1fU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1769])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1773] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[2U] 
                      >> 0x1fU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1772])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1776] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[2U] 
                         >> 0x1eU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1775])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1779] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[2U] 
                      >> 0x1eU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1778])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1782] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[2U] 
                         >> 0x1dU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1781])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1785] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[2U] 
                      >> 0x1dU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1784])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1788] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[2U] 
                         >> 0x1cU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1787])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1791] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[2U] 
                      >> 0x1cU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1790])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1794] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[2U] 
                         >> 0x1bU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1793])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1797] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[2U] 
                      >> 0x1bU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1796])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1800] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[2U] 
                         >> 0x1aU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1799])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1803] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[2U] 
                      >> 0x1aU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1802])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1806] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[2U] 
                         >> 0x19U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1805])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1809] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[2U] 
                      >> 0x19U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1808])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1812] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[2U] 
                         >> 0x18U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1811])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1815] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[2U] 
                      >> 0x18U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1814])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1818] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[2U] 
                         >> 0x16U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1817])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1821] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[2U] 
                      >> 0x16U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1820])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1824] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[2U] 
                         >> 0x15U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1823])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1827] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[2U] 
                      >> 0x15U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1826])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1830] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[2U] 
                         >> 0x14U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1829])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1833] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[2U] 
                      >> 0x14U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1832])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1836] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[2U] 
                         >> 0x13U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1835])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1839] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[2U] 
                      >> 0x13U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1838])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1842] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[2U] 
                         >> 0x12U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1841])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1845] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[2U] 
                      >> 0x12U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1844])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1848] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[2U] 
                         >> 0x11U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1847])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1851] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[2U] 
                      >> 0x11U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1850])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1854] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[2U] 
                         >> 0x10U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1853])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1857] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[2U] 
                      >> 0x10U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1856])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1860] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[2U] 
                         >> 0xfU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1859])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1863] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[2U] 
                      >> 0xfU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1862])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1866] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[2U] 
                         >> 0xeU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1865])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1869] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[2U] 
                      >> 0xeU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1868])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1872] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[2U] 
                         >> 0xdU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1871])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1875] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[2U] 
                      >> 0xdU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1874])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1878] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[2U] 
                         >> 0xcU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1877])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1881] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[2U] 
                      >> 0xcU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1880])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1884] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[2U] 
                         >> 0xbU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1883])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1887] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[2U] 
                      >> 0xbU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1886])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1890] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[2U] 
                         >> 0xaU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1889])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1893] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[2U] 
                      >> 0xaU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1892])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1896] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[2U] 
                         >> 9U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1895])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1899] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[2U] 
                      >> 9U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1898])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1902] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[2U] 
                         >> 8U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1901])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1905] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[2U] 
                      >> 8U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1904])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1908] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[1U] 
                         >> 3U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1907])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1911] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[1U] 
                      >> 3U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1910])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1914] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[0U] 
                         >> 0x16U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1913])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1917] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[0U] 
                      >> 0x16U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1916])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1920] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[0U] 
                         >> 0x15U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1919])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1923] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[0U] 
                      >> 0x15U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1922])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1926] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[0U] 
                         >> 0x14U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1925])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1929] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[0U] 
                      >> 0x14U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1928])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1932] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[0U] 
                         >> 0x13U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1931])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1935] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[0U] 
                      >> 0x13U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1934])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1938] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[0U] 
                         >> 0x12U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1937])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1941] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[0U] 
                      >> 0x12U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1940])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1944] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[0U] 
                         >> 0x10U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1943])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1947] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[0U] 
                      >> 0x10U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1946])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1950] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[0U] 
                         >> 0xeU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1949])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1953] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[0U] 
                      >> 0xeU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1952])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1956] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[0U] 
                         >> 0xdU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1955])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1959] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[0U] 
                      >> 0xdU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1958])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1962] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[0U] 
                         >> 0xbU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1961])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1965] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[0U] 
                      >> 0xbU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1964])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1968] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[0U] 
                         >> 5U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1967])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1971] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 10 + BATCH_SIZE * 78)[0U] 
                      >> 5U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 1970])));
    }

    void Top::_eval_initial(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_eval_initial\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Body
        vlTOPp->_initial__TOP__1(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
#pragma omp parallel for
        for(size_t i = 0; i < BATCH_SIZE; ++i) {
            // Final
            _csignals[i + BATCH_SIZE * 1975] = _csignals[i + BATCH_SIZE * 988];
        }
    }

    void Top::final() {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::final\n"); );
        // Variables
#pragma omp parallel for
        for(size_t i = 0; i < BATCH_SIZE; ++i) {
            Top__Syms* __restrict vlSymsp = this->__VlSymsp;
            Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        }
    }

    __global__
    void _eval_settle(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_eval_settle\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Body
        vlTOPp->_settle__TOP__2(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    void Top::_ctor_var_reset(CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_ctor_var_reset\n"); );
#pragma omp parallel for
        for(size_t i = 0; i < BATCH_SIZE; ++i) {
            // Body
            if (false) {}  // Prevent unused
            _csignals[i + 988 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 989 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 990 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 991 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 992 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 993 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 994 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 995 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 996 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 997 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 998 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 999 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1000 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1001 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1002 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1003 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1004 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1005 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1006 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1007 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1008 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1009 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1010 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1011 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1012 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1013 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1014 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1015 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1016 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1017 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1018 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1019 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1020 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1021 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1022 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1023 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1024 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1025 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1026 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1027 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1028 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1029 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1030 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1031 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1032 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1033 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1034 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1035 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1036 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1037 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1038 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1039 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1040 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1041 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1042 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1043 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1044 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1045 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1046 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1047 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1048 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1049 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1050 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1051 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1052 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1053 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1054 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1055 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1056 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1057 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1058 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1059 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1060 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1061 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1062 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1063 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1064 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1065 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1066 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1067 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1068 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1069 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1070 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1071 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1072 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1073 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1074 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1075 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1076 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1077 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1078 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1079 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1080 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1081 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1082 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1083 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1084 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1085 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1086 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1087 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1088 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1089 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1090 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1091 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1092 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1093 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1094 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1095 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1096 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1097 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1098 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1099 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1100 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1101 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1102 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1103 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1104 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1105 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1106 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1107 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1108 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1109 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1110 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1111 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1112 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1113 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1114 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1115 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1116 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1117 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1118 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1119 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1120 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1121 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1122 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1123 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1124 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1125 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1126 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1127 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1128 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1129 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1130 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1131 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1132 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1133 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1134 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1135 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1136 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1137 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1138 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1139 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1140 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1141 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1142 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1143 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1144 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1145 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1146 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1147 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1148 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1149 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1150 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1151 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1152 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1153 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1154 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1155 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1156 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            VL_RAND_RESET_W(192, _isignals + i * 6 + 72 * BATCH_SIZE);
            VL_RAND_RESET_W(301, _isignals + i * 10 + 78 * BATCH_SIZE);
            _csignals[i + 1157 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1158 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1159 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1160 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1161 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1162 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1163 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 1164 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1165 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1166 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1167 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1168 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1169 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _ssignals[i + 7 * BATCH_SIZE] = VL_RAND_RESET_I(14);
            _csignals[i + 1170 * BATCH_SIZE] = VL_RAND_RESET_I(3);
            _csignals[i + 1171 * BATCH_SIZE] = VL_RAND_RESET_I(3);
            _csignals[i + 1172 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1173 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1174 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1175 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1176 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1177 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1178 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1179 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1180 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _ssignals[i + 8 * BATCH_SIZE] = VL_RAND_RESET_I(16);
            _csignals[i + 1181 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1182 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1183 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1184 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1185 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1186 * BATCH_SIZE] = VL_RAND_RESET_I(4);
            _csignals[i + 1187 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1188 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1189 * BATCH_SIZE] = VL_RAND_RESET_I(5);
            _csignals[i + 1190 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1191 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1192 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1193 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1194 * BATCH_SIZE] = VL_RAND_RESET_I(3);
            _csignals[i + 1195 * BATCH_SIZE] = VL_RAND_RESET_I(3);
            _csignals[i + 1196 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 88 * BATCH_SIZE] = VL_RAND_RESET_I(25);
            VL_RAND_RESET_W(72, _isignals + i * 3 + 89 * BATCH_SIZE);
            VL_RAND_RESET_W(83, _isignals + i * 3 + 92 * BATCH_SIZE);
            _csignals[i + 1197 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1198 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1199 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1200 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1201 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1202 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1203 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1204 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1205 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 95 * BATCH_SIZE] = VL_RAND_RESET_I(28);
            _csignals[i + 1206 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1207 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1208 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1209 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1210 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _qsignals[i + 1 * BATCH_SIZE] = VL_RAND_RESET_Q(54);
            _csignals[i + 1211 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1212 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1213 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1214 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1215 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            VL_RAND_RESET_W(80, _isignals + i * 3 + 96 * BATCH_SIZE);
            _csignals[i + 1216 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1217 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1218 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1219 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            VL_RAND_RESET_W(117, _isignals + i * 4 + 99 * BATCH_SIZE);
            _csignals[i + 1220 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1221 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1222 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1223 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1224 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            VL_RAND_RESET_W(159, _isignals + i * 5 + 103 * BATCH_SIZE);
            _csignals[i + 1225 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1226 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1227 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            VL_RAND_RESET_W(190, _isignals + i * 6 + 108 * BATCH_SIZE);
            _csignals[i + 1228 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 114 * BATCH_SIZE] = VL_RAND_RESET_I(32);
            _csignals[i + 1229 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1230 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1231 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1232 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1233 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1234 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1235 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1236 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1237 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            VL_RAND_RESET_W(219, _isignals + i * 7 + 115 * BATCH_SIZE);
            _csignals[i + 1238 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1239 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            VL_RAND_RESET_W(283, _isignals + i * 9 + 122 * BATCH_SIZE);
            _ssignals[i + 9 * BATCH_SIZE] = VL_RAND_RESET_I(16);
            _ssignals[i + 10 * BATCH_SIZE] = VL_RAND_RESET_I(14);
            _csignals[i + 1240 * BATCH_SIZE] = VL_RAND_RESET_I(4);
            _csignals[i + 1241 * BATCH_SIZE] = VL_RAND_RESET_I(4);
            _csignals[i + 1242 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _ssignals[i + 11 * BATCH_SIZE] = VL_RAND_RESET_I(16);
            _csignals[i + 1243 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1244 * BATCH_SIZE] = VL_RAND_RESET_I(3);
            _ssignals[i + 12 * BATCH_SIZE] = VL_RAND_RESET_I(9);
            _csignals[i + 1245 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _ssignals[i + 13 * BATCH_SIZE] = VL_RAND_RESET_I(9);
            _csignals[i + 1246 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1247 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1248 * BATCH_SIZE] = VL_RAND_RESET_I(3);
            _csignals[i + 1249 * BATCH_SIZE] = VL_RAND_RESET_I(3);
            _csignals[i + 1250 * BATCH_SIZE] = VL_RAND_RESET_I(2);
            _csignals[i + 1251 * BATCH_SIZE] = VL_RAND_RESET_I(7);
            _isignals[i + 131 * BATCH_SIZE] = VL_RAND_RESET_I(29);
            _csignals[i + 1252 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1253 * BATCH_SIZE] = VL_RAND_RESET_I(3);
            _csignals[i + 1254 * BATCH_SIZE] = VL_RAND_RESET_I(2);
            _csignals[i + 1255 * BATCH_SIZE] = VL_RAND_RESET_I(7);
            VL_RAND_RESET_W(128, _isignals + i * 4 + 132 * BATCH_SIZE);
            _csignals[i + 1256 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1257 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1258 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1259 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1260 * BATCH_SIZE] = VL_RAND_RESET_I(4);
            _csignals[i + 1261 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1262 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1263 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1264 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1265 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1266 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1267 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1268 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1269 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1270 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1271 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1272 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1273 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1274 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1275 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1276 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1277 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1278 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1279 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1280 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1281 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1282 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            VL_RAND_RESET_W(128, _isignals + i * 4 + 136 * BATCH_SIZE);
            _csignals[i + 1283 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            VL_RAND_RESET_W(128, _isignals + i * 4 + 140 * BATCH_SIZE);
            _csignals[i + 1284 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1285 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1286 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1287 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1288 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1289 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1290 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1291 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1292 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1293 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1294 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1295 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1296 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1297 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1298 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1299 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1300 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1301 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1302 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1303 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1304 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1305 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1306 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1307 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1308 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1309 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1310 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1311 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1312 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1313 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1314 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1315 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1316 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1317 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1318 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1319 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1320 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1321 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1322 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1323 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1324 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1325 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1326 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1327 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1328 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1329 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1330 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1331 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1332 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1333 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1334 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1335 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1336 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1337 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1338 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1339 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1340 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1341 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1342 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1343 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1344 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1345 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1346 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1347 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1348 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1349 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1350 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1351 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1352 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1353 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1354 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1355 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1356 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1357 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1358 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1359 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1360 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1361 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1362 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1363 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1364 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1365 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1366 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1367 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1368 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1369 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1370 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1371 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1372 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1373 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1374 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1375 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1376 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1377 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1378 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1379 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1380 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1381 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1382 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1383 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1384 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1385 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1386 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1387 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1388 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1389 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1390 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1391 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1392 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1393 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1394 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1395 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1396 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1397 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1398 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1399 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1400 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1401 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1402 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1403 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1404 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1405 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1406 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1407 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1408 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1409 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1410 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1411 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1412 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1413 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1414 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1415 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1416 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1417 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1418 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1419 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1420 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1421 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1422 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1423 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1424 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1425 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1426 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1427 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1428 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1429 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1430 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1431 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1432 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1433 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1434 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1435 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1436 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1437 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1438 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1439 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1440 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1441 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1442 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1443 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1444 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1445 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1446 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1447 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1448 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1449 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1450 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1451 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1452 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1453 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1454 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1455 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1456 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1457 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1458 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1459 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1460 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1461 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1462 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1463 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1464 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1465 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1466 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1467 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1468 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1469 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1470 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1471 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1472 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1473 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1474 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1475 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1476 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1477 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1478 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1479 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1480 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1481 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1482 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1483 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1484 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1485 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1486 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1487 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1488 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1489 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1490 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1491 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1492 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1493 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1494 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1495 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1496 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1497 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1498 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1499 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1500 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1501 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1502 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1503 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1504 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1505 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1506 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1507 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1508 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1509 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1510 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1511 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1512 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1513 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1514 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1515 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1516 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1517 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1518 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1519 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1520 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1521 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1522 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1523 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1524 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1525 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1526 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1527 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1528 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1529 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1530 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1531 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1532 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1533 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1534 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1535 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1536 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1537 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1538 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1539 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1540 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1541 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1542 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1543 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1544 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1545 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1546 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1547 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1548 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1549 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1550 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1551 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1552 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1553 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1554 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1555 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1556 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1557 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1558 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1559 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1560 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1561 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1562 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1563 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1564 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1565 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1566 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1567 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1568 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1569 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1570 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1571 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1572 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1573 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1574 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1575 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1576 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1577 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1578 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1579 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1580 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1581 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1582 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1583 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1584 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1585 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1586 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1587 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1588 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1589 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1590 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1591 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1592 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1593 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1594 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1595 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1596 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1597 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1598 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1599 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1600 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1601 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1602 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1603 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1604 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1605 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1606 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1607 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1608 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1609 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1610 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1611 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1612 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1613 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1614 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1615 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1616 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1617 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1618 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1619 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1620 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1621 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1622 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1623 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1624 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1625 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1626 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1627 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1628 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1629 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1630 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1631 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1632 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1633 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1634 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1635 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1636 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1637 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1638 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1639 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1640 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1641 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1642 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1643 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1644 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1645 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1646 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1647 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1648 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1649 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1650 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1651 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1652 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1653 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1654 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1655 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1656 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1657 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1658 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1659 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1660 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1661 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1662 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1663 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1664 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1665 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1666 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1667 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1668 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1669 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1670 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1671 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1672 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1673 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1674 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1675 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1676 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1677 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1678 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1679 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1680 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1681 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1682 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1683 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1684 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1685 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1686 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1687 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1688 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1689 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1690 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1691 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1692 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1693 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1694 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1695 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1696 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1697 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1698 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1699 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1700 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1701 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1702 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1703 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1704 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1705 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1706 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1707 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1708 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1709 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1710 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1711 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1712 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1713 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1714 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1715 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1716 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1717 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1718 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1719 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1720 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1721 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1722 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1723 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1724 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1725 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1726 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1727 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1728 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1729 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1730 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1731 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1732 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1733 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1734 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1735 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1736 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1737 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1738 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1739 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1740 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1741 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1742 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1743 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1744 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1745 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1746 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1747 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1748 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1749 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1750 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1751 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1752 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1753 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1754 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1755 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1756 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1757 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1758 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1759 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1760 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1761 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1762 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1763 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1764 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1765 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1766 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1767 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1768 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1769 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1770 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1771 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1772 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1773 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1774 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1775 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1776 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1777 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1778 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1779 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1780 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1781 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1782 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1783 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1784 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1785 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1786 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1787 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1788 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1789 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1790 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1791 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1792 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1793 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1794 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1795 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1796 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1797 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1798 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1799 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1800 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1801 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1802 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1803 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1804 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1805 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1806 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1807 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1808 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1809 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1810 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1811 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1812 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1813 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1814 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1815 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1816 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1817 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1818 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1819 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1820 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1821 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1822 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1823 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1824 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1825 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1826 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1827 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1828 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1829 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1830 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1831 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1832 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1833 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1834 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1835 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1836 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1837 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1838 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1839 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1840 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1841 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1842 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1843 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1844 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1845 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1846 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1847 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1848 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1849 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1850 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1851 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1852 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1853 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1854 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1855 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1856 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1857 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1858 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1859 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1860 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1861 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1862 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1863 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1864 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1865 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1866 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1867 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1868 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1869 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1870 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1871 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1872 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1873 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1874 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1875 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1876 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1877 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1878 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1879 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1880 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1881 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1882 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1883 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1884 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1885 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1886 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1887 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1888 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1889 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1890 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1891 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1892 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1893 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1894 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1895 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1896 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1897 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1898 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1899 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1900 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1901 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1902 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1903 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1904 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1905 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1906 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1907 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1908 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1909 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1910 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1911 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1912 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1913 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1914 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1915 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1916 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1917 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1918 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1919 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1920 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1921 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1922 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1923 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1924 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1925 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1926 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1927 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1928 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1929 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1930 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1931 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1932 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1933 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1934 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1935 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1936 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1937 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1938 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1939 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1940 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1941 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1942 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1943 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1944 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1945 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1946 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1947 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1948 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1949 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1950 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1951 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1952 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1953 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1954 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1955 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1956 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1957 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1958 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1959 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1960 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1961 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1962 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1963 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1964 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1965 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1966 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1967 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1968 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1969 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1970 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1971 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1972 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1973 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 1974 * BATCH_SIZE] = VL_RAND_RESET_I(3);
        }
    }
} // end of namespace RF ========================================
