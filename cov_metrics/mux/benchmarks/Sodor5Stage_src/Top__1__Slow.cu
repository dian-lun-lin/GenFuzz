#include "hip/hip_runtime.h"
// Verilated -*- C++ -*-
// DESCRIPTION: Verilator output: Design implementation internals
// See Top.h for the primary calling header

#include "Top.h"
#include "Top__Syms.h"
// begin of namespace RF =====================================
namespace RF {

    //==========

    __device__
    void Top::_settle__TOP__3(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_settle__TOP__3\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Body
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2912] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0x1cU] 
                         >> 0xfU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2911])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2915] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0x1cU] 
                      >> 0xfU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2914])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2918] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0x1cU] 
                         >> 0xeU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2917])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2921] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0x1cU] 
                      >> 0xeU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2920])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2924] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0x1cU] 
                         >> 0xdU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2923])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2927] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0x1cU] 
                      >> 0xdU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2926])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2930] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0x1cU] 
                         >> 0xcU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2929])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2933] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0x1cU] 
                      >> 0xcU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2932])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2936] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0x1cU] 
                         >> 0xbU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2935])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2939] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0x1cU] 
                      >> 0xbU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2938])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2942] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0x1cU] 
                         >> 0xaU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2941])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2945] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0x1cU] 
                      >> 0xaU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2944])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2948] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0x1cU] 
                         >> 9U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2947])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2951] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0x1cU] 
                      >> 9U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2950])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2954] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0x1cU] 
                         >> 8U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2953])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2957] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0x1cU] 
                      >> 8U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2956])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2960] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0x1cU] 
                         >> 7U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2959])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2963] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0x1cU] 
                      >> 7U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2962])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2966] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0x1cU] 
                         >> 6U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2965])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2969] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0x1cU] 
                      >> 6U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2968])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2972] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0x1cU] 
                         >> 5U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2971])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2975] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0x1cU] 
                      >> 5U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2974])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2978] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0x1cU] 
                         >> 4U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2977])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2981] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0x1cU] 
                      >> 4U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2980])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2984] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0x1cU] 
                         >> 3U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2983])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2987] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0x1cU] 
                      >> 3U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2986])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2990] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0x1cU] 
                         >> 2U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2989])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2993] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0x1cU] 
                      >> 2U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2992])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2996] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0x1cU] 
                         >> 1U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2995])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2999] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0x1cU] 
                      >> 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 2998])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3002] 
            = (1U & ((~ (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0x1cU]) 
                     | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3001])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3005] 
            = (1U & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0x1cU] 
                     | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3004])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3008] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xdU] 
                         >> 0xeU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3007])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3011] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xdU] 
                      >> 0xeU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3010])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3014] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xdU] 
                         >> 0xdU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3013])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3017] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xdU] 
                      >> 0xdU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3016])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3020] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xdU] 
                         >> 0xcU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3019])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3023] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xdU] 
                      >> 0xcU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3022])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3026] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xdU] 
                         >> 0xbU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3025])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3029] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xdU] 
                      >> 0xbU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3028])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3032] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xdU] 
                         >> 0xaU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3031])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3035] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xdU] 
                      >> 0xaU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3034])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3038] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xdU] 
                         >> 9U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3037])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3041] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xdU] 
                      >> 9U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3040])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3044] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xdU] 
                         >> 8U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3043])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3047] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xdU] 
                      >> 8U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3046])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3050] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xdU] 
                         >> 7U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3049])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3053] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xdU] 
                      >> 7U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3052])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3056] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xdU] 
                         >> 6U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3055])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3059] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xdU] 
                      >> 6U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3058])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3062] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xdU] 
                         >> 5U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3061])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3065] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xdU] 
                      >> 5U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3064])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3068] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xdU] 
                         >> 4U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3067])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3071] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xdU] 
                      >> 4U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3070])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3074] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xdU] 
                         >> 3U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3073])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3077] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xdU] 
                      >> 3U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3076])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3080] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xdU] 
                         >> 2U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3079])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3083] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xdU] 
                      >> 2U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3082])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3086] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xdU] 
                         >> 1U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3085])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3089] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xdU] 
                      >> 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3088])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3092] 
            = (1U & ((~ (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xdU]) 
                     | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3091])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3095] 
            = (1U & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xdU] 
                     | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3094])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3098] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xcU] 
                         >> 0x1fU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3097])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3101] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xcU] 
                      >> 0x1fU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3100])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3104] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xcU] 
                         >> 0x1eU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3103])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3107] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xcU] 
                      >> 0x1eU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3106])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3110] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xcU] 
                         >> 0x19U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3109])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3113] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xcU] 
                      >> 0x19U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3112])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3116] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xcU] 
                         >> 0x16U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3115])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3119] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xcU] 
                      >> 0x16U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3118])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3122] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xcU] 
                         >> 0x14U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3121])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3125] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xcU] 
                      >> 0x14U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3124])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3128] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xcU] 
                         >> 9U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3127])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3131] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xcU] 
                      >> 9U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3130])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3134] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xcU] 
                         >> 2U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3133])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3137] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xcU] 
                      >> 2U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3136])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3140] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xcU] 
                         >> 1U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3139])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3143] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xcU] 
                      >> 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3142])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3146] 
            = (1U & ((~ (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xcU]) 
                     | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3145])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3149] 
            = (1U & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xcU] 
                     | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3148])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3152] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xbU] 
                         >> 0x1fU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3151])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3155] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xbU] 
                      >> 0x1fU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3154])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3158] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xbU] 
                         >> 0x1eU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3157])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3161] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xbU] 
                      >> 0x1eU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3160])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3164] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xbU] 
                         >> 0x1dU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3163])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3167] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xbU] 
                      >> 0x1dU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3166])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3170] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xbU] 
                         >> 0x1cU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3169])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3173] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xbU] 
                      >> 0x1cU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3172])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3176] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xbU] 
                         >> 0x1aU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3175])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3179] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xbU] 
                      >> 0x1aU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3178])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3182] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xbU] 
                         >> 0x18U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3181])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3185] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xbU] 
                      >> 0x18U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3184])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3188] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xbU] 
                         >> 0x14U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3187])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3191] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xbU] 
                      >> 0x14U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3190])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3194] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xbU] 
                         >> 0x12U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3193])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3197] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xbU] 
                      >> 0x12U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3196])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3200] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xbU] 
                         >> 0x11U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3199])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3203] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xbU] 
                      >> 0x11U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3202])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3206] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xbU] 
                         >> 0x10U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3205])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3209] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xbU] 
                      >> 0x10U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3208])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3212] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xbU] 
                         >> 0xfU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3211])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3215] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xbU] 
                      >> 0xfU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3214])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3218] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xbU] 
                         >> 0xeU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3217])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3221] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xbU] 
                      >> 0xeU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3220])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3224] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xbU] 
                         >> 0xdU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3223])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3227] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xbU] 
                      >> 0xdU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3226])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3230] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xbU] 
                         >> 0xcU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3229])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3233] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xbU] 
                      >> 0xcU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3232])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3236] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xbU] 
                         >> 0xbU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3235])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3239] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xbU] 
                      >> 0xbU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3238])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3242] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xbU] 
                         >> 0xaU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3241])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3245] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xbU] 
                      >> 0xaU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3244])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3248] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xbU] 
                         >> 9U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3247])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3251] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xbU] 
                      >> 9U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3250])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3254] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xbU] 
                         >> 8U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3253])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3257] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xbU] 
                      >> 8U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3256])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3260] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xbU] 
                         >> 7U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3259])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3263] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xbU] 
                      >> 7U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3262])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3266] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xbU] 
                         >> 6U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3265])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3269] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xbU] 
                      >> 6U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3268])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3272] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xbU] 
                         >> 5U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3271])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3275] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xbU] 
                      >> 5U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3274])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3278] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xbU] 
                         >> 4U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3277])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3281] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xbU] 
                      >> 4U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3280])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3284] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xbU] 
                         >> 3U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3283])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3287] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xbU] 
                      >> 3U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3286])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3290] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xbU] 
                         >> 2U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3289])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3293] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xbU] 
                      >> 2U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3292])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3296] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xbU] 
                         >> 1U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3295])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3299] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xbU] 
                      >> 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3298])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3302] 
            = (1U & ((~ (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xbU]) 
                     | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3301])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3305] 
            = (1U & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xbU] 
                     | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3304])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3308] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xaU] 
                         >> 0x1fU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3307])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3311] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xaU] 
                      >> 0x1fU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3310])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3314] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xaU] 
                         >> 0x1eU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3313])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3317] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xaU] 
                      >> 0x1eU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3316])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3320] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xaU] 
                         >> 0x1dU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3319])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3323] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xaU] 
                      >> 0x1dU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3322])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3326] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xaU] 
                         >> 0x1cU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3325])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3329] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xaU] 
                      >> 0x1cU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3328])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3332] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xaU] 
                         >> 0x1bU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3331])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3335] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xaU] 
                      >> 0x1bU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3334])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3338] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xaU] 
                         >> 0x1aU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3337])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3341] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xaU] 
                      >> 0x1aU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3340])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3344] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xaU] 
                         >> 0x19U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3343])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3347] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xaU] 
                      >> 0x19U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3346])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3350] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xaU] 
                         >> 0x18U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3349])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3353] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xaU] 
                      >> 0x18U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3352])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3356] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xaU] 
                         >> 0x17U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3355])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3359] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xaU] 
                      >> 0x17U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3358])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3362] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xaU] 
                         >> 0x16U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3361])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3365] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xaU] 
                      >> 0x16U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3364])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3368] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xaU] 
                         >> 0x15U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3367])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3371] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xaU] 
                      >> 0x15U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3370])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3374] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xaU] 
                         >> 0x14U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3373])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3377] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xaU] 
                      >> 0x14U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3376])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3380] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xaU] 
                         >> 0x13U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3379])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3383] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xaU] 
                      >> 0x13U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3382])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3386] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xaU] 
                         >> 0x12U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3385])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3389] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xaU] 
                      >> 0x12U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3388])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3392] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xaU] 
                         >> 0x11U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3391])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3395] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xaU] 
                      >> 0x11U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3394])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3398] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xaU] 
                         >> 0x10U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3397])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3401] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xaU] 
                      >> 0x10U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3400])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3404] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xaU] 
                         >> 0xfU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3403])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3407] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xaU] 
                      >> 0xfU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3406])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3410] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xaU] 
                         >> 0xeU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3409])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3413] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xaU] 
                      >> 0xeU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3412])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3416] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xaU] 
                         >> 0xdU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3415])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3419] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xaU] 
                      >> 0xdU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3418])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3422] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xaU] 
                         >> 0xcU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3421])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3425] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xaU] 
                      >> 0xcU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3424])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3428] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xaU] 
                         >> 0xbU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3427])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3431] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xaU] 
                      >> 0xbU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3430])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3434] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xaU] 
                         >> 0xaU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3433])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3437] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xaU] 
                      >> 0xaU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3436])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3440] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xaU] 
                         >> 9U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3439])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3443] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xaU] 
                      >> 9U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3442])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3446] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xaU] 
                         >> 8U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3445])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3449] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xaU] 
                      >> 8U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3448])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3452] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xaU] 
                         >> 7U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3451])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3455] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xaU] 
                      >> 7U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3454])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3458] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xaU] 
                         >> 6U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3457])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3461] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xaU] 
                      >> 6U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3460])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3464] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xaU] 
                         >> 5U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3463])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3467] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xaU] 
                      >> 5U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3466])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3470] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xaU] 
                         >> 4U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3469])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3473] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xaU] 
                      >> 4U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3472])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3476] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xaU] 
                         >> 3U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3475])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3479] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xaU] 
                      >> 3U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3478])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3482] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xaU] 
                         >> 2U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3481])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3485] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xaU] 
                      >> 2U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3484])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3488] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xaU] 
                         >> 1U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3487])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3491] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xaU] 
                      >> 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3490])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3494] 
            = (1U & ((~ (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xaU]) 
                     | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3493])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3497] 
            = (1U & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0xaU] 
                     | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3496])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3500] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[9U] 
                         >> 0x1fU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3499])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3503] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[9U] 
                      >> 0x1fU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3502])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3506] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[9U] 
                         >> 0x1eU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3505])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3509] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[9U] 
                      >> 0x1eU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3508])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3512] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[9U] 
                         >> 0x1dU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3511])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3515] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[9U] 
                      >> 0x1dU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3514])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3518] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[9U] 
                         >> 0x1cU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3517])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3521] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[9U] 
                      >> 0x1cU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3520])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3524] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[9U] 
                         >> 0x1bU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3523])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3527] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[9U] 
                      >> 0x1bU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3526])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3530] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[9U] 
                         >> 0x1aU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3529])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3533] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[9U] 
                      >> 0x1aU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3532])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3536] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[9U] 
                         >> 0x19U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3535])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3539] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[9U] 
                      >> 0x19U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3538])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3542] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[9U] 
                         >> 0x18U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3541])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3545] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[9U] 
                      >> 0x18U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3544])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3548] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[9U] 
                         >> 0x17U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3547])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3551] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[9U] 
                      >> 0x17U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3550])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3554] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[9U] 
                         >> 0x16U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3553])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3557] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[9U] 
                      >> 0x16U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3556])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3560] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[9U] 
                         >> 0x15U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3559])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3563] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[9U] 
                      >> 0x15U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3562])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3566] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[9U] 
                         >> 0x14U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3565])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3569] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[9U] 
                      >> 0x14U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3568])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3572] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[9U] 
                         >> 0x13U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3571])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3575] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[9U] 
                      >> 0x13U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3574])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3578] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[9U] 
                         >> 0x12U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3577])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3581] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[9U] 
                      >> 0x12U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3580])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3584] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[9U] 
                         >> 0x11U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3583])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3587] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[9U] 
                      >> 0x11U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3586])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3590] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[9U] 
                         >> 0x10U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3589])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3593] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[9U] 
                      >> 0x10U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3592])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3596] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[9U] 
                         >> 0xfU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3595])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3599] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[9U] 
                      >> 0xfU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3598])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3602] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[9U] 
                         >> 0xeU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3601])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3605] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[9U] 
                      >> 0xeU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3604])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3608] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[9U] 
                         >> 0xdU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3607])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3611] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[9U] 
                      >> 0xdU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3610])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3614] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[9U] 
                         >> 0xcU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3613])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3617] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[9U] 
                      >> 0xcU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3616])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3620] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[9U] 
                         >> 0xbU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3619])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3623] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[9U] 
                      >> 0xbU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3622])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3626] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[9U] 
                         >> 0xaU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3625])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3629] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[9U] 
                      >> 0xaU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3628])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3632] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[9U] 
                         >> 9U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3631])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3635] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[9U] 
                      >> 9U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3634])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3638] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[9U] 
                         >> 8U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3637])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3641] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[9U] 
                      >> 8U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3640])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3644] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[9U] 
                         >> 7U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3643])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3647] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[9U] 
                      >> 7U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3646])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3650] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[9U] 
                         >> 6U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3649])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3653] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[9U] 
                      >> 6U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3652])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3656] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[9U] 
                         >> 5U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3655])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3659] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[9U] 
                      >> 5U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3658])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3662] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[9U] 
                         >> 4U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3661])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3665] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[9U] 
                      >> 4U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3664])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3668] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[9U] 
                         >> 3U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3667])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3671] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[9U] 
                      >> 3U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3670])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3674] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[9U] 
                         >> 2U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3673])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3677] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[9U] 
                      >> 2U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3676])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3680] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[9U] 
                         >> 1U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3679])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3683] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[9U] 
                      >> 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3682])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3686] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[6U] 
                         >> 0x19U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3685])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3689] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[6U] 
                      >> 0x19U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3688])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3692] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[6U] 
                         >> 0x15U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3691])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3695] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[6U] 
                      >> 0x15U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3694])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3698] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[6U] 
                         >> 0xaU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3697])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3701] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[6U] 
                      >> 0xaU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3700])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3704] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[4U] 
                         >> 0xaU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3703])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3707] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[4U] 
                      >> 0xaU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3706])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3710] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[4U] 
                         >> 9U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3709])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3713] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[4U] 
                      >> 9U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3712])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3716] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[4U] 
                         >> 8U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3715])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3719] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[4U] 
                      >> 8U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3718])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3722] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[4U] 
                         >> 7U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3721])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3725] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[4U] 
                      >> 7U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3724])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3728] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[4U] 
                         >> 6U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3727])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3731] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[4U] 
                      >> 6U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3730])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3734] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[4U] 
                         >> 5U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3733])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3737] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[4U] 
                      >> 5U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3736])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3740] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[4U] 
                         >> 4U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3739])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3743] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[4U] 
                      >> 4U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3742])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3746] 
            = (1U & ((~ (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[4U]) 
                     | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3745])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3749] 
            = (1U & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[4U] 
                     | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3748])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3752] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[3U] 
                         >> 0x1fU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3751])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3755] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[3U] 
                      >> 0x1fU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3754])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3758] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[3U] 
                         >> 0x1eU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3757])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3761] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[3U] 
                      >> 0x1eU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3760])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3764] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[3U] 
                         >> 0x1dU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3763])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3767] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[3U] 
                      >> 0x1dU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3766])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3770] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[3U] 
                         >> 0x1cU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3769])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3773] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[3U] 
                      >> 0x1cU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3772])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3776] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[3U] 
                         >> 0x1bU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3775])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3779] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[3U] 
                      >> 0x1bU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3778])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3782] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[3U] 
                         >> 0x1aU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3781])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3785] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[3U] 
                      >> 0x1aU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3784])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3788] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[3U] 
                         >> 0x19U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3787])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3791] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[3U] 
                      >> 0x19U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3790])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3794] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[3U] 
                         >> 0x18U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3793])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3797] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[3U] 
                      >> 0x18U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3796])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3800] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[3U] 
                         >> 0x17U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3799])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3803] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[3U] 
                      >> 0x17U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3802])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3806] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[3U] 
                         >> 0x16U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3805])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3809] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[3U] 
                      >> 0x16U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3808])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3812] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[3U] 
                         >> 0x15U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3811])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3815] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[3U] 
                      >> 0x15U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3814])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3818] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[3U] 
                         >> 0x14U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3817])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3821] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[3U] 
                      >> 0x14U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3820])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3824] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[3U] 
                         >> 0x13U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3823])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3827] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[3U] 
                      >> 0x13U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3826])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3830] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[3U] 
                         >> 0x12U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3829])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3833] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[3U] 
                      >> 0x12U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3832])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3836] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[3U] 
                         >> 0x11U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3835])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3839] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[3U] 
                      >> 0x11U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3838])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3842] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[3U] 
                         >> 0x10U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3841])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3845] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[3U] 
                      >> 0x10U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3844])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3848] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[3U] 
                         >> 0xfU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3847])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3851] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[3U] 
                      >> 0xfU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3850])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3854] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[3U] 
                         >> 0xcU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3853])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3857] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[3U] 
                      >> 0xcU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3856])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3860] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[3U] 
                         >> 0xbU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3859])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3863] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[3U] 
                      >> 0xbU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3862])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3866] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[3U] 
                         >> 0xaU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3865])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3869] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[3U] 
                      >> 0xaU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3868])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3872] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[3U] 
                         >> 9U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3871])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3875] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[3U] 
                      >> 9U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3874])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3878] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[3U] 
                         >> 8U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3877])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3881] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[3U] 
                      >> 8U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3880])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3884] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[3U] 
                         >> 7U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3883])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3887] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[3U] 
                      >> 7U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3886])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3890] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[3U] 
                         >> 6U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3889])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3893] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[3U] 
                      >> 6U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3892])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3896] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[3U] 
                         >> 4U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3895])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3899] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[3U] 
                      >> 4U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3898])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3902] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[3U] 
                         >> 2U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3901])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3905] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[3U] 
                      >> 2U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3904])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3908] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[3U] 
                         >> 1U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3907])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3911] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[3U] 
                      >> 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3910])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3914] 
            = (1U & ((~ (_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[3U]) 
                     | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3913])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3917] 
            = (1U & ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[3U] 
                     | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3916])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3920] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[2U] 
                         >> 0x1fU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3919])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3923] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[2U] 
                      >> 0x1fU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3922])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3926] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[2U] 
                         >> 0x1eU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3925])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3929] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[2U] 
                      >> 0x1eU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3928])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3932] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[2U] 
                         >> 0x1cU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3931])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3935] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[2U] 
                      >> 0x1cU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3934])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3938] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[2U] 
                         >> 0x1bU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3937])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3941] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[2U] 
                      >> 0x1bU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3940])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3944] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[2U] 
                         >> 5U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3943])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3947] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[2U] 
                      >> 5U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3946])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3950] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[1U] 
                         >> 0x1dU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3949])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3953] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[1U] 
                      >> 0x1dU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3952])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3956] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[1U] 
                         >> 0x1cU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3955])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3959] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[1U] 
                      >> 0x1cU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3958])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3962] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[1U] 
                         >> 0xcU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3961])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3965] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[1U] 
                      >> 0xcU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3964])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3968] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0U] 
                         >> 0xbU)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3967])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3971] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0U] 
                      >> 0xbU) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3970])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3974] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0U] 
                         >> 7U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3973])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3977] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0U] 
                      >> 7U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3976])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3980] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0U] 
                         >> 6U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3979])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3983] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0U] 
                      >> 6U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3982])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3986] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0U] 
                         >> 5U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3985])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3989] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0U] 
                      >> 5U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3988])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3992] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0U] 
                         >> 3U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3991])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3995] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0U] 
                      >> 3U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3994])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3998] 
            = (1U & ((~ ((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0U] 
                         >> 1U)) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 3997])));
        _csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 4001] 
            = (1U & (((_isignals + (blockDim.x * blockIdx.x + threadIdx.x) * 30 + BATCH_SIZE * 877)[0U] 
                      >> 1U) | (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 4000])));
    }

    void Top::_eval_initial(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_eval_initial\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Body
        vlTOPp->_initial__TOP__1(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
#pragma omp parallel for
        for(size_t i = 0; i < BATCH_SIZE; ++i) {
            // Final
            _csignals[i + BATCH_SIZE * 4017] = _csignals[i + BATCH_SIZE * 2009];
        }
    }

    void Top::final() {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::final\n"); );
        // Variables
#pragma omp parallel for
        for(size_t i = 0; i < BATCH_SIZE; ++i) {
            Top__Syms* __restrict vlSymsp = this->__VlSymsp;
            Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        }
    }

    __global__
    void _eval_settle(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_eval_settle\n"); );
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        // Body
        vlTOPp->_settle__TOP__2(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        vlTOPp->_settle__TOP__3(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    void Top::_ctor_var_reset(CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals) {
        VL_DEBUG_IF(VL_DBG_MSGF("+    Top::_ctor_var_reset\n"); );
#pragma omp parallel for
        for(size_t i = 0; i < BATCH_SIZE; ++i) {
            // Body
            if (false) {}  // Prevent unused
            _csignals[i + 2009 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2010 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2011 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2012 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2013 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2014 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2015 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2016 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2017 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2018 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2019 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2020 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2021 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2022 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2023 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2024 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2025 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2026 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2027 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2028 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2029 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2030 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2031 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2032 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2033 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2034 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2035 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2036 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2037 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2038 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2039 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2040 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2041 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2042 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2043 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2044 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2045 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2046 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2047 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2048 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2049 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2050 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2051 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2052 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2053 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2054 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2055 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2056 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2057 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2058 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2059 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2060 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2061 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2062 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2063 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2064 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2065 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2066 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2067 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2068 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2069 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2070 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2071 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2072 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2073 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2074 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2075 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2076 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2077 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2078 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2079 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2080 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2081 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2082 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2083 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2084 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2085 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2086 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2087 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2088 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2089 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2090 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2091 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2092 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2093 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2094 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2095 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2096 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2097 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2098 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2099 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2100 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2101 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2102 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2103 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2104 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2105 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2106 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2107 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2108 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2109 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2110 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2111 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2112 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2113 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2114 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2115 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2116 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2117 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2118 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2119 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2120 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2121 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2122 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2123 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2124 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2125 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2126 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2127 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2128 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2129 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2130 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2131 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2132 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2133 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2134 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2135 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2136 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2137 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2138 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2139 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2140 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2141 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2142 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2143 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2144 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2145 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2146 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2147 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2148 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2149 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2150 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2151 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2152 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2153 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2154 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2155 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2156 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2157 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2158 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2159 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2160 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2161 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2162 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2163 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2164 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2165 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2166 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2167 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2168 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2169 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2170 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2171 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2172 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2173 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2174 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2175 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2176 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2177 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2178 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2179 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2180 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2181 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2182 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2183 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2184 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2185 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2186 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2187 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2188 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2189 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2190 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2191 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2192 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2193 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2194 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2195 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2196 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2197 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2198 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2199 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2200 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2201 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2202 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2203 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2204 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2205 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2206 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2207 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2208 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2209 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2210 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2211 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2212 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2213 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2214 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2215 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2216 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2217 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2218 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2219 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2220 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2221 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2222 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2223 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2224 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2225 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2226 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2227 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2228 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2229 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2230 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2231 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2232 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2233 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2234 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2235 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2236 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2237 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2238 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2239 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2240 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2241 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _qsignals[i + 91 * BATCH_SIZE] = VL_RAND_RESET_Q(64);
            VL_RAND_RESET_W(945, _isignals + i * 30 + 877 * BATCH_SIZE);
            _isignals[i + 907 * BATCH_SIZE] = VL_RAND_RESET_I(19);
            _qsignals[i + 92 * BATCH_SIZE] = VL_RAND_RESET_Q(36);
            _qsignals[i + 93 * BATCH_SIZE] = VL_RAND_RESET_Q(53);
            VL_RAND_RESET_W(70, _isignals + i * 3 + 908 * BATCH_SIZE);
            VL_RAND_RESET_W(87, _isignals + i * 3 + 911 * BATCH_SIZE);
            VL_RAND_RESET_W(104, _isignals + i * 4 + 914 * BATCH_SIZE);
            VL_RAND_RESET_W(121, _isignals + i * 4 + 918 * BATCH_SIZE);
            VL_RAND_RESET_W(138, _isignals + i * 5 + 922 * BATCH_SIZE);
            VL_RAND_RESET_W(155, _isignals + i * 5 + 927 * BATCH_SIZE);
            VL_RAND_RESET_W(172, _isignals + i * 6 + 932 * BATCH_SIZE);
            VL_RAND_RESET_W(189, _isignals + i * 6 + 938 * BATCH_SIZE);
            VL_RAND_RESET_W(206, _isignals + i * 7 + 944 * BATCH_SIZE);
            VL_RAND_RESET_W(223, _isignals + i * 7 + 951 * BATCH_SIZE);
            VL_RAND_RESET_W(240, _isignals + i * 8 + 958 * BATCH_SIZE);
            VL_RAND_RESET_W(257, _isignals + i * 9 + 966 * BATCH_SIZE);
            VL_RAND_RESET_W(274, _isignals + i * 9 + 975 * BATCH_SIZE);
            VL_RAND_RESET_W(291, _isignals + i * 10 + 984 * BATCH_SIZE);
            VL_RAND_RESET_W(308, _isignals + i * 10 + 994 * BATCH_SIZE);
            VL_RAND_RESET_W(325, _isignals + i * 11 + 1004 * BATCH_SIZE);
            VL_RAND_RESET_W(342, _isignals + i * 11 + 1015 * BATCH_SIZE);
            VL_RAND_RESET_W(360, _isignals + i * 12 + 1026 * BATCH_SIZE);
            VL_RAND_RESET_W(378, _isignals + i * 12 + 1038 * BATCH_SIZE);
            VL_RAND_RESET_W(395, _isignals + i * 13 + 1050 * BATCH_SIZE);
            VL_RAND_RESET_W(412, _isignals + i * 13 + 1063 * BATCH_SIZE);
            VL_RAND_RESET_W(431, _isignals + i * 14 + 1076 * BATCH_SIZE);
            VL_RAND_RESET_W(448, _isignals + i * 14 + 1090 * BATCH_SIZE);
            VL_RAND_RESET_W(465, _isignals + i * 15 + 1104 * BATCH_SIZE);
            VL_RAND_RESET_W(484, _isignals + i * 16 + 1119 * BATCH_SIZE);
            VL_RAND_RESET_W(501, _isignals + i * 16 + 1135 * BATCH_SIZE);
            VL_RAND_RESET_W(522, _isignals + i * 17 + 1151 * BATCH_SIZE);
            VL_RAND_RESET_W(548, _isignals + i * 18 + 1168 * BATCH_SIZE);
            VL_RAND_RESET_W(576, _isignals + i * 18 + 1186 * BATCH_SIZE);
            VL_RAND_RESET_W(591, _isignals + i * 19 + 1204 * BATCH_SIZE);
            VL_RAND_RESET_W(606, _isignals + i * 19 + 1223 * BATCH_SIZE);
            VL_RAND_RESET_W(621, _isignals + i * 20 + 1242 * BATCH_SIZE);
            VL_RAND_RESET_W(636, _isignals + i * 20 + 1262 * BATCH_SIZE);
            VL_RAND_RESET_W(651, _isignals + i * 21 + 1282 * BATCH_SIZE);
            VL_RAND_RESET_W(666, _isignals + i * 21 + 1303 * BATCH_SIZE);
            VL_RAND_RESET_W(681, _isignals + i * 22 + 1324 * BATCH_SIZE);
            VL_RAND_RESET_W(696, _isignals + i * 22 + 1346 * BATCH_SIZE);
            VL_RAND_RESET_W(711, _isignals + i * 23 + 1368 * BATCH_SIZE);
            VL_RAND_RESET_W(726, _isignals + i * 23 + 1391 * BATCH_SIZE);
            VL_RAND_RESET_W(741, _isignals + i * 24 + 1414 * BATCH_SIZE);
            VL_RAND_RESET_W(790, _isignals + i * 25 + 1438 * BATCH_SIZE);
            VL_RAND_RESET_W(829, _isignals + i * 26 + 1463 * BATCH_SIZE);
            VL_RAND_RESET_W(859, _isignals + i * 27 + 1489 * BATCH_SIZE);
            VL_RAND_RESET_W(903, _isignals + i * 29 + 1516 * BATCH_SIZE);
            _csignals[i + 2242 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2243 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2244 * BATCH_SIZE] = VL_RAND_RESET_I(2);
            _csignals[i + 2245 * BATCH_SIZE] = VL_RAND_RESET_I(3);
            _csignals[i + 2246 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2247 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2248 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2249 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2250 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2251 * BATCH_SIZE] = VL_RAND_RESET_I(5);
            _csignals[i + 2252 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2253 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2254 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2255 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2256 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2257 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2258 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2259 * BATCH_SIZE] = VL_RAND_RESET_I(4);
            _csignals[i + 2260 * BATCH_SIZE] = VL_RAND_RESET_I(4);
            _csignals[i + 2261 * BATCH_SIZE] = VL_RAND_RESET_I(2);
            _csignals[i + 2262 * BATCH_SIZE] = VL_RAND_RESET_I(2);
            _csignals[i + 2263 * BATCH_SIZE] = VL_RAND_RESET_I(3);
            _csignals[i + 2264 * BATCH_SIZE] = VL_RAND_RESET_I(3);
            _csignals[i + 2265 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2266 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2267 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2268 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2269 * BATCH_SIZE] = VL_RAND_RESET_I(4);
            _csignals[i + 2270 * BATCH_SIZE] = VL_RAND_RESET_I(4);
            _csignals[i + 2271 * BATCH_SIZE] = VL_RAND_RESET_I(2);
            _csignals[i + 2272 * BATCH_SIZE] = VL_RAND_RESET_I(2);
            _csignals[i + 2273 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2274 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2275 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2276 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2277 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2278 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2279 * BATCH_SIZE] = VL_RAND_RESET_I(3);
            _csignals[i + 2280 * BATCH_SIZE] = VL_RAND_RESET_I(3);
            _csignals[i + 2281 * BATCH_SIZE] = VL_RAND_RESET_I(3);
            _csignals[i + 2282 * BATCH_SIZE] = VL_RAND_RESET_I(3);
            _csignals[i + 2283 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2284 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2285 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2286 * BATCH_SIZE] = VL_RAND_RESET_I(2);
            _csignals[i + 2287 * BATCH_SIZE] = VL_RAND_RESET_I(2);
            _csignals[i + 2288 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2289 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2290 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2291 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2292 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2293 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1545 * BATCH_SIZE] = VL_RAND_RESET_I(32);
            _isignals[i + 1546 * BATCH_SIZE] = VL_RAND_RESET_I(32);
            _isignals[i + 1547 * BATCH_SIZE] = VL_RAND_RESET_I(32);
            _isignals[i + 1548 * BATCH_SIZE] = VL_RAND_RESET_I(32);
            _isignals[i + 1549 * BATCH_SIZE] = VL_RAND_RESET_I(32);
            _csignals[i + 2294 * BATCH_SIZE] = VL_RAND_RESET_I(5);
            _isignals[i + 1550 * BATCH_SIZE] = VL_RAND_RESET_I(32);
            _isignals[i + 1551 * BATCH_SIZE] = VL_RAND_RESET_I(32);
            _isignals[i + 1552 * BATCH_SIZE] = VL_RAND_RESET_I(32);
            _csignals[i + 2295 * BATCH_SIZE] = VL_RAND_RESET_I(4);
            _csignals[i + 2296 * BATCH_SIZE] = VL_RAND_RESET_I(4);
            _csignals[i + 2297 * BATCH_SIZE] = VL_RAND_RESET_I(2);
            _csignals[i + 2298 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2299 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2300 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2301 * BATCH_SIZE] = VL_RAND_RESET_I(3);
            _csignals[i + 2302 * BATCH_SIZE] = VL_RAND_RESET_I(3);
            _isignals[i + 1553 * BATCH_SIZE] = VL_RAND_RESET_I(32);
            _isignals[i + 1554 * BATCH_SIZE] = VL_RAND_RESET_I(32);
            _isignals[i + 1555 * BATCH_SIZE] = VL_RAND_RESET_I(32);
            _csignals[i + 2303 * BATCH_SIZE] = VL_RAND_RESET_I(5);
            _isignals[i + 1556 * BATCH_SIZE] = VL_RAND_RESET_I(32);
            _csignals[i + 2304 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2305 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2306 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2307 * BATCH_SIZE] = VL_RAND_RESET_I(3);
            _csignals[i + 2308 * BATCH_SIZE] = VL_RAND_RESET_I(2);
            _csignals[i + 2309 * BATCH_SIZE] = VL_RAND_RESET_I(3);
            _csignals[i + 2310 * BATCH_SIZE] = VL_RAND_RESET_I(5);
            _isignals[i + 1557 * BATCH_SIZE] = VL_RAND_RESET_I(32);
            _csignals[i + 2311 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1558 * BATCH_SIZE] = VL_RAND_RESET_I(32);
            _csignals[i + 2312 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2313 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1559 * BATCH_SIZE] = VL_RAND_RESET_I(32);
            _qsignals[i + 94 * BATCH_SIZE] = VL_RAND_RESET_Q(33);
            _ssignals[i + 33 * BATCH_SIZE] = VL_RAND_RESET_I(12);
            _ssignals[i + 34 * BATCH_SIZE] = VL_RAND_RESET_I(12);
            _isignals[i + 1560 * BATCH_SIZE] = VL_RAND_RESET_I(20);
            _csignals[i + 2314 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2315 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2316 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _qsignals[i + 95 * BATCH_SIZE] = VL_RAND_RESET_Q(33);
            _isignals[i + 1561 * BATCH_SIZE] = VL_RAND_RESET_I(32);
            _isignals[i + 1562 * BATCH_SIZE] = VL_RAND_RESET_I(32);
            _isignals[i + 1563 * BATCH_SIZE] = VL_RAND_RESET_I(32);
            _csignals[i + 2317 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2318 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _qsignals[i + 96 * BATCH_SIZE] = VL_RAND_RESET_Q(63);
            _isignals[i + 1564 * BATCH_SIZE] = VL_RAND_RESET_I(32);
            _isignals[i + 1565 * BATCH_SIZE] = VL_RAND_RESET_I(32);
            _csignals[i + 2319 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2320 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2321 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2322 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2323 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2324 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2325 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2326 * BATCH_SIZE] = VL_RAND_RESET_I(2);
            _isignals[i + 1566 * BATCH_SIZE] = VL_RAND_RESET_I(32);
            _csignals[i + 2327 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            for (int __Vi0=0; __Vi0<32; ++__Vi0) {
                _isignals[i * 32 + 1567 * BATCH_SIZE + __Vi0] = VL_RAND_RESET_I(32);
            }
            _csignals[i + 2328 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2329 * BATCH_SIZE] = VL_RAND_RESET_I(5);
            _csignals[i + 2330 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2331 * BATCH_SIZE] = VL_RAND_RESET_I(5);
            _csignals[i + 2332 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2333 * BATCH_SIZE] = VL_RAND_RESET_I(5);
            _csignals[i + 2334 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2335 * BATCH_SIZE] = VL_RAND_RESET_I(5);
            _csignals[i + 2336 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2337 * BATCH_SIZE] = VL_RAND_RESET_I(5);
            _csignals[i + 2338 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2339 * BATCH_SIZE] = VL_RAND_RESET_I(5);
            _csignals[i + 2340 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2341 * BATCH_SIZE] = VL_RAND_RESET_I(5);
            _csignals[i + 2342 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2343 * BATCH_SIZE] = VL_RAND_RESET_I(5);
            _csignals[i + 2344 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2345 * BATCH_SIZE] = VL_RAND_RESET_I(5);
            _csignals[i + 2346 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2347 * BATCH_SIZE] = VL_RAND_RESET_I(5);
            _csignals[i + 2348 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2349 * BATCH_SIZE] = VL_RAND_RESET_I(5);
            _csignals[i + 2350 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2351 * BATCH_SIZE] = VL_RAND_RESET_I(5);
            _csignals[i + 2352 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2353 * BATCH_SIZE] = VL_RAND_RESET_I(5);
            _csignals[i + 2354 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2355 * BATCH_SIZE] = VL_RAND_RESET_I(5);
            _csignals[i + 2356 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2357 * BATCH_SIZE] = VL_RAND_RESET_I(5);
            _csignals[i + 2358 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2359 * BATCH_SIZE] = VL_RAND_RESET_I(5);
            _csignals[i + 2360 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2361 * BATCH_SIZE] = VL_RAND_RESET_I(5);
            _csignals[i + 2362 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2363 * BATCH_SIZE] = VL_RAND_RESET_I(5);
            _csignals[i + 2364 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2365 * BATCH_SIZE] = VL_RAND_RESET_I(5);
            _csignals[i + 2366 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2367 * BATCH_SIZE] = VL_RAND_RESET_I(5);
            _csignals[i + 2368 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2369 * BATCH_SIZE] = VL_RAND_RESET_I(5);
            _csignals[i + 2370 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2371 * BATCH_SIZE] = VL_RAND_RESET_I(5);
            _csignals[i + 2372 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2373 * BATCH_SIZE] = VL_RAND_RESET_I(5);
            _csignals[i + 2374 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2375 * BATCH_SIZE] = VL_RAND_RESET_I(5);
            _csignals[i + 2376 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2377 * BATCH_SIZE] = VL_RAND_RESET_I(5);
            _csignals[i + 2378 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2379 * BATCH_SIZE] = VL_RAND_RESET_I(5);
            _csignals[i + 2380 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2381 * BATCH_SIZE] = VL_RAND_RESET_I(5);
            _csignals[i + 2382 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2383 * BATCH_SIZE] = VL_RAND_RESET_I(5);
            _csignals[i + 2384 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2385 * BATCH_SIZE] = VL_RAND_RESET_I(5);
            _csignals[i + 2386 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2387 * BATCH_SIZE] = VL_RAND_RESET_I(5);
            _csignals[i + 2388 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2389 * BATCH_SIZE] = VL_RAND_RESET_I(5);
            _csignals[i + 2390 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2391 * BATCH_SIZE] = VL_RAND_RESET_I(5);
            _csignals[i + 2392 * BATCH_SIZE] = VL_RAND_RESET_I(6);
            _ssignals[i + 35 * BATCH_SIZE] = VL_RAND_RESET_I(16);
            _ssignals[i + 36 * BATCH_SIZE] = VL_RAND_RESET_I(16);
            _isignals[i + 1599 * BATCH_SIZE] = VL_RAND_RESET_I(32);
            _ssignals[i + 37 * BATCH_SIZE] = VL_RAND_RESET_I(16);
            _csignals[i + 2393 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2394 * BATCH_SIZE] = VL_RAND_RESET_I(4);
            _ssignals[i + 38 * BATCH_SIZE] = VL_RAND_RESET_I(16);
            _ssignals[i + 39 * BATCH_SIZE] = VL_RAND_RESET_I(16);
            _isignals[i + 1600 * BATCH_SIZE] = VL_RAND_RESET_I(32);
            _ssignals[i + 40 * BATCH_SIZE] = VL_RAND_RESET_I(16);
            _csignals[i + 2395 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2396 * BATCH_SIZE] = VL_RAND_RESET_I(4);
            _csignals[i + 2397 * BATCH_SIZE] = VL_RAND_RESET_I(6);
            _isignals[i + 1601 * BATCH_SIZE] = VL_RAND_RESET_I(32);
            _ssignals[i + 41 * BATCH_SIZE] = VL_RAND_RESET_I(16);
            _csignals[i + 2398 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2399 * BATCH_SIZE] = VL_RAND_RESET_I(4);
            _csignals[i + 2400 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2401 * BATCH_SIZE] = VL_RAND_RESET_I(6);
            _csignals[i + 2402 * BATCH_SIZE] = VL_RAND_RESET_I(2);
            _csignals[i + 2403 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2404 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1602 * BATCH_SIZE] = VL_RAND_RESET_I(32);
            _isignals[i + 1603 * BATCH_SIZE] = VL_RAND_RESET_I(32);
            _isignals[i + 1604 * BATCH_SIZE] = VL_RAND_RESET_I(32);
            _isignals[i + 1605 * BATCH_SIZE] = VL_RAND_RESET_I(32);
            _isignals[i + 1606 * BATCH_SIZE] = VL_RAND_RESET_I(32);
            _csignals[i + 2405 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2406 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2407 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2408 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2409 * BATCH_SIZE] = VL_RAND_RESET_I(6);
            _qsignals[i + 97 * BATCH_SIZE] = VL_RAND_RESET_Q(58);
            _csignals[i + 2410 * BATCH_SIZE] = VL_RAND_RESET_I(6);
            _qsignals[i + 98 * BATCH_SIZE] = VL_RAND_RESET_Q(58);
            _qsignals[i + 99 * BATCH_SIZE] = VL_RAND_RESET_Q(40);
            _qsignals[i + 100 * BATCH_SIZE] = VL_RAND_RESET_Q(40);
            _qsignals[i + 101 * BATCH_SIZE] = VL_RAND_RESET_Q(40);
            _qsignals[i + 102 * BATCH_SIZE] = VL_RAND_RESET_Q(40);
            _qsignals[i + 103 * BATCH_SIZE] = VL_RAND_RESET_Q(40);
            _qsignals[i + 104 * BATCH_SIZE] = VL_RAND_RESET_Q(40);
            _qsignals[i + 105 * BATCH_SIZE] = VL_RAND_RESET_Q(40);
            _qsignals[i + 106 * BATCH_SIZE] = VL_RAND_RESET_Q(40);
            _qsignals[i + 107 * BATCH_SIZE] = VL_RAND_RESET_Q(40);
            _qsignals[i + 108 * BATCH_SIZE] = VL_RAND_RESET_Q(40);
            _qsignals[i + 109 * BATCH_SIZE] = VL_RAND_RESET_Q(40);
            _qsignals[i + 110 * BATCH_SIZE] = VL_RAND_RESET_Q(40);
            _qsignals[i + 111 * BATCH_SIZE] = VL_RAND_RESET_Q(40);
            _qsignals[i + 112 * BATCH_SIZE] = VL_RAND_RESET_Q(40);
            _qsignals[i + 113 * BATCH_SIZE] = VL_RAND_RESET_Q(40);
            _qsignals[i + 114 * BATCH_SIZE] = VL_RAND_RESET_Q(40);
            _qsignals[i + 115 * BATCH_SIZE] = VL_RAND_RESET_Q(40);
            _qsignals[i + 116 * BATCH_SIZE] = VL_RAND_RESET_Q(40);
            _qsignals[i + 117 * BATCH_SIZE] = VL_RAND_RESET_Q(40);
            _qsignals[i + 118 * BATCH_SIZE] = VL_RAND_RESET_Q(40);
            _qsignals[i + 119 * BATCH_SIZE] = VL_RAND_RESET_Q(40);
            _qsignals[i + 120 * BATCH_SIZE] = VL_RAND_RESET_Q(40);
            _qsignals[i + 121 * BATCH_SIZE] = VL_RAND_RESET_Q(40);
            _qsignals[i + 122 * BATCH_SIZE] = VL_RAND_RESET_Q(40);
            _qsignals[i + 123 * BATCH_SIZE] = VL_RAND_RESET_Q(40);
            _qsignals[i + 124 * BATCH_SIZE] = VL_RAND_RESET_Q(40);
            _qsignals[i + 125 * BATCH_SIZE] = VL_RAND_RESET_Q(40);
            _qsignals[i + 126 * BATCH_SIZE] = VL_RAND_RESET_Q(40);
            _qsignals[i + 127 * BATCH_SIZE] = VL_RAND_RESET_Q(40);
            _qsignals[i + 128 * BATCH_SIZE] = VL_RAND_RESET_Q(40);
            _qsignals[i + 129 * BATCH_SIZE] = VL_RAND_RESET_Q(40);
            _qsignals[i + 130 * BATCH_SIZE] = VL_RAND_RESET_Q(40);
            _isignals[i + 1607 * BATCH_SIZE] = VL_RAND_RESET_I(32);
            _isignals[i + 1608 * BATCH_SIZE] = VL_RAND_RESET_I(32);
            _csignals[i + 2411 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2412 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _qsignals[i + 131 * BATCH_SIZE] = VL_RAND_RESET_Q(58);
            _qsignals[i + 132 * BATCH_SIZE] = VL_RAND_RESET_Q(64);
            _csignals[i + 2413 * BATCH_SIZE] = VL_RAND_RESET_I(7);
            _qsignals[i + 133 * BATCH_SIZE] = VL_RAND_RESET_Q(58);
            _qsignals[i + 134 * BATCH_SIZE] = VL_RAND_RESET_Q(64);
            _csignals[i + 2414 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2415 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2416 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1609 * BATCH_SIZE] = VL_RAND_RESET_I(32);
            _csignals[i + 2417 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2418 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2419 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1610 * BATCH_SIZE] = VL_RAND_RESET_I(32);
            _csignals[i + 2420 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2421 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _qsignals[i + 135 * BATCH_SIZE] = VL_RAND_RESET_Q(64);
            _qsignals[i + 136 * BATCH_SIZE] = VL_RAND_RESET_Q(64);
            _qsignals[i + 137 * BATCH_SIZE] = VL_RAND_RESET_Q(64);
            _qsignals[i + 138 * BATCH_SIZE] = VL_RAND_RESET_Q(64);
            _qsignals[i + 139 * BATCH_SIZE] = VL_RAND_RESET_Q(64);
            _qsignals[i + 140 * BATCH_SIZE] = VL_RAND_RESET_Q(64);
            _qsignals[i + 141 * BATCH_SIZE] = VL_RAND_RESET_Q(64);
            _qsignals[i + 142 * BATCH_SIZE] = VL_RAND_RESET_Q(64);
            _qsignals[i + 143 * BATCH_SIZE] = VL_RAND_RESET_Q(64);
            _qsignals[i + 144 * BATCH_SIZE] = VL_RAND_RESET_Q(64);
            _qsignals[i + 145 * BATCH_SIZE] = VL_RAND_RESET_Q(64);
            _qsignals[i + 146 * BATCH_SIZE] = VL_RAND_RESET_Q(64);
            _qsignals[i + 147 * BATCH_SIZE] = VL_RAND_RESET_Q(64);
            _qsignals[i + 148 * BATCH_SIZE] = VL_RAND_RESET_Q(41);
            _qsignals[i + 149 * BATCH_SIZE] = VL_RAND_RESET_Q(41);
            _qsignals[i + 150 * BATCH_SIZE] = VL_RAND_RESET_Q(41);
            _qsignals[i + 151 * BATCH_SIZE] = VL_RAND_RESET_Q(41);
            _qsignals[i + 152 * BATCH_SIZE] = VL_RAND_RESET_Q(41);
            _qsignals[i + 153 * BATCH_SIZE] = VL_RAND_RESET_Q(41);
            _qsignals[i + 154 * BATCH_SIZE] = VL_RAND_RESET_Q(41);
            _qsignals[i + 155 * BATCH_SIZE] = VL_RAND_RESET_Q(41);
            _qsignals[i + 156 * BATCH_SIZE] = VL_RAND_RESET_Q(41);
            _qsignals[i + 157 * BATCH_SIZE] = VL_RAND_RESET_Q(41);
            _qsignals[i + 158 * BATCH_SIZE] = VL_RAND_RESET_Q(41);
            _qsignals[i + 159 * BATCH_SIZE] = VL_RAND_RESET_Q(41);
            _qsignals[i + 160 * BATCH_SIZE] = VL_RAND_RESET_Q(41);
            _qsignals[i + 161 * BATCH_SIZE] = VL_RAND_RESET_Q(41);
            _qsignals[i + 162 * BATCH_SIZE] = VL_RAND_RESET_Q(41);
            _qsignals[i + 163 * BATCH_SIZE] = VL_RAND_RESET_Q(41);
            _qsignals[i + 164 * BATCH_SIZE] = VL_RAND_RESET_Q(41);
            _qsignals[i + 165 * BATCH_SIZE] = VL_RAND_RESET_Q(41);
            _qsignals[i + 166 * BATCH_SIZE] = VL_RAND_RESET_Q(41);
            _qsignals[i + 167 * BATCH_SIZE] = VL_RAND_RESET_Q(41);
            _qsignals[i + 168 * BATCH_SIZE] = VL_RAND_RESET_Q(41);
            _qsignals[i + 169 * BATCH_SIZE] = VL_RAND_RESET_Q(41);
            _qsignals[i + 170 * BATCH_SIZE] = VL_RAND_RESET_Q(41);
            _qsignals[i + 171 * BATCH_SIZE] = VL_RAND_RESET_Q(41);
            _qsignals[i + 172 * BATCH_SIZE] = VL_RAND_RESET_Q(41);
            _qsignals[i + 173 * BATCH_SIZE] = VL_RAND_RESET_Q(41);
            _qsignals[i + 174 * BATCH_SIZE] = VL_RAND_RESET_Q(41);
            _qsignals[i + 175 * BATCH_SIZE] = VL_RAND_RESET_Q(41);
            _qsignals[i + 176 * BATCH_SIZE] = VL_RAND_RESET_Q(41);
            _qsignals[i + 177 * BATCH_SIZE] = VL_RAND_RESET_Q(41);
            _qsignals[i + 178 * BATCH_SIZE] = VL_RAND_RESET_Q(41);
            _qsignals[i + 179 * BATCH_SIZE] = VL_RAND_RESET_Q(41);
            _qsignals[i + 180 * BATCH_SIZE] = VL_RAND_RESET_Q(64);
            _isignals[i + 1611 * BATCH_SIZE] = VL_RAND_RESET_I(32);
            _qsignals[i + 181 * BATCH_SIZE] = VL_RAND_RESET_Q(63);
            for (int __Vi0=0; __Vi0<32; ++__Vi0) {
                _csignals[i * 32 + 2422 * BATCH_SIZE + __Vi0] = VL_RAND_RESET_I(8);
            }
            _csignals[i + 2454 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1612 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2455 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1613 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2456 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1614 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2457 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1615 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2458 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1616 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2459 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1617 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2460 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1618 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2461 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1619 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2462 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1620 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2463 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1621 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2464 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1622 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2465 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1623 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2466 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1624 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2467 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1625 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2468 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1626 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2469 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1627 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2470 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1628 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2471 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1629 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2472 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1630 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2473 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1631 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2474 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1632 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2475 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1633 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2476 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1634 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2477 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1635 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2478 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1636 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2479 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1637 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2480 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1638 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2481 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1639 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2482 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1640 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2483 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1641 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2484 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1642 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2485 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1643 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2486 * BATCH_SIZE] = VL_RAND_RESET_I(6);
            _ssignals[i + 42 * BATCH_SIZE] = VL_RAND_RESET_I(16);
            _ssignals[i + 43 * BATCH_SIZE] = VL_RAND_RESET_I(16);
            _isignals[i + 1644 * BATCH_SIZE] = VL_RAND_RESET_I(32);
            _ssignals[i + 44 * BATCH_SIZE] = VL_RAND_RESET_I(16);
            _csignals[i + 2487 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2488 * BATCH_SIZE] = VL_RAND_RESET_I(4);
            _csignals[i + 2489 * BATCH_SIZE] = VL_RAND_RESET_I(6);
            _ssignals[i + 45 * BATCH_SIZE] = VL_RAND_RESET_I(16);
            _ssignals[i + 46 * BATCH_SIZE] = VL_RAND_RESET_I(16);
            _isignals[i + 1645 * BATCH_SIZE] = VL_RAND_RESET_I(32);
            _ssignals[i + 47 * BATCH_SIZE] = VL_RAND_RESET_I(16);
            _csignals[i + 2490 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2491 * BATCH_SIZE] = VL_RAND_RESET_I(4);
            _csignals[i + 2492 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2493 * BATCH_SIZE] = VL_RAND_RESET_I(6);
            for (int __Vi0=0; __Vi0<32; ++__Vi0) {
                _csignals[i * 32 + 2494 * BATCH_SIZE + __Vi0] = VL_RAND_RESET_I(8);
            }
            _csignals[i + 2526 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1646 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2527 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1647 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2528 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1648 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2529 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1649 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2530 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1650 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2531 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1651 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2532 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1652 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2533 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1653 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2534 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1654 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2535 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1655 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2536 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1656 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2537 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1657 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2538 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1658 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2539 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1659 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2540 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1660 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2541 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1661 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2542 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1662 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2543 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1663 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2544 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1664 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2545 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1665 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2546 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1666 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2547 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1667 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2548 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1668 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2549 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1669 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2550 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1670 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2551 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1671 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2552 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1672 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2553 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1673 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2554 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1674 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2555 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1675 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2556 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1676 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2557 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1677 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2558 * BATCH_SIZE] = VL_RAND_RESET_I(6);
            _ssignals[i + 48 * BATCH_SIZE] = VL_RAND_RESET_I(16);
            _ssignals[i + 49 * BATCH_SIZE] = VL_RAND_RESET_I(16);
            _isignals[i + 1678 * BATCH_SIZE] = VL_RAND_RESET_I(32);
            _ssignals[i + 50 * BATCH_SIZE] = VL_RAND_RESET_I(16);
            _csignals[i + 2559 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2560 * BATCH_SIZE] = VL_RAND_RESET_I(4);
            _csignals[i + 2561 * BATCH_SIZE] = VL_RAND_RESET_I(6);
            _ssignals[i + 51 * BATCH_SIZE] = VL_RAND_RESET_I(16);
            _ssignals[i + 52 * BATCH_SIZE] = VL_RAND_RESET_I(16);
            _isignals[i + 1679 * BATCH_SIZE] = VL_RAND_RESET_I(32);
            _ssignals[i + 53 * BATCH_SIZE] = VL_RAND_RESET_I(16);
            _csignals[i + 2562 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2563 * BATCH_SIZE] = VL_RAND_RESET_I(4);
            _csignals[i + 2564 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2565 * BATCH_SIZE] = VL_RAND_RESET_I(6);
            for (int __Vi0=0; __Vi0<32; ++__Vi0) {
                _csignals[i * 32 + 2566 * BATCH_SIZE + __Vi0] = VL_RAND_RESET_I(8);
            }
            _csignals[i + 2598 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1680 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2599 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1681 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2600 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1682 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2601 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1683 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2602 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1684 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2603 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1685 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2604 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1686 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2605 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1687 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2606 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1688 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2607 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1689 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2608 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1690 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2609 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1691 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2610 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1692 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2611 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1693 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2612 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1694 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2613 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1695 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2614 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1696 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2615 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1697 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2616 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1698 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2617 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1699 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2618 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1700 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2619 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1701 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2620 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1702 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2621 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1703 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2622 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1704 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2623 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1705 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2624 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1706 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2625 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1707 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2626 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1708 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2627 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1709 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2628 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1710 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2629 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1711 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2630 * BATCH_SIZE] = VL_RAND_RESET_I(6);
            _ssignals[i + 54 * BATCH_SIZE] = VL_RAND_RESET_I(16);
            _ssignals[i + 55 * BATCH_SIZE] = VL_RAND_RESET_I(16);
            _isignals[i + 1712 * BATCH_SIZE] = VL_RAND_RESET_I(32);
            _ssignals[i + 56 * BATCH_SIZE] = VL_RAND_RESET_I(16);
            _csignals[i + 2631 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2632 * BATCH_SIZE] = VL_RAND_RESET_I(4);
            _csignals[i + 2633 * BATCH_SIZE] = VL_RAND_RESET_I(6);
            _ssignals[i + 57 * BATCH_SIZE] = VL_RAND_RESET_I(16);
            _ssignals[i + 58 * BATCH_SIZE] = VL_RAND_RESET_I(16);
            _isignals[i + 1713 * BATCH_SIZE] = VL_RAND_RESET_I(32);
            _ssignals[i + 59 * BATCH_SIZE] = VL_RAND_RESET_I(16);
            _csignals[i + 2634 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2635 * BATCH_SIZE] = VL_RAND_RESET_I(4);
            _csignals[i + 2636 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2637 * BATCH_SIZE] = VL_RAND_RESET_I(6);
            for (int __Vi0=0; __Vi0<32; ++__Vi0) {
                _csignals[i * 32 + 2638 * BATCH_SIZE + __Vi0] = VL_RAND_RESET_I(8);
            }
            _csignals[i + 2670 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1714 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2671 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1715 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2672 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1716 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2673 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1717 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2674 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1718 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2675 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1719 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2676 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1720 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2677 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1721 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2678 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1722 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2679 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1723 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2680 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1724 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2681 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1725 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2682 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1726 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2683 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1727 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2684 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1728 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2685 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1729 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2686 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1730 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2687 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1731 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2688 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1732 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2689 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1733 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2690 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1734 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2691 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1735 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2692 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1736 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2693 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1737 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2694 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1738 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2695 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1739 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2696 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1740 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2697 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1741 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2698 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1742 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2699 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1743 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2700 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1744 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2701 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1745 * BATCH_SIZE] = VL_RAND_RESET_I(21);
            _csignals[i + 2702 * BATCH_SIZE] = VL_RAND_RESET_I(6);
            _ssignals[i + 60 * BATCH_SIZE] = VL_RAND_RESET_I(16);
            _ssignals[i + 61 * BATCH_SIZE] = VL_RAND_RESET_I(16);
            _isignals[i + 1746 * BATCH_SIZE] = VL_RAND_RESET_I(32);
            _ssignals[i + 62 * BATCH_SIZE] = VL_RAND_RESET_I(16);
            _csignals[i + 2703 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2704 * BATCH_SIZE] = VL_RAND_RESET_I(4);
            _csignals[i + 2705 * BATCH_SIZE] = VL_RAND_RESET_I(6);
            _ssignals[i + 63 * BATCH_SIZE] = VL_RAND_RESET_I(16);
            _ssignals[i + 64 * BATCH_SIZE] = VL_RAND_RESET_I(16);
            _isignals[i + 1747 * BATCH_SIZE] = VL_RAND_RESET_I(32);
            _ssignals[i + 65 * BATCH_SIZE] = VL_RAND_RESET_I(16);
            _csignals[i + 2706 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 2707 * BATCH_SIZE] = VL_RAND_RESET_I(4);
            _csignals[i + 2708 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2709 * BATCH_SIZE] = VL_RAND_RESET_I(6);
            _csignals[i + 2710 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2711 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2712 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2713 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2714 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2715 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2716 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2717 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2718 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2719 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2720 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2721 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2722 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2723 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2724 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2725 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2726 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2727 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2728 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2729 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2730 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2731 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2732 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2733 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2734 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2735 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2736 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2737 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2738 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2739 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2740 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2741 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2742 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2743 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2744 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2745 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2746 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2747 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2748 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2749 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2750 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2751 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2752 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2753 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2754 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2755 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2756 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2757 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2758 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2759 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2760 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2761 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2762 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2763 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2764 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2765 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2766 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2767 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2768 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2769 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2770 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2771 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2772 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2773 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2774 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2775 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2776 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2777 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2778 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2779 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2780 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2781 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2782 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2783 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2784 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2785 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2786 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2787 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2788 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2789 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2790 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2791 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2792 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2793 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2794 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2795 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2796 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2797 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2798 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2799 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2800 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2801 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2802 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2803 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2804 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2805 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2806 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2807 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2808 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2809 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2810 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2811 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2812 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2813 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2814 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2815 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2816 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2817 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2818 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2819 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2820 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2821 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2822 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2823 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2824 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2825 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2826 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2827 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2828 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2829 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2830 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2831 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2832 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2833 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2834 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2835 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2836 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2837 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2838 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2839 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2840 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2841 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2842 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2843 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2844 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2845 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2846 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2847 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2848 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2849 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2850 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2851 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2852 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2853 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2854 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2855 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2856 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2857 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2858 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2859 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2860 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2861 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2862 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2863 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2864 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2865 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2866 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2867 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2868 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2869 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2870 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2871 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2872 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2873 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2874 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2875 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2876 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2877 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2878 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2879 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2880 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2881 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2882 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2883 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2884 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2885 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2886 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2887 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2888 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2889 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2890 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2891 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2892 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2893 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2894 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2895 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2896 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2897 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2898 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2899 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2900 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2901 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2902 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2903 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2904 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2905 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2906 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2907 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2908 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2909 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2910 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2911 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2912 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2913 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2914 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2915 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2916 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2917 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2918 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2919 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2920 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2921 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2922 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2923 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2924 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2925 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2926 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2927 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2928 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2929 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2930 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2931 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2932 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2933 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2934 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2935 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2936 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2937 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2938 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2939 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2940 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2941 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2942 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2943 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2944 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2945 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2946 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2947 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2948 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2949 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2950 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2951 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2952 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2953 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2954 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2955 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2956 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2957 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2958 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2959 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2960 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2961 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2962 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2963 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2964 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2965 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2966 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2967 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2968 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2969 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2970 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2971 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2972 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2973 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2974 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2975 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2976 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2977 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2978 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2979 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2980 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2981 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2982 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2983 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2984 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2985 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2986 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2987 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2988 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2989 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2990 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2991 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2992 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2993 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2994 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2995 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2996 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2997 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2998 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 2999 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3000 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3001 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3002 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3003 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3004 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3005 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3006 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3007 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3008 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3009 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3010 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3011 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3012 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3013 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3014 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3015 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3016 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3017 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3018 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3019 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3020 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3021 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3022 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3023 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3024 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3025 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3026 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3027 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3028 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3029 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3030 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3031 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3032 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3033 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3034 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3035 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3036 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3037 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3038 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3039 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3040 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3041 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3042 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3043 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3044 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3045 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3046 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3047 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3048 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3049 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3050 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3051 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3052 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3053 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3054 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3055 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3056 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3057 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3058 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3059 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3060 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3061 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3062 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3063 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3064 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3065 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3066 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3067 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3068 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3069 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3070 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3071 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3072 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3073 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3074 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3075 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3076 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3077 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3078 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3079 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3080 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3081 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3082 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3083 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3084 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3085 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3086 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3087 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3088 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3089 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3090 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3091 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3092 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3093 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3094 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3095 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3096 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3097 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3098 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3099 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3100 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3101 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3102 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3103 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3104 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3105 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3106 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3107 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3108 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3109 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3110 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3111 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3112 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3113 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3114 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3115 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3116 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3117 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3118 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3119 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3120 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3121 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3122 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3123 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3124 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3125 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3126 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3127 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3128 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3129 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3130 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3131 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3132 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3133 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3134 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3135 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3136 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3137 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3138 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3139 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3140 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3141 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3142 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3143 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3144 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3145 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3146 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3147 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3148 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3149 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3150 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3151 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3152 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3153 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3154 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3155 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3156 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3157 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3158 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3159 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3160 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3161 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3162 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3163 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3164 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3165 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3166 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3167 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3168 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3169 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3170 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3171 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3172 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3173 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3174 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3175 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3176 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3177 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3178 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3179 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3180 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3181 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3182 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3183 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3184 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3185 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3186 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3187 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3188 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3189 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3190 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3191 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3192 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3193 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3194 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3195 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3196 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3197 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3198 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3199 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3200 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3201 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3202 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3203 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3204 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3205 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3206 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3207 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3208 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3209 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3210 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3211 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3212 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3213 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3214 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3215 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3216 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3217 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3218 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3219 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3220 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3221 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3222 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3223 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3224 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3225 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3226 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3227 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3228 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3229 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3230 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3231 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3232 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3233 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3234 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3235 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3236 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3237 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3238 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3239 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3240 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3241 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3242 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3243 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3244 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3245 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3246 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3247 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3248 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3249 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3250 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3251 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3252 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3253 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3254 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3255 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3256 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3257 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3258 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3259 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3260 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3261 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3262 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3263 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3264 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3265 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3266 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3267 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3268 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3269 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3270 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3271 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3272 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3273 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3274 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3275 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3276 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3277 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3278 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3279 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3280 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3281 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3282 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3283 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3284 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3285 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3286 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3287 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3288 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3289 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3290 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3291 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3292 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3293 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3294 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3295 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3296 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3297 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3298 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3299 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3300 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3301 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3302 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3303 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3304 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3305 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3306 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3307 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3308 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3309 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3310 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3311 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3312 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3313 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3314 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3315 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3316 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3317 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3318 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3319 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3320 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3321 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3322 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3323 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3324 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3325 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3326 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3327 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3328 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3329 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3330 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3331 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3332 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3333 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3334 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3335 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3336 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3337 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3338 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3339 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3340 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3341 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3342 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3343 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3344 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3345 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3346 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3347 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3348 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3349 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3350 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3351 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3352 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3353 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3354 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3355 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3356 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3357 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3358 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3359 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3360 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3361 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3362 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3363 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3364 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3365 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3366 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3367 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3368 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3369 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3370 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3371 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3372 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3373 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3374 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3375 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3376 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3377 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3378 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3379 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3380 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3381 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3382 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3383 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3384 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3385 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3386 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3387 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3388 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3389 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3390 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3391 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3392 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3393 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3394 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3395 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3396 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3397 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3398 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3399 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3400 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3401 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3402 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3403 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3404 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3405 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3406 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3407 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3408 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3409 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3410 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3411 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3412 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3413 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3414 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3415 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3416 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3417 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3418 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3419 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3420 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3421 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3422 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3423 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3424 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3425 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3426 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3427 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3428 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3429 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3430 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3431 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3432 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3433 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3434 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3435 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3436 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3437 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3438 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3439 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3440 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3441 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3442 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3443 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3444 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3445 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3446 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3447 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3448 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3449 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3450 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3451 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3452 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3453 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3454 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3455 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3456 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3457 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3458 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3459 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3460 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3461 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3462 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3463 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3464 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3465 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3466 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3467 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3468 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3469 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3470 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3471 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3472 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3473 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3474 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3475 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3476 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3477 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3478 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3479 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3480 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3481 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3482 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3483 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3484 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3485 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3486 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3487 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3488 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3489 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3490 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3491 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3492 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3493 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3494 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3495 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3496 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3497 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3498 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3499 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3500 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3501 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3502 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3503 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3504 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3505 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3506 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3507 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3508 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3509 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3510 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3511 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3512 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3513 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3514 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3515 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3516 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3517 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3518 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3519 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3520 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3521 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3522 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3523 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3524 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3525 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3526 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3527 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3528 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3529 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3530 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3531 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3532 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3533 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3534 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3535 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3536 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3537 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3538 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3539 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3540 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3541 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3542 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3543 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3544 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3545 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3546 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3547 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3548 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3549 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3550 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3551 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3552 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3553 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3554 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3555 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3556 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3557 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3558 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3559 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3560 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3561 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3562 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3563 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3564 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3565 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3566 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3567 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3568 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3569 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3570 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3571 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3572 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3573 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3574 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3575 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3576 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3577 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3578 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3579 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3580 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3581 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3582 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3583 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3584 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3585 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3586 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3587 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3588 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3589 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3590 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3591 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3592 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3593 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3594 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3595 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3596 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3597 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3598 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3599 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3600 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3601 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3602 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3603 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3604 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3605 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3606 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3607 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3608 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3609 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3610 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3611 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3612 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3613 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3614 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3615 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3616 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3617 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3618 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3619 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3620 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3621 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3622 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3623 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3624 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3625 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3626 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3627 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3628 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3629 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3630 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3631 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3632 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3633 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3634 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3635 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3636 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3637 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3638 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3639 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3640 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3641 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3642 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3643 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3644 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3645 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3646 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3647 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3648 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3649 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3650 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3651 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3652 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3653 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3654 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3655 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3656 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3657 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3658 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3659 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3660 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3661 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3662 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3663 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3664 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3665 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3666 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3667 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3668 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3669 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3670 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3671 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3672 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3673 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3674 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3675 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3676 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3677 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3678 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3679 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3680 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3681 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3682 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3683 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3684 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3685 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3686 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3687 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3688 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3689 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3690 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3691 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3692 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3693 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3694 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3695 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3696 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3697 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3698 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3699 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3700 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3701 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3702 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3703 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3704 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3705 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3706 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3707 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3708 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3709 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3710 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3711 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3712 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3713 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3714 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3715 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3716 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3717 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3718 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3719 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3720 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3721 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3722 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3723 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3724 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3725 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3726 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3727 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3728 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3729 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3730 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3731 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3732 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3733 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3734 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3735 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3736 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3737 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3738 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3739 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3740 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3741 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3742 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3743 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3744 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3745 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3746 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3747 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3748 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3749 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3750 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3751 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3752 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3753 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3754 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3755 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3756 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3757 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3758 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3759 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3760 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3761 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3762 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3763 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3764 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3765 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3766 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3767 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3768 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3769 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3770 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3771 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3772 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3773 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3774 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3775 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3776 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3777 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3778 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3779 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3780 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3781 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3782 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3783 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3784 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3785 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3786 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3787 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3788 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3789 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3790 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3791 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3792 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3793 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3794 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3795 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3796 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3797 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3798 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3799 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3800 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3801 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3802 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3803 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3804 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3805 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3806 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3807 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3808 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3809 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3810 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3811 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3812 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3813 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3814 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3815 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3816 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3817 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3818 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3819 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3820 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3821 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3822 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3823 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3824 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3825 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3826 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3827 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3828 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3829 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3830 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3831 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3832 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3833 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3834 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3835 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3836 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3837 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3838 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3839 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3840 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3841 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3842 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3843 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3844 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3845 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3846 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3847 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3848 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3849 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3850 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3851 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3852 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3853 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3854 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3855 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3856 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3857 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3858 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3859 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3860 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3861 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3862 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3863 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3864 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3865 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3866 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3867 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3868 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3869 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3870 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3871 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3872 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3873 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3874 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3875 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3876 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3877 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3878 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3879 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3880 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3881 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3882 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3883 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3884 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3885 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3886 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3887 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3888 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3889 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3890 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3891 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3892 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3893 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3894 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3895 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3896 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3897 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3898 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3899 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3900 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3901 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3902 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3903 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3904 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3905 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3906 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3907 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3908 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3909 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3910 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3911 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3912 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3913 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3914 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3915 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3916 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3917 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3918 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3919 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3920 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3921 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3922 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3923 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3924 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3925 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3926 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3927 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3928 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3929 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3930 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3931 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3932 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3933 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3934 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3935 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3936 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3937 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3938 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3939 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3940 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3941 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3942 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3943 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3944 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3945 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3946 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3947 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3948 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3949 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3950 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3951 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3952 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3953 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3954 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3955 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3956 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3957 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3958 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3959 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3960 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3961 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3962 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3963 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3964 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3965 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3966 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3967 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3968 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3969 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3970 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3971 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3972 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3973 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3974 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3975 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3976 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3977 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3978 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3979 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3980 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3981 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3982 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3983 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3984 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3985 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3986 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3987 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3988 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3989 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3990 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3991 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3992 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3993 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3994 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3995 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3996 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3997 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3998 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 3999 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 4000 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 4001 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _csignals[i + 4002 * BATCH_SIZE] = VL_RAND_RESET_I(1);
            _isignals[i + 1748 * BATCH_SIZE] = VL_RAND_RESET_I(32);
            _isignals[i + 1749 * BATCH_SIZE] = VL_RAND_RESET_I(32);
            _isignals[i + 1750 * BATCH_SIZE] = VL_RAND_RESET_I(32);
            _isignals[i + 1751 * BATCH_SIZE] = VL_RAND_RESET_I(32);
            _isignals[i + 1752 * BATCH_SIZE] = VL_RAND_RESET_I(32);
            _csignals[i + 4003 * BATCH_SIZE] = 0;
            _isignals[i + 1753 * BATCH_SIZE] = VL_RAND_RESET_I(32);
            _csignals[i + 4004 * BATCH_SIZE] = 0;
            _csignals[i + 4005 * BATCH_SIZE] = 0;
            _csignals[i + 4006 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 4007 * BATCH_SIZE] = 0;
            _csignals[i + 4008 * BATCH_SIZE] = 0;
            _csignals[i + 4009 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 4010 * BATCH_SIZE] = 0;
            _csignals[i + 4011 * BATCH_SIZE] = 0;
            _csignals[i + 4012 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 4013 * BATCH_SIZE] = 0;
            _csignals[i + 4014 * BATCH_SIZE] = 0;
            _csignals[i + 4015 * BATCH_SIZE] = VL_RAND_RESET_I(8);
            _csignals[i + 4016 * BATCH_SIZE] = 0;
        }
    }
} // end of namespace RF ========================================
