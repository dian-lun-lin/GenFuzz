#include "hip/hip_runtime.h"
#include <rtlflow.h>
#include "coverage.hpp"
#include <rf_heavy.h>

#include <chrono>
#include <iostream>
//#include <algorithm/pipeline.hpp>
#include <sstream>
#include <fstream>
#include <unordered_map>
#include <assert.h>
#include "src/adapters/tile_adapter.hpp"



void read(
  const std::filesystem::path& filename,
  std::unordered_map<unsigned long long, unsigned long long>& output
) {
  std::ifstream file(filename);
  std::stringstream ss;

  if(file) {
    ss << file.rdbuf(); 
    file.close();
    unsigned long long key, value;
    char c;

    if (ss >> c && c == '{') {
      while (ss >> key)
      {
        if ((ss >> c && c == ':') && ss >> value) {
          output[key] = value;
        }
        if (ss >> c && c != ',') {
          break;
        }
      }
    }

  }
  else {
    assert(false);
  }
}

void write(
  const std::filesystem::path& filename,
  std::unordered_map<unsigned long long, unsigned long long>& output
) {
  std::ofstream file(filename);
  std::stringstream ss;

  if(file) {
    unsigned long long key, value;
    char c;

    ss << '{';
    for(auto&& pair: output) {
      ss << '\"';  
      ss << pair.first;
      ss << '\"';
      ss << ':';
      ss << pair.second;
      ss << ',';
    }
    ss.seekp(-1, std::ios_base::end);
    ss << '}';

    file << ss.rdbuf(); 
    file.close();

  }
  else {
    assert(false);
  }
}

unsigned long long read_tohost_addr(const std::filesystem::path& filename) {
  std::ifstream file(filename);
  std::stringstream ss;
  unsigned long long addr;

  if(file) {
    ss << file.rdbuf(); 
    file.close();
    ss >> addr;
  }
  else {
    assert(false);
  }

  return addr;
}

void reset(
  RF::RTLflow& rtlflow, 
  RF::Top* dut, 
  //std::vector<gf::tileAdapter>& adapters, 
  size_t NUM_STIMULI=1
) {

  for(size_t i = 0; i < NUM_STIMULI; ++i) {
    *rtlflow.get(dut->metaReset, i) = 1;
  }

  // meta reset
  for(size_t c = 0; c < 5; ++c) {
    for(size_t i = 0; i < NUM_STIMULI; ++i) {
      *rtlflow.get(dut->clock, i) = 0;
    }
    rtlflow.run();

    for(size_t i = 0; i < NUM_STIMULI; ++i) {
      *rtlflow.get(dut->clock, i) = 1;
    }
    rtlflow.run();
  }


  for(size_t i = 0; i < NUM_STIMULI; ++i) {
    *rtlflow.get(dut->metaReset, i) = 0;
    *rtlflow.get(dut->reset, i) = 1;
  }

  // reset
  for(size_t c = 0; c < 5; ++c) {
    for(size_t i = 0; i < NUM_STIMULI; ++i) {
    *rtlflow.get(dut->clock, i) = 0;
    }
    rtlflow.run();

    for(size_t i = 0; i < NUM_STIMULI; ++i) {
    *rtlflow.get(dut->clock, i) = 1;
    }
    rtlflow.run();
  }


  for(size_t i = 0; i < NUM_STIMULI; ++i) {
    *rtlflow.get(dut->reset, i) = 0;
  }
}


int main(int argc, char** argv) {

  RF::Verilated::commandArgs(argc, argv); 
  const size_t NUM_STIMULI = std::stoi(argv[1]);
  const size_t NUM_CYCLES      = std::stoi(argv[2]);
  const std::filesystem::path ints_dir{argv[3]}; 
  const std::filesystem::path memory_dir{argv[4]}; 
  const std::filesystem::path tohost_addr_dir{argv[5]}; 
  const std::filesystem::path trace_dir{argv[6]}; 
  const std::filesystem::path output_dir{argv[7]};
  const std::filesystem::path map_file{argv[8]};

  std::vector<std::unordered_map<unsigned long long, unsigned long long>> ints(NUM_STIMULI);
  std::vector<std::unordered_map<unsigned long long, unsigned long long>> memorys(NUM_STIMULI);
  std::vector<unsigned long long> tohosts(NUM_STIMULI);
  std::vector<unsigned long long> tohost_addrs(NUM_STIMULI);

  //std::cerr << ints_dir << " " << memory_dir << " " << tohost_addr_dir << " " << trace_dir << " " << output_dir << " " << map_file << "\n";
  for(size_t i = 0; i < NUM_STIMULI; ++i) {
    read(ints_dir / (std::to_string(i) + ".ints"), ints[i]);
    read(memory_dir / (std::to_string(i) + ".mem"), memorys[i]);
    tohost_addrs[i] = read_tohost_addr(tohost_addr_dir / (std::to_string(i) + ".tohost"));
  }

  //memory.insert({tohost_addr, 0});
  //for(auto&& v: memory) {  
    //std::cerr << v.first << ": " << v.second << "\n";
  //}

  auto dut = new RF::Top;
  RF::RTLflow rtlflow(dut);
  RF::RegCoverage cov(NUM_STIMULI);

  rtlflow.initialize();

  //std::cerr << "initialize...\n";

  for(size_t i = 0; i < NUM_STIMULI; ++i) {
    *rtlflow.get(dut->clock, i) = 0;
  }
  rtlflow.run();
  for(size_t i = 0; i < NUM_STIMULI; ++i) {
    *rtlflow.get(dut->clock, i) = 1;
  }
  rtlflow.run();


  std::vector<gf::tileAdapter> adapters(NUM_STIMULI, gf::tileAdapter(rtlflow, dut));
  for(size_t i = 0; i < NUM_STIMULI; ++i) {
    adapters[i].set(i);
  }
  
  //std::cerr << "reset...\n";
  reset(rtlflow, dut, NUM_STIMULI);
  //rtlflow.load_coverage_map(cov.get_coverage_map());
  cov.load_coverage(map_file, rtlflow);

  std::vector<size_t> cycles(NUM_STIMULI);

  std::chrono::microseconds sim_duration(0);
  std::chrono::microseconds eval_duration(0);
  std::chrono::time_point<std::chrono::steady_clock> eval_tic;
  std::chrono::time_point<std::chrono::steady_clock> eval_toc;
  for(size_t i = 0; i < NUM_STIMULI; ++i) {
    adapters[i].start(memorys[i], ints[i]);
  }
  std::chrono::time_point<std::chrono::steady_clock> sim_tic = std::chrono::steady_clock::now();

  size_t all_finished{0};
  std::vector<bool> finishes(NUM_STIMULI, false);

  //std::cerr << "start simulation...\n";
  size_t c;
  for(c = 0; c < NUM_CYCLES; ++c) {
    if(c % 10000 == 0) {
      std::cout << "cycles: " << c << "\n";
    }

    for(size_t i = 0; i < NUM_STIMULI; ++i) {
      *(rtlflow.get(dut->clock, i)) = 0;
    }

    eval_tic = std::chrono::steady_clock::now();
    rtlflow.run();
    eval_toc = std::chrono::steady_clock::now();
    eval_duration +=  std::chrono::duration_cast<std::chrono::microseconds>(eval_toc - eval_tic);

    for(size_t i = 0; i < NUM_STIMULI; ++i) {
      *(rtlflow.get(dut->clock, i)) = 1;
    }

    eval_tic = std::chrono::steady_clock::now();
    rtlflow.run();
    eval_toc = std::chrono::steady_clock::now();
    eval_duration +=  std::chrono::duration_cast<std::chrono::microseconds>(eval_toc - eval_tic);
    

    for(size_t i = 0; i < NUM_STIMULI; ++i) {
      if(!finishes[i]) {
        adapters[i].drive_input(memorys[i], ints[i]);
        adapters[i].update();

        if(c % 10000 == 0) {
          auto tohost = memorys[i][tohost_addrs[i]];
          if(tohost != 0) {
            rtlflow.done[i] = true;
            finishes[i] = true;
            ++all_finished;
            
          }
          else {
            adapters[i].probe_tohost(tohost_addrs[i]);
          }
        }
      }
    }

    if(all_finished == NUM_STIMULI) {
      break;
    }
  }

  std::chrono::time_point<std::chrono::steady_clock> sim_toc = std::chrono::steady_clock::now();
  sim_duration =  std::chrono::duration_cast<std::chrono::microseconds>(sim_toc - sim_tic);

  //std::cout << "cycles: " << c << "\n";
  //std::cout << "Simulation time: " << std::chrono::duration_cast<std::chrono::microseconds>(sim_duration).count() << "\n";
  //std::cout << "Evaluation time: " << std::chrono::duration_cast<std::chrono::microseconds>(eval_duration).count() << "\n";
  //std::cout << "Driving input time: " << std::chrono::duration_cast<std::chrono::microseconds>(sim_duration - eval_duration).count() << "\n";
  //std::cerr << "finish simulation, start to terminate...\n";

  // stop for all =======================================
  std::vector<int> goings(NUM_STIMULI, -1);
  size_t finish_going{0};
  std::vector<int> runnings(NUM_STIMULI, -1);
  size_t finish_running{0};

  for(size_t i = 0; i < NUM_STIMULI; ++i) {
    adapters[i].drive_stop();
    rtlflow.done[i] = false;
  }

  //std::cerr << "stop simulation......\n";
  while(finish_going < NUM_STIMULI) {
    for(size_t i = 0; i < NUM_STIMULI; ++i) {
      if(!adapters[i].onGoing() && goings[i] == -1) {
        goings[i] = 0;
        rtlflow.done[i] = true;
      }
    }

    for(size_t i = 0; i < NUM_STIMULI; ++i) {
      *(rtlflow.get(dut->clock, i)) = 0;
    }
    rtlflow.run();
    for(size_t i = 0; i < NUM_STIMULI; ++i) {
      *(rtlflow.get(dut->clock, i)) = 1;
    }
    rtlflow.run();

    for(size_t i = 0; i < NUM_STIMULI; ++i) {
      if(goings[i] == -1) {
        adapters[i].drive_input(memorys[i], ints[i]);
        adapters[i].update();
      }
      else if(goings[i] == 0) {
        goings[i] = 1;
        ++finish_going;
      }
    }
  }

  for(size_t i = 0; i < NUM_STIMULI; ++i) {
    adapters[i].stop();
    rtlflow.done[i] = false;
  }


  while(finish_running < NUM_STIMULI) {
    for(size_t i = 0; i < NUM_STIMULI; ++i) {
      if(!adapters[i].isRunning() && runnings[i] == -1) {
        runnings[i] = 0;
        rtlflow.done[i] = true;
      }
    }

    for(size_t i = 0; i < NUM_STIMULI; ++i) {
      *(rtlflow.get(dut->clock, i)) = 0;
    }
    rtlflow.run();
    for(size_t i = 0; i < NUM_STIMULI; ++i) {
      *(rtlflow.get(dut->clock, i)) = 1;
    }
    rtlflow.run();

    for(size_t i = 0; i < NUM_STIMULI; ++i) {
      if(runnings[i] == -1) {
        adapters[i].drive_input(memorys[i], ints[i]);
        adapters[i].update();
      }
      else if(runnings[i] == 0) {
        runnings[i] = 1;
        ++finish_running;
      }
    }
  }
  

  for(size_t i = 0; i < NUM_STIMULI; ++i) {
    adapters[i].clear();
  }

  //std::cerr << "caculate_coverage\n";
  cov.caculate_coverage(rtlflow);
  for(size_t i = 0; i < NUM_STIMULI; ++i) {
    write(output_dir / (std::to_string(i) + ".mem"), memorys[i]);
    cov.save_one_coverage(output_dir / (std::to_string(i) + ".cov"), i);
  }

  cov.save_total_coverage(output_dir / "union.cov");
}
