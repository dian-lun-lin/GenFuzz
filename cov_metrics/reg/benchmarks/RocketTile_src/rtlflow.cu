#include "hip/hip_runtime.h"

#include <taskflow.hpp>

#include <cuda/algorithm/reduce.hpp>

#include "rtlflow.h"


#include "Top.h"


#include <hiprand.h>

#include <assert.h>

// begin of namespace RF =====================================
namespace RF {
    inline
    hipError_t checkCuda(hipError_t result) {
        if (result != hipSuccess) {
            using namespace std::literals::string_literals;
            throw std::runtime_error("CUDA Runtime Error: "s + hipGetErrorString(result));
        }
        return result;
    }
    
    __global__ void _eval_settle(Top__Syms* __restrict vlSymsp, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals);
    
    RTLflow::RTLflow(Top* dut): dut{dut} {
        checkCuda(hipMallocManaged(&_csignals, BATCH_SIZE * cuda_cmem_size * sizeof(CData)));
        checkCuda(hipMallocManaged(&_ssignals, BATCH_SIZE * cuda_smem_size * sizeof(SData)));
        checkCuda(hipMallocManaged(&_qsignals, BATCH_SIZE * cuda_qmem_size * sizeof(QData)));
        checkCuda(hipMallocManaged(&_isignals, BATCH_SIZE * cuda_imem_size * sizeof(IData)));
        checkCuda(hipMallocManaged(&change, BATCH_SIZE * sizeof(IData)));
        checkCuda(hipMallocManaged(&done, BATCH_SIZE * sizeof(bool)));
        checkCuda(hipMemset(change, 1, BATCH_SIZE * sizeof(IData)));
        checkCuda(hipMemset(done, 0, BATCH_SIZE * sizeof(bool)));
    }
    RTLflow::RTLflow(const RTLflow& rtlflow) {
        dut = rtlflow.dut;
        checkCuda(hipMallocManaged(&_csignals, BATCH_SIZE * cuda_cmem_size * sizeof(CData)));
        checkCuda(hipMallocManaged(&_ssignals, BATCH_SIZE * cuda_smem_size * sizeof(SData)));
        checkCuda(hipMallocManaged(&_qsignals, BATCH_SIZE * cuda_qmem_size * sizeof(QData)));
        checkCuda(hipMallocManaged(&_isignals, BATCH_SIZE * cuda_imem_size * sizeof(IData)));
        checkCuda(hipMallocManaged(&change, BATCH_SIZE * sizeof(IData)));
        checkCuda(hipMallocManaged(&done, BATCH_SIZE * sizeof(bool)));
        checkCuda(hipMemset(change, 1, BATCH_SIZE * sizeof(IData)));
        checkCuda(hipMemset(done, 0, BATCH_SIZE * sizeof(bool)));
    }
    RTLflow::~RTLflow() {
        checkCuda(hipFree(_csignals));
        checkCuda(hipFree(_ssignals));
        checkCuda(hipFree(_qsignals));
        checkCuda(hipFree(_isignals));
        checkCuda(hipFree(change));
        checkCuda(hipFree(done));
    }
    
    // idx: index of testbenches
    CData* RTLflow::get(CDataLoc cdl, size_t idx) {
        return _csignals + idx * cdl.size + cdl.memloc;
    }
    SData* RTLflow::get(SDataLoc sdl, size_t idx) {
        return _ssignals + idx * sdl.size + sdl.memloc;
    }
    QData* RTLflow::get(QDataLoc qdl, size_t idx) {
        return _qsignals + idx * qdl.size + qdl.memloc;
    }
    IData* RTLflow::get(IDataLoc idl, size_t idx) {
        return _isignals + idx * idl.size + idl.memloc;
    }
    
    void RTLflow::_randomize() {
        hiprandGenerator_t generator;
        hiprandCreateGenerator(&generator,HIPRAND_RNG_PSEUDO_XORWOW);
        hiprandSetPseudoRandomGeneratorSeed(generator,(int)time(NULL));
        hiprandGenerate(generator, (unsigned int*)_csignals, BATCH_SIZE * cuda_cmem_size / 4);
        hiprandGenerate(generator, (unsigned int*)_ssignals, BATCH_SIZE * cuda_smem_size / 2);
        hiprandGenerate(generator, (unsigned int*)_isignals, BATCH_SIZE * cuda_imem_size);
        hiprandGenerate(generator, (unsigned int*)_qsignals, BATCH_SIZE * cuda_qmem_size * 2);
    }
    
    void RTLflow::run() { _executor.run(_taskflow).wait(); }
    
    tf::Taskflow& RTLflow::taskflow() { return _taskflow; }
    
    void RTLflow::_ctor_var_reset(Top__Syms* VlSymsp) {
        size_t offsetc{0};
        size_t offsets{0};
        size_t offseti{0};
        size_t offsetq{0};
        offsetc = 0;
        offsets = 0;
        offseti = 0;
        offsetq = 0;
        VlSymsp->TOPp->_ctor_var_reset(_csignals + offsetc, _ssignals + offsets, _isignals + offseti, _qsignals + offsetq);
    }
    
    void RTLflow::initialize() {
        Top__Syms* VlSymsp = dut-> __VlSymsp;
         _ctor_var_reset(VlSymsp);
        const size_t num_threads = (BATCH_SIZE < 128) ? BATCH_SIZE : 128;
        const size_t num_blocks = (num_threads < 128) ? 1 : BATCH_SIZE / num_threads;
        auto reset_cut = _cudaflow.memset(change, 1, sizeof(IData) * BATCH_SIZE);
        auto change_cut = _cudaflow.kernel(num_blocks, num_threads, 0, _change_request, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change);
        auto last_assign_cut = _cudaflow.kernel(num_blocks, num_threads, 0, _last_assign, VlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        auto reduce_cut = _cudaflow.reduce(change, change + BATCH_SIZE, change, [] __device__ (IData a, IData b){ return a | b; });
        last_assign_cut.precede(change_cut);
        
        change_cut.precede(reduce_cut);
        
        auto id_3_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__3, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_3");
        auto id_4_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__4, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_4");
        auto id_5_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__5, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_5");
        auto id_8_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__8, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_8");
        auto id_13_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__13, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_13");
        auto id_14_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__14, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_14");
        auto id_16_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__16, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_16");
        auto id_19_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__19, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_19");
        auto id_20_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__20, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_20");
        auto id_21_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__21, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_21");
        auto id_22_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__22, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_22");
        auto id_23_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__23, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_23");
        auto id_24_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__24, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_24");
        auto id_25_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__25, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_25");
        auto id_26_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__26, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_26");
        auto id_27_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__27, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_27");
        auto id_28_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__28, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_28");
        auto id_36_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__36, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_36");
        auto id_39_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__39, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_39");
        auto id_40_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__40, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_40");
        auto id_41_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__41, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_41");
        auto id_42_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__42, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_42");
        auto id_53_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__53, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_53");
        auto id_54_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__54, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_54");
        auto id_55_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__55, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_55");
        auto id_67_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__67, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_67");
        auto id_86_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__86, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_86");
        auto id_2_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__2, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_2");
        auto id_9_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__9, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_9");
        auto id_17_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__17, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_17");
        auto id_29_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__29, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_29");
        auto id_34_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__34, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_34");
        auto id_37_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__37, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_37");
        auto id_38_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__38, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_38");
        auto id_46_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__46, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_46");
        auto id_50_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__50, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_50");
        auto id_51_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__51, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_51");
        auto id_58_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__58, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_58");
        auto id_59_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__59, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_59");
        auto id_65_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__65, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_65");
        auto id_69_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__69, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_69");
        auto id_99_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__99, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_99");
        auto id_116_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__116, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_116");
        auto id_11_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__11, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_11");
        auto id_15_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__15, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_15");
        auto id_18_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__18, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_18");
        auto id_35_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__35, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_35");
        auto id_45_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__45, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_45");
        auto id_48_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__48, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_48");
        auto id_49_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__49, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_49");
        auto id_52_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__52, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_52");
        auto id_62_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__62, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_62");
        auto id_64_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__64, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_64");
        auto id_70_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__70, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_70");
        auto id_76_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__76, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_76");
        auto id_79_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__79, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_79");
        auto id_81_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__81, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_81");
        auto id_92_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__92, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_92");
        auto id_93_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__93, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_93");
        auto id_96_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__96, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_96");
        auto id_102_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__102, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_102");
        auto id_10_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__10, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_10");
        auto id_12_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__12, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_12");
        auto id_33_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__33, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_33");
        auto id_47_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__47, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_47");
        auto id_57_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__57, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_57");
        auto id_71_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__71, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_71");
        auto id_72_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__72, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_72");
        auto id_73_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__73, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_73");
        auto id_84_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__84, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_84");
        auto id_98_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__98, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_98");
        auto id_104_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__104, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_104");
        auto id_108_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__108, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_108");
        auto id_114_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__114, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_114");
        auto id_30_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__30, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_30");
        auto id_31_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__31, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_31");
        auto id_43_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__43, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_43");
        auto id_44_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__44, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_44");
        auto id_56_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__56, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_56");
        auto id_60_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__60, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_60");
        auto id_66_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__66, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_66");
        auto id_68_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__68, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_68");
        auto id_75_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__75, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_75");
        auto id_77_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__77, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_77");
        auto id_80_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__80, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_80");
        auto id_83_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__83, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_83");
        auto id_85_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__85, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_85");
        auto id_89_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__89, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_89");
        auto id_90_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__90, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_90");
        auto id_91_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__91, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_91");
        auto id_94_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__94, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_94");
        auto id_95_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__95, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_95");
        auto id_97_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__97, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_97");
        auto id_100_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__100, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_100");
        auto id_103_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__103, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_103");
        auto id_105_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__105, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_105");
        auto id_109_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__109, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_109");
        auto id_111_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__111, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_111");
        auto id_112_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__112, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_112");
        auto id_113_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__113, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_113");
        auto id_122_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__122, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_122");
        auto id_125_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__125, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_125");
        auto id_61_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__61, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_61");
        auto id_63_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__63, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_63");
        auto id_78_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__78, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_78");
        auto id_87_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__87, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_87");
        auto id_88_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__88, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_88");
        auto id_101_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__101, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_101");
        auto id_107_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__107, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_107");
        auto id_121_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__121, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_121");
        auto id_123_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__123, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_123");
        auto id_124_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__124, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_124");
        auto id_128_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__128, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_128");
        auto id_129_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__129, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_129");
        auto id_74_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__74, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_74");
        auto id_82_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__82, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_82");
        auto id_110_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__110, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_110");
        auto id_115_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__115, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_115");
        auto id_117_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__117, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_117");
        auto id_118_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__118, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_118");
        auto id_119_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__119, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_119");
        auto id_120_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__120, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_120");
        auto id_126_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__126, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_126");
        auto id_127_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__127, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_127");
        auto id_106_cut = _cudaflow.kernel(num_blocks, num_threads, 0, __Vmtask__106, VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change, done).name("task_106");
        id_3_cut.precede(id_30_cut);
        id_3_cut.precede(id_31_cut);
        id_3_cut.precede(id_85_cut);
        reset_cut.precede(id_3_cut);
        id_4_cut.precede(id_72_cut);
        id_4_cut.precede(id_73_cut);
        id_4_cut.precede(id_98_cut);
        reset_cut.precede(id_4_cut);
        id_5_cut.precede(id_14_cut);
        id_5_cut.precede(id_19_cut);
        id_5_cut.precede(id_22_cut);
        id_5_cut.precede(id_23_cut);
        id_5_cut.precede(id_24_cut);
        id_5_cut.precede(id_25_cut);
        id_5_cut.precede(id_26_cut);
        id_5_cut.precede(id_27_cut);
        id_5_cut.precede(id_28_cut);
        id_5_cut.precede(id_53_cut);
        id_5_cut.precede(id_54_cut);
        id_5_cut.precede(id_67_cut);
        id_5_cut.precede(id_9_cut);
        reset_cut.precede(id_5_cut);
        id_8_cut.precede(id_13_cut);
        id_8_cut.precede(id_14_cut);
        id_8_cut.precede(id_16_cut);
        id_8_cut.precede(id_19_cut);
        id_8_cut.precede(id_21_cut);
        id_8_cut.precede(id_22_cut);
        id_8_cut.precede(id_23_cut);
        id_8_cut.precede(id_24_cut);
        id_8_cut.precede(id_27_cut);
        id_8_cut.precede(id_41_cut);
        id_8_cut.precede(id_53_cut);
        id_8_cut.precede(id_54_cut);
        id_8_cut.precede(id_67_cut);
        id_8_cut.precede(id_42_cut);
        id_8_cut.precede(id_9_cut);
        reset_cut.precede(id_8_cut);
        id_13_cut.precede(id_45_cut);
        id_14_cut.precede(id_46_cut);
        id_14_cut.precede(id_59_cut);
        id_14_cut.precede(id_69_cut);
        id_14_cut.precede(id_15_cut);
        id_14_cut.precede(id_35_cut);
        id_16_cut.precede(id_36_cut);
        id_16_cut.precede(id_50_cut);
        id_16_cut.precede(id_11_cut);
        id_19_cut.precede(id_20_cut);
        id_19_cut.precede(id_39_cut);
        id_19_cut.precede(id_40_cut);
        id_19_cut.precede(id_86_cut);
        id_19_cut.precede(id_2_cut);
        id_19_cut.precede(id_29_cut);
        id_19_cut.precede(id_34_cut);
        id_19_cut.precede(id_37_cut);
        id_19_cut.precede(id_38_cut);
        id_19_cut.precede(id_11_cut);
        id_20_cut.precede(id_52_cut);
        id_20_cut.precede(id_33_cut);
        id_20_cut.precede(id_31_cut);
        id_21_cut.precede(id_36_cut);
        id_21_cut.precede(id_50_cut);
        id_21_cut.precede(id_11_cut);
        id_22_cut.precede(id_39_cut);
        id_22_cut.precede(id_29_cut);
        id_22_cut.precede(id_37_cut);
        id_23_cut.precede(id_29_cut);
        id_23_cut.precede(id_46_cut);
        id_23_cut.precede(id_69_cut);
        id_23_cut.precede(id_99_cut);
        id_23_cut.precede(id_15_cut);
        id_24_cut.precede(id_29_cut);
        id_24_cut.precede(id_50_cut);
        id_24_cut.precede(id_11_cut);
        id_25_cut.precede(id_40_cut);
        id_25_cut.precede(id_29_cut);
        id_25_cut.precede(id_49_cut);
        id_26_cut.precede(id_39_cut);
        id_26_cut.precede(id_40_cut);
        id_26_cut.precede(id_29_cut);
        id_26_cut.precede(id_59_cut);
        id_27_cut.precede(id_39_cut);
        id_27_cut.precede(id_29_cut);
        id_27_cut.precede(id_37_cut);
        id_27_cut.precede(id_15_cut);
        id_28_cut.precede(id_39_cut);
        id_28_cut.precede(id_29_cut);
        id_28_cut.precede(id_59_cut);
        id_36_cut.precede(id_51_cut);
        id_36_cut.precede(id_58_cut);
        id_36_cut.precede(id_65_cut);
        id_36_cut.precede(id_35_cut);
        id_36_cut.precede(id_45_cut);
        id_39_cut.precede(id_92_cut);
        id_39_cut.precede(id_93_cut);
        id_39_cut.precede(id_71_cut);
        id_40_cut.precede(id_76_cut);
        id_40_cut.precede(id_93_cut);
        id_41_cut.precede(id_34_cut);
        id_41_cut.precede(id_37_cut);
        id_41_cut.precede(id_38_cut);
        id_41_cut.precede(id_11_cut);
        id_41_cut.precede(id_15_cut);
        id_41_cut.precede(id_79_cut);
        id_42_cut.precede(id_99_cut);
        id_42_cut.precede(id_68_cut);
        id_53_cut.precede(id_55_cut);
        id_53_cut.precede(id_17_cut);
        id_53_cut.precede(id_34_cut);
        id_53_cut.precede(id_50_cut);
        id_54_cut.precede(id_38_cut);
        id_54_cut.precede(id_50_cut);
        id_55_cut.precede(id_59_cut);
        id_55_cut.precede(id_69_cut);
        id_55_cut.precede(id_15_cut);
        id_67_cut.precede(id_29_cut);
        id_67_cut.precede(id_38_cut);
        id_67_cut.precede(id_11_cut);
        id_86_cut.precede(id_45_cut);
        id_86_cut.precede(id_52_cut);
        id_86_cut.precede(id_64_cut);
        id_2_cut.precede(id_33_cut);
        id_2_cut.precede(id_31_cut);
        id_9_cut.precede(id_52_cut);
        id_17_cut.precede(id_51_cut);
        id_17_cut.precede(id_59_cut);
        id_17_cut.precede(id_65_cut);
        id_17_cut.precede(id_69_cut);
        id_17_cut.precede(id_11_cut);
        id_17_cut.precede(id_15_cut);
        id_29_cut.precede(id_76_cut);
        id_29_cut.precede(id_71_cut);
        id_34_cut.precede(id_52_cut);
        id_34_cut.precede(id_64_cut);
        id_34_cut.precede(id_93_cut);
        id_34_cut.precede(id_33_cut);
        id_37_cut.precede(id_46_cut);
        id_37_cut.precede(id_59_cut);
        id_37_cut.precede(id_69_cut);
        id_38_cut.precede(id_51_cut);
        id_38_cut.precede(id_59_cut);
        id_38_cut.precede(id_69_cut);
        id_46_cut.precede(id_45_cut);
        id_46_cut.precede(id_49_cut);
        id_50_cut.precede(id_51_cut);
        id_50_cut.precede(id_58_cut);
        id_50_cut.precede(id_59_cut);
        id_50_cut.precede(id_65_cut);
        id_50_cut.precede(id_35_cut);
        id_51_cut.precede(id_99_cut);
        id_51_cut.precede(id_52_cut);
        id_51_cut.precede(id_62_cut);
        id_51_cut.precede(id_70_cut);
        id_51_cut.precede(id_93_cut);
        id_58_cut.precede(id_52_cut);
        id_58_cut.precede(id_62_cut);
        id_58_cut.precede(id_70_cut);
        id_59_cut.precede(id_99_cut);
        id_59_cut.precede(id_18_cut);
        id_59_cut.precede(id_45_cut);
        id_59_cut.precede(id_48_cut);
        id_59_cut.precede(id_49_cut);
        id_59_cut.precede(id_81_cut);
        id_59_cut.precede(id_10_cut);
        id_65_cut.precede(id_52_cut);
        id_65_cut.precede(id_62_cut);
        id_65_cut.precede(id_70_cut);
        id_65_cut.precede(id_79_cut);
        id_69_cut.precede(id_45_cut);
        id_69_cut.precede(id_52_cut);
        id_69_cut.precede(id_81_cut);
        id_69_cut.precede(id_10_cut);
        id_99_cut.precede(id_116_cut);
        id_99_cut.precede(id_96_cut);
        id_99_cut.precede(id_44_cut);
        id_116_cut.precede(id_109_cut);
        id_116_cut.precede(id_122_cut);
        id_11_cut.precede(id_52_cut);
        id_11_cut.precede(id_70_cut);
        id_11_cut.precede(id_93_cut);
        id_11_cut.precede(id_33_cut);
        id_11_cut.precede(id_72_cut);
        id_15_cut.precede(id_64_cut);
        id_15_cut.precede(id_81_cut);
        id_15_cut.precede(id_10_cut);
        id_18_cut.precede(id_76_cut);
        id_18_cut.precede(id_93_cut);
        id_18_cut.precede(id_57_cut);
        id_18_cut.precede(id_71_cut);
        id_35_cut.precede(id_52_cut);
        id_35_cut.precede(id_62_cut);
        id_35_cut.precede(id_70_cut);
        id_35_cut.precede(id_79_cut);
        id_35_cut.precede(id_33_cut);
        id_45_cut.precede(id_93_cut);
        id_45_cut.precede(id_96_cut);
        id_45_cut.precede(id_33_cut);
        id_48_cut.precede(id_57_cut);
        id_49_cut.precede(id_76_cut);
        id_49_cut.precede(id_93_cut);
        id_49_cut.precede(id_33_cut);
        id_52_cut.precede(id_57_cut);
        id_52_cut.precede(id_71_cut);
        id_52_cut.precede(id_104_cut);
        id_52_cut.precede(id_61_cut);
        id_62_cut.precede(id_71_cut);
        id_64_cut.precede(id_12_cut);
        id_64_cut.precede(id_57_cut);
        id_64_cut.precede(id_30_cut);
        id_64_cut.precede(id_31_cut);
        id_64_cut.precede(id_77_cut);
        id_64_cut.precede(id_85_cut);
        id_70_cut.precede(id_84_cut);
        id_70_cut.precede(id_43_cut);
        id_70_cut.precede(id_77_cut);
        id_76_cut.precede(id_92_cut);
        id_76_cut.precede(id_12_cut);
        id_76_cut.precede(id_104_cut);
        id_79_cut.precede(id_84_cut);
        id_79_cut.precede(id_43_cut);
        id_81_cut.precede(id_93_cut);
        id_81_cut.precede(id_12_cut);
        id_81_cut.precede(id_57_cut);
        id_81_cut.precede(id_72_cut);
        id_81_cut.precede(id_73_cut);
        id_81_cut.precede(id_98_cut);
        id_92_cut.precede(id_43_cut);
        id_92_cut.precede(id_44_cut);
        id_92_cut.precede(id_60_cut);
        id_92_cut.precede(id_80_cut);
        id_93_cut.precede(id_102_cut);
        id_93_cut.precede(id_104_cut);
        id_93_cut.precede(id_56_cut);
        id_93_cut.precede(id_68_cut);
        id_93_cut.precede(id_90_cut);
        id_93_cut.precede(id_112_cut);
        id_96_cut.precede(id_114_cut);
        id_96_cut.precede(id_89_cut);
        id_96_cut.precede(id_90_cut);
        id_102_cut.precede(id_91_cut);
        id_10_cut.precede(id_12_cut);
        id_10_cut.precede(id_57_cut);
        id_10_cut.precede(id_104_cut);
        id_12_cut.precede(id_114_cut);
        id_12_cut.precede(id_56_cut);
        id_12_cut.precede(id_80_cut);
        id_12_cut.precede(id_83_cut);
        id_12_cut.precede(id_105_cut);
        id_33_cut.precede(id_47_cut);
        id_33_cut.precede(id_57_cut);
        id_33_cut.precede(id_71_cut);
        id_33_cut.precede(id_98_cut);
        id_33_cut.precede(id_114_cut);
        id_47_cut.precede(id_43_cut);
        id_47_cut.precede(id_56_cut);
        id_47_cut.precede(id_80_cut);
        id_47_cut.precede(id_85_cut);
        id_47_cut.precede(id_94_cut);
        id_57_cut.precede(id_43_cut);
        id_57_cut.precede(id_56_cut);
        id_57_cut.precede(id_83_cut);
        id_57_cut.precede(id_94_cut);
        id_57_cut.precede(id_97_cut);
        id_57_cut.precede(id_111_cut);
        id_57_cut.precede(id_113_cut);
        id_71_cut.precede(id_84_cut);
        id_71_cut.precede(id_43_cut);
        id_71_cut.precede(id_44_cut);
        id_71_cut.precede(id_56_cut);
        id_71_cut.precede(id_60_cut);
        id_71_cut.precede(id_68_cut);
        id_71_cut.precede(id_91_cut);
        id_72_cut.precede(id_30_cut);
        id_72_cut.precede(id_85_cut);
        id_73_cut.precede(id_30_cut);
        id_73_cut.precede(id_31_cut);
        id_73_cut.precede(id_85_cut);
        id_84_cut.precede(last_assign_cut);
        id_98_cut.precede(id_30_cut);
        id_98_cut.precede(id_85_cut);
        id_104_cut.precede(id_108_cut);
        id_104_cut.precede(id_80_cut);
        id_104_cut.precede(id_91_cut);
        id_104_cut.precede(id_94_cut);
        id_108_cut.precede(id_100_cut);
        id_114_cut.precede(id_94_cut);
        id_114_cut.precede(id_100_cut);
        id_114_cut.precede(id_103_cut);
        id_30_cut.precede(id_66_cut);
        id_30_cut.precede(id_75_cut);
        id_30_cut.precede(id_100_cut);
        id_31_cut.precede(id_66_cut);
        id_43_cut.precede(id_103_cut);
        id_43_cut.precede(id_115_cut);
        id_44_cut.precede(id_77_cut);
        id_44_cut.precede(id_89_cut);
        id_44_cut.precede(id_90_cut);
        id_44_cut.precede(id_95_cut);
        id_44_cut.precede(id_97_cut);
        id_56_cut.precede(id_128_cut);
        id_60_cut.precede(id_97_cut);
        id_60_cut.precede(id_103_cut);
        id_60_cut.precede(id_113_cut);
        id_60_cut.precede(id_129_cut);
        id_66_cut.precede(id_83_cut);
        id_66_cut.precede(id_112_cut);
        id_66_cut.precede(id_61_cut);
        id_68_cut.precede(id_89_cut);
        id_68_cut.precede(id_95_cut);
        id_68_cut.precede(id_109_cut);
        id_75_cut.precede(id_112_cut);
        id_75_cut.precede(id_61_cut);
        id_77_cut.precede(id_94_cut);
        id_77_cut.precede(id_103_cut);
        id_77_cut.precede(id_109_cut);
        id_77_cut.precede(id_111_cut);
        id_80_cut.precede(id_100_cut);
        id_80_cut.precede(id_63_cut);
        id_80_cut.precede(id_129_cut);
        id_83_cut.precede(id_63_cut);
        id_83_cut.precede(id_78_cut);
        id_83_cut.precede(id_107_cut);
        id_83_cut.precede(id_121_cut);
        id_83_cut.precede(id_123_cut);
        id_85_cut.precede(id_112_cut);
        id_85_cut.precede(id_61_cut);
        id_89_cut.precede(id_94_cut);
        id_89_cut.precede(id_100_cut);
        id_89_cut.precede(id_103_cut);
        id_89_cut.precede(id_111_cut);
        id_90_cut.precede(id_94_cut);
        id_90_cut.precede(id_100_cut);
        id_90_cut.precede(id_103_cut);
        id_90_cut.precede(id_109_cut);
        id_90_cut.precede(id_122_cut);
        id_91_cut.precede(id_97_cut);
        id_91_cut.precede(id_100_cut);
        id_91_cut.precede(id_105_cut);
        id_91_cut.precede(id_113_cut);
        id_91_cut.precede(id_63_cut);
        id_94_cut.precede(id_125_cut);
        id_94_cut.precede(id_118_cut);
        id_95_cut.precede(id_103_cut);
        id_97_cut.precede(id_101_cut);
        id_100_cut.precede(id_120_cut);
        id_103_cut.precede(id_125_cut);
        id_103_cut.precede(id_101_cut);
        id_103_cut.precede(id_74_cut);
        id_105_cut.precede(id_124_cut);
        id_105_cut.precede(id_115_cut);
        id_109_cut.precede(id_125_cut);
        id_111_cut.precede(id_124_cut);
        id_112_cut.precede(id_63_cut);
        id_112_cut.precede(id_78_cut);
        id_112_cut.precede(id_123_cut);
        id_113_cut.precede(last_assign_cut);
        id_122_cut.precede(id_125_cut);
        id_125_cut.precede(last_assign_cut);
        id_61_cut.precede(id_78_cut);
        id_61_cut.precede(id_107_cut);
        id_61_cut.precede(id_121_cut);
        id_61_cut.precede(id_123_cut);
        id_63_cut.precede(id_128_cut);
        id_63_cut.precede(id_74_cut);
        id_63_cut.precede(id_115_cut);
        id_63_cut.precede(id_117_cut);
        id_63_cut.precede(id_119_cut);
        id_63_cut.precede(id_120_cut);
        id_78_cut.precede(id_87_cut);
        id_78_cut.precede(id_88_cut);
        id_78_cut.precede(id_82_cut);
        id_78_cut.precede(id_119_cut);
        id_78_cut.precede(id_127_cut);
        id_87_cut.precede(id_126_cut);
        id_87_cut.precede(id_106_cut);
        id_88_cut.precede(id_126_cut);
        id_101_cut.precede(last_assign_cut);
        id_107_cut.precede(id_82_cut);
        id_107_cut.precede(id_110_cut);
        id_107_cut.precede(id_120_cut);
        id_121_cut.precede(id_74_cut);
        id_121_cut.precede(id_110_cut);
        id_121_cut.precede(id_118_cut);
        id_121_cut.precede(id_119_cut);
        id_123_cut.precede(id_124_cut);
        id_123_cut.precede(id_82_cut);
        id_123_cut.precede(id_115_cut);
        id_123_cut.precede(id_117_cut);
        id_123_cut.precede(id_127_cut);
        id_124_cut.precede(id_128_cut);
        id_124_cut.precede(id_129_cut);
        id_124_cut.precede(id_74_cut);
        id_124_cut.precede(id_118_cut);
        id_124_cut.precede(id_119_cut);
        id_124_cut.precede(id_120_cut);
        id_128_cut.precede(last_assign_cut);
        id_129_cut.precede(last_assign_cut);
        id_74_cut.precede(last_assign_cut);
        id_82_cut.precede(last_assign_cut);
        id_110_cut.precede(id_115_cut);
        id_110_cut.precede(id_127_cut);
        id_110_cut.precede(id_106_cut);
        id_115_cut.precede(last_assign_cut);
        id_117_cut.precede(id_126_cut);
        id_118_cut.precede(last_assign_cut);
        id_119_cut.precede(last_assign_cut);
        id_120_cut.precede(last_assign_cut);
        id_126_cut.precede(last_assign_cut);
        id_127_cut.precede(last_assign_cut);
        id_106_cut.precede(last_assign_cut);
        auto start_t = _taskflow.emplace([=](){
                if(VL_UNLIKELY(!init)) {
                    Top::_eval_initial(VlSymsp, _csignals, _ssignals, _isignals, _qsignals);
                    int device;
                    checkCuda(hipGetDevice(&device));
                    checkCuda(hipMemPrefetchAsync(_csignals, BATCH_SIZE * cuda_cmem_size * sizeof(CData), device));
                    checkCuda(hipMemPrefetchAsync(_ssignals, BATCH_SIZE * cuda_smem_size * sizeof(SData), device));
                    checkCuda(hipMemPrefetchAsync(_isignals, BATCH_SIZE * cuda_imem_size * sizeof(IData), device));
                    checkCuda(hipMemPrefetchAsync(_qsignals, BATCH_SIZE * cuda_qmem_size * sizeof(QData), device));
                    checkCuda(hipMemPrefetchAsync(change, BATCH_SIZE * sizeof(IData), device));
                    checkCuda(hipMemPrefetchAsync(done, BATCH_SIZE * sizeof(bool), device));
                    init = true;
                    return 0;
                }
                else {
                    return 1;
                }
        });
        
        auto init_detect_t = _taskflow.emplace([=](){
                if(++loop > 100) {
                    _change_request<<<num_blocks, num_threads, 0>>>(VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change);
                    checkCuda(hipDeviceSynchronize());
                    VL_FATAL_MT("add.v", 2, "",
                        "Verilated model didn't converge"
                        "- See https://verilator.org/warn/DIDNOTCONVERGE");
                }
                return (bool)change[0];
        });
        auto init_sim_t = _taskflow.emplace([=](){
                _eval_settle<<<num_blocks, num_threads, 0>>>(VlSymsp, _csignals, _ssignals, _isignals, _qsignals);
                checkCuda(hipDeviceSynchronize());
                _cudaflow.offload();
        });
        auto sim_t = _taskflow.emplace([=](){
                _cudaflow.offload();
        });
        auto end_t = _taskflow.emplace([=](){
                loop = 0;
        });
        
        auto detect_t = _taskflow.emplace([=](){
                if(++loop > 100) {
                    _change_request<<<num_blocks, num_threads, 0>>>(VlSymsp, _csignals, _ssignals, _isignals, _qsignals, change);
                    checkCuda(hipDeviceSynchronize());
                    VL_FATAL_MT("add.v", 2, "",
                        "Verilated model didn't converge"
                        "- See https://verilator.org/warn/DIDNOTCONVERGE");
                }
                return (bool)change[0];
        });
        start_t.precede(init_sim_t, sim_t);
        init_sim_t.precede(init_detect_t);
        init_detect_t.precede(end_t, init_sim_t);
        
        sim_t.precede(detect_t);
        detect_t.precede(end_t, sim_t);
        std::ofstream ofs("./cudaflow.out");
        _cudaflow.dump(ofs);
    }
} // end of namespace RF ==================================== 
