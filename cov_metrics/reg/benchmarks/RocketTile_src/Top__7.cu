#include "hip/hip_runtime.h"
// Verilated -*- C++ -*-
// DESCRIPTION: Verilator output: Design implementation internals
// See Top.h for the primary calling header

#include "Top.h"
#include "Top__Syms.h"
// begin of namespace RF =====================================
namespace RF {

    //==========

    __global__
    void __Vmtask__44(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__129(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__56(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__130(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__131(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__132(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__133(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__134(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__60(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__135(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__66(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__136(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__137(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__138(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__139(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__140(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__68(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__141(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__75(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__142(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__77(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__143(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__144(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__145(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__146(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__147(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__148(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__149(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__150(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__151(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__80(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__152(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__153(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__154(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__155(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__156(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__83(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__157(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__158(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__159(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__160(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__161(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__162(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__163(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__85(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__164(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__165(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__166(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__167(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__168(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__89(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__169(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__90(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__170(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__91(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__171(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__172(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__173(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__94(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__174(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__175(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__176(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__177(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__178(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__179(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__180(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__95(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__181(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__97(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__182(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__100(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__183(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__103(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__184(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__185(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__186(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__187(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__188(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__189(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__105(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__190(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__191(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        vlTOPp->_combo__TOP__193(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__194(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__109(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__195(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__111(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__196(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__112(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__197(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__198(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__199(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__200(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__201(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__113(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__202(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__122(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__203(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__125(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__204(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__205(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__61(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        vlTOPp->_combo__TOP__207(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__208(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__209(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__210(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__63(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__211(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__212(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__213(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__78(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        vlTOPp->_combo__TOP__214(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__215(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__216(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__217(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__218(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__219(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__220(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__87(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__221(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__222(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__88(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        vlTOPp->_combo__TOP__223(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__101(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__224(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__225(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__226(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__107(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        vlTOPp->_combo__TOP__228(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__229(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__230(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__231(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__121(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__232(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__233(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__234(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__235(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__123(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__236(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__124(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__237(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__128(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__238(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__129(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__239(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__74(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__240(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__82(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        vlTOPp->_combo__TOP__241(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__242(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__110(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        vlTOPp->_combo__TOP__243(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__244(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__245(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__115(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__246(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__247(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__248(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__249(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__250(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__251(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__252(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__253(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__254(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__255(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__256(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__117(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__257(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__258(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__118(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__259(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__260(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__261(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__262(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__263(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__264(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__265(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__119(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__266(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__267(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__268(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__120(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        vlTOPp->_combo__TOP__269(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__270(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__271(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__272(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__273(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__274(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
    }

    __global__
    void __Vmtask__126(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        vlTOPp->_combo__TOP__275(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__127(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        vlTOPp->_combo__TOP__276(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        if (((IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 10505955]) 
             & (~ (IData)(_csignals[(blockDim.x * blockIdx.x + threadIdx.x) + BATCH_SIZE * 21011909])))) {
            vlTOPp->_sequent__TOP__277(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
        }
        vlTOPp->_combo__TOP__278(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }

    __global__
    void __Vmtask__106(void* symtab, CData* _csignals, SData* _ssignals, IData* _isignals, QData* _qsignals, IData* change, bool* done) {
        if(done[blockDim.x * blockIdx.x + threadIdx.x] || !change[blockDim.x * blockIdx.x + threadIdx.x]) return;
        Top__Syms* __restrict vlSymsp = (Top__Syms*)symtab;
        Top* const __restrict vlTOPp VL_ATTR_UNUSED = vlSymsp->TOPp;
        vlTOPp->_combo__TOP__279(vlSymsp, _csignals, _ssignals, _isignals, _qsignals);
    }
} // end of namespace RF ========================================
